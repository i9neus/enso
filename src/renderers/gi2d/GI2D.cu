#include "hip/hip_runtime.h"
#include "GI2D.cuh"
#include "generic/Math.h"
#include "kernels/CudaVector.cuh"
#include "kernels/gi2d/CudaGI2DOverlay.cuh"
#include "kernels/gi2d/CudaPrimitive2D.cuh"

using namespace Cuda;

// All renderer objects must be declared and stored in the source file to avoid compilation errors between nvcc and Visual Studio
struct CudaObjects
{
    Cuda::AssetHandle<Cuda::Host::GI2DOverlay>                  overlayRenderer;
    Cuda::AssetHandle<Cuda::Host::Vector<Cuda::LineSegment>>    hostLineSegments;

    Cuda::GI2DOverlayParams                                     overlayParams;
    Cuda::AssetHandle<Cuda::Host::BIH2DAsset>                   sceneBIH;
    Cuda::AssetHandle<Cuda::Host::BIH2DAsset>                   newObjctBIH;
};

GI2D::GI2D() :
    m_objectsPtr(std::make_unique<CudaObjects>()),
    m_objects(*m_objectsPtr)
{
    m_uiGraph.DeclareState("kIdleState", this, &GI2D::OnIdleState);

    // Create path
    m_uiGraph.DeclareState("kCreatePathOpen", this, &GI2D::OnCreatePath);      
    m_uiGraph.DeclareState("kCreatePathHover", this, &GI2D::OnCreatePath);
    m_uiGraph.DeclareState("kCreatePathAppend", this, &GI2D::OnCreatePath);
    m_uiGraph.DeclareState("kCreatePathClose", this, &GI2D::OnCreatePath);
    m_uiGraph.DeclareDeterministicTransition("kIdleState", "kCreatePathOpen", KeyboardButtonMap({ {'Q', kOnButtonDepressed}, {VK_CONTROL, kButtonDown} }), nullptr, 0);
    m_uiGraph.DeclareDeterministicTransition("kCreatePathHover", "kCreatePathAppend", nullptr, MouseButtonMap(kMouseLButton, kOnButtonDepressed), 0);
    m_uiGraph.DeclareDeterministicTransition("kCreatePathHover", "kCreatePathClose", KeyboardButtonMap(VK_ESCAPE, kOnButtonDepressed), nullptr, 0);

    // Select path
    m_uiGraph.DeclareState("kSelectPathDragging", this, &GI2D::OnSelectPath);
    m_uiGraph.DeclareState("kSelectPathEnd", this, &GI2D::OnSelectPath);
    m_uiGraph.DeclareDeterministicTransition("kIdleState", "kSelectPathDragging", nullptr, MouseButtonMap(kMouseLButton, kOnButtonDepressed), 0);
    m_uiGraph.DeclareDeterministicTransition("kSelectPathDragging", "kSelectPathDragging", nullptr, MouseButtonMap(kMouseLButton, kButtonDown), kUITriggerOnMouseMove);
    m_uiGraph.DeclareDeterministicTransition("kSelectPathDragging", "kSelectPathEnd", nullptr, MouseButtonMap(kMouseLButton, kOnButtonReleased), 0);

    m_uiGraph.Finalise();
}

GI2D::~GI2D()
{
    Destroy();
}

uint GI2D::OnIdleState(const UIStateTransition& transition)
{
    Log::Success("Back home!");
    return kUIStateOkay;
}

uint GI2D::OnSelectPath(const UIStateTransition& transition)
{   
    if (!transition.HasDeterministicTarget()) { return kUIStateError; }

    const std::string stateID = m_uiGraph.GetTargetStateID(transition);
    if (stateID == "kSelectPathDragging")
    {
        auto& selection = m_objects.overlayParams.selection;
        if (!selection.showBounds)
        {
            selection.bBox = BBox2f(m_objects.overlayParams.mousePosView, m_objects.overlayParams.mousePosView);
            selection.showBounds = true;            
        }
        else
        {
            selection.bBox[1] = m_objects.overlayParams.mousePosView;
        }
        Log::Success("Dragging!");
    }
    else if (stateID == "kSelectPathEnd")
    {
        m_objects.overlayParams.selection.showBounds = false;
        Log::Success("Finished!");

        m_uiGraph.SetState("kIdleState");
    }
    else
    {
        return kUIStateError;        
    }

    SetDirtyFlags(kGI2DDirtyParams);
    return kUIStateOkay;
}

uint GI2D::OnCreatePath(const UIStateTransition& transition)
{
    if (!transition.HasDeterministicTarget()) { return kUIStateError; }

    const std::string stateID = m_uiGraph.GetTargetStateID(transition);
    if (stateID == "kCreatePathOpen")
    {
        // Record the index of the starting segment on the path 
        m_newPath.pathStartIdx = m_objects.hostLineSegments->Size() - 1;
        m_newPath.numVertices = 0;
        
        m_uiGraph.SetState("kCreatePathHover");
    }
    else if (stateID == "kCreatePathHover")
    {
        if (m_newPath.numVertices >= 2)
        {
            m_objects.hostLineSegments->Back()[1] = m_objects.overlayParams.mousePosView;
            SetDirtyFlags(kGI2DDirtyLineSegments);
        }
    }
    else if (stateID == "kCreatePathAppend")
    {
        if (m_newPath.numVertices == 0)
        {
            // Create a zero-length segment that will be manipulated later
            m_objects.hostLineSegments->EmplaceBack(m_objects.overlayParams.mousePosView, m_objects.overlayParams.mousePosView, 0);
            m_newPath.numVertices = 2;
            SetDirtyFlags(kGI2DDirtyLineSegments);
        }
        else if (m_newPath.numVertices >= 2)
        {
            // Any more and we simply reuse the last vertex on the path as the start of the next segment
            m_objects.hostLineSegments->EmplaceBack(m_objects.hostLineSegments->Back()[1], m_objects.overlayParams.mousePosView, 0);
            m_newPath.numVertices++;
            SetDirtyFlags(kGI2DDirtyLineSegments);
        }

        m_uiGraph.SetState("kCreatePathHover");
    }
    else if (stateID == "kCreatePathClose")
    {
        m_uiGraph.SetState("kIdleState");
    }
    else
    {
        return kUIStateError;
    }
    
    return kUIStateOkay;
}

std::shared_ptr<RendererInterface> GI2D::Instantiate()
{
    return std::make_shared<GI2D>();
}

void GI2D::RebuildBIH()
{
    // Synchronise the segments
    Host::Vector<LineSegment>& segments = *m_objects.hostLineSegments;
    segments.Synchronise(kVectorSyncUpload);

    // Create a segment list ready for building
    // TODO: It's probably faster if we build on the already-sorted index list
    auto& primIdxs = m_objects.sceneBIH->GetPrimitiveIndices();
    primIdxs.resize(segments.Size());
    for (uint idx = 0; idx < primIdxs.size(); ++idx) { primIdxs[idx] = idx; }

    // Construct the BIH
    std::function<BBox2f(uint)> getPrimitiveBBox = [&segments](const uint& idx) -> BBox2f
    {
        return Grow(segments[idx].GetBoundingBox(), 0.001f);
    };
    m_objects.sceneBIH->Build(getPrimitiveBBox);

    SetDirtyFlags(kGI2DDirtyLineSegments);
}

void GI2D::OnInitialise()
{
    m_view.trans = vec2(0.f);
    m_view.scale = 1.0f;
    m_view.rotate = 0.0;
    m_objects.overlayParams.viewMatrix = ConstructViewMatrix(m_view.trans, m_view.rotate, m_view.scale) * m_clientToNormMatrix;
    m_view.zoomSpeed = 10.0f;   

    //m_primitiveContainer.Create(m_renderStream);

    m_objects.hostLineSegments = CreateAsset<Host::Vector<LineSegment>>("id_lineSegments", kVectorHostAlloc, m_renderStream);
    
    m_objects.sceneBIH = CreateAsset<Host::BIH2DAsset>("id_gi2DBIH");
    m_objects.overlayRenderer = CreateAsset<Host::GI2DOverlay>("id_gi2DOverlay", m_objects.sceneBIH, m_objects.hostLineSegments);

    SetDirtyFlags(kGI2DDirtyLineSegments | kGI2DDirtyParams);
}

void GI2D::OnDestroy()
{
    m_objects.overlayRenderer.DestroyAsset();
    m_objects.sceneBIH.DestroyAsset();
    m_objects.hostLineSegments.DestroyAsset();
}

void GI2D::OnRender()
{
    //std::this_thread::sleep_for(std::chrono::milliseconds(50));
    //Log::Write("Tick");

    if (m_dirtyFlags & kGI2DDirtyParams)
    {
        std::lock_guard <std::mutex> lock(m_resourceMutex);

        m_objects.overlayRenderer->SetParams(m_objects.overlayParams);

        ClearDirtyFlags(kGI2DDirtyParams);
    }
    if (m_dirtyFlags & kGI2DDirtyLineSegments)
    {
        RebuildBIH();

        ClearDirtyFlags(kGI2DDirtyLineSegments);
    }

    m_objects.overlayRenderer->Render(m_compositeImage);
}

void GI2D::OnKey(const uint code, const bool isSysKey, const bool isDown)
{

}

void GI2D::OnMouseButton(const uint code, const bool isDown)
{
    // Is the view being changed? 
    if (code == kMouseMButton)
    {
        m_view.dragAnchor = vec2(m_mouse.pos);
        m_view.rotAxis = normalize(m_view.dragAnchor - vec2(m_clientWidth, m_clientHeight) * 0.5f);
        m_view.transAnchor = m_view.trans;
        m_view.scaleAnchor = m_view.scale;
        m_view.rotAnchor = m_view.rotate;
    } 
}

mat3 GI2D::ConstructViewMatrix(const vec2& trans, const float rotate, const float scale) const
{
    const float sinTheta = std::sin(rotate);
    const float cosTheta = std::cos(rotate);
    mat3 m = mat3::Indentity();
    m.i00 = scale * cosTheta; m.i01 = scale * sinTheta;
    m.i10 = scale * sinTheta; m.i11 = scale * -cosTheta;
    m.i02 = trans.x;
    m.i12 = trans.y;
    return m;
}

void GI2D::OnMouseMove()
{
    // Dragging?
    if (IsMouseButtonDown(kMouseMButton))
    {
        OnViewChange();
    }
    
    {
        std::lock_guard <std::mutex> lock(m_resourceMutex);
        m_objects.overlayParams.mousePosView = m_objects.overlayParams.viewMatrix * vec2(m_mouse.pos);
    }

    // Mark the scene as dirty
    SetDirtyFlags(kGI2DDirtyParams);
}

void GI2D::OnViewChange()
{
    // Zooming?
    if (IsKeyDown(VK_CONTROL))
    {
        float logScaleAnchor = std::log2(::math::max(1e-10f, m_view.scaleAnchor));
        logScaleAnchor += m_view.zoomSpeed * float(m_mouse.pos.y - m_view.dragAnchor.y) / m_clientHeight;
        m_view.scale = std::pow(2.0, logScaleAnchor);

        //Log::Write("Scale: %f", m_view.scale);
    }
    // Rotating?
    else if (IsKeyDown(VK_SHIFT))
    {
        const vec2 delta = normalize(vec2(m_mouse.pos) - vec2(m_clientWidth, m_clientHeight) * 0.5f);
        const float theta = std::acos(dot(delta, m_view.rotAxis)) * (float(dot(delta, vec2(m_view.rotAxis.y, -m_view.rotAxis.x)) < 0.0f) * 2.0 - 1.0f);
        m_view.rotate = m_view.rotAnchor + theta;

        if (std::abs(std::fmod(m_view.rotate, kHalfPi)) < 0.05f) { m_view.rotate = std::round(m_view.rotate / kHalfPi) * kHalfPi; }

        //Log::Write("Theta: %f", m_view.rotate);
    }
    // Translating
    else
    {
        // Update the transformation
        m_objects.overlayParams.viewMatrix = ConstructViewMatrix(m_view.transAnchor, m_view.rotate, m_view.scale) * m_clientToNormMatrix;
        const vec2 dragDelta = (m_objects.overlayParams.viewMatrix * vec2(m_view.dragAnchor)) - (m_objects.overlayParams.viewMatrix * vec2(m_mouse.pos));
        m_view.trans = m_view.transAnchor + dragDelta;

        //Log::Write("Trans: %s", m_view.trans.format());
    }

    // Update the parameters in the overlay renderer
    {
        std::lock_guard <std::mutex> lock(m_resourceMutex);
        m_objects.overlayParams.viewMatrix = ConstructViewMatrix(m_view.trans, m_view.rotate, m_view.scale) * m_clientToNormMatrix;
        m_objects.overlayParams.viewScale = m_view.scale;
    }
}

void GI2D::OnMouseWheel()
{

}

void GI2D::OnResizeClient()
{
}