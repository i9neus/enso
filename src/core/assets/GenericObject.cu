﻿#include "GenericObject.cuh"
#include "io/json/JsonUtils.h"
#include "io/FilesystemUtils.h"

namespace Enso
{    
    __host__ __device__ GenericObjectParams::GenericObjectParams()
        {}

    __host__ void GenericObjectParams::ToJson(Json::Node& node) const
    {
        //flags.ToJson("objectFlags", node);
    }

    __host__ uint GenericObjectParams::FromJson(const Json::Node& node, const uint flags)
    {
        return 0u;
    }

    __host__ void GenericObjectParams::Randomise(const vec2& range)
    {
    }

    __host__ Host::GenericObject::GenericObject(const Asset::InitCtx& initCtx) :
        Dirtyable(initCtx),
        m_genericObjectFlags(0),
        m_isFinalised(false),
        m_isConstructed(false)
    {
    }
}