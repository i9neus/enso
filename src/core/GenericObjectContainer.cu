﻿#include "GenericObjectContainer.cuh"

namespace Enso
{
    __host__ void GenericObjectContainer::Finalise() const
    {
        Log::Debug("Finalising...\n");
        Log::Indent indent;

        Log::Debug("DAG map:\n");
        {
            Log::Indent indent;
            for (auto& object : m_dagMap)
            {
                Log::Debug("%s\n", object.first);
            }
        }
    }

    __host__ void GenericObjectContainer::Emplace(AssetHandle<Host::GenericObject>& newObject, const bool requireDAGPath)
    {
        AssertMsgFmt(!Exists(newObject->GetAssetID()), "A render object with ID '%s' already exists in the object container.\n", newObject->GetAssetID().c_str());

        // Store a strong reference to the object in the object map
        m_objectMap[newObject->GetAssetID()] = newObject;
        // Store a weak indexable reference in the object vector
        m_objectVector.emplace_back(newObject.GetWeakHandle());
        // Link the ID with the vector index
        m_idToIdxMap[newObject->GetAssetID()] = m_objectVector.size() - 1;
        ++m_uniqueIdx;

        // If the object has a DAG path, add it to the map alongside its weak reference
        if (newObject->HasDAGPath())
        {
            if (m_dagMap.find(newObject->GetDAGPath()) == m_dagMap.end())
            {
                m_dagMap[newObject->GetDAGPath()] = newObject.GetWeakHandle();
            }
            else
            {
                Log::Error("Internal error: object '%s' has the same DAG path (%s) as another object.\n", newObject->GetAssetID(), newObject->GetDAGPath());
            }
        }
        // Child objects don't need to have DAG paths because they aren't user-referenceable
        else if (requireDAGPath && !newObject->IsChildObject())
        {
            Log::Warning("Warning: instantiated object '%s' does not have a valid DAG path. (Did you forget to call UpdateDAGPath() during FromJson()?)\n", newObject->GetAssetID());
            return;
        }
    }

    __host__ void GenericObjectContainer::Erase(const Host::GenericObject& obj)
    {
        Erase(obj.GetAssetID());
    }

    __host__ void GenericObjectContainer::Erase(const std::string& id)
    {
        auto it = m_idToIdxMap.find(id);
        AssertMsgFmt(it != m_idToIdxMap.end(), "Invalid asset ID '%s'", id.c_str());
        Erase(it->second);
    }

    __host__ void GenericObjectContainer::Erase(const uint objectIdx)
    {
        AssertMsg(objectIdx < m_objectVector.size(), "Render object index out of bounds.");
        AssertMsg(!m_objectVector[objectIdx].expired(), "Internal error: render object expired unexpectedly");

        AssetHandle<Host::GenericObject> obj(m_objectVector[objectIdx]);
        Assert(obj);

        // Erase the object from the DAG map
        if (obj->HasDAGPath())
        {
            const auto& dag = obj->GetDAGPath();
            m_objectMap.erase(dag);
        }

        // Erase the object from the main asset map
        const auto& id = obj->GetAssetID();
        AssertMsgFmt(m_objectMap.erase(id), "Internal error: object map and object list have gone out of sync with object '%s'", id.c_str());

        // Erase the object from the indexed list
        m_objectVector[objectIdx] = m_objectVector.back();
        m_objectVector.pop_back();

        // Destroy the asset
        obj.DestroyAsset();
    } 

    __host__ AssetHandle<Host::GenericObject> GenericObjectContainer::operator[](const uint objectIdx)
    {
        AssertMsg(objectIdx < m_objectVector.size(), "Render object index out of bounds.");
        AssertMsg(!m_objectVector[objectIdx].expired(), "Internal error: render object expired unexpectedly");

        return AssetHandle<Host::GenericObject>(m_objectVector[objectIdx]);
    }

    __host__ void GenericObjectContainer::Bind()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Bind(*this);
        }
    }

    __host__ void GenericObjectContainer::Synchronise()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Synchronise();
        }
    }

    __host__ void GenericObjectContainer::OnDestroyAsset()
    {
        Log::Debug("Unloading scene graph...");
        
        constexpr int kMaxAttempts = 10;
        std::vector<std::string> activeList;
        for (int i = 0; !m_objectMap.empty() && i < kMaxAttempts; i++)
        {
            //Log::Indent indent(tfm::format("Pass %i...", i + 1));
            for (RenderObjectMap::iterator it = m_objectMap.begin(); it != m_objectMap.end();)
            {
                uint flags = kAssetCleanupPass;
                if (i == kMaxAttempts - 1)
                {
                    flags |= kAssetForceDestroy | kAssetAssertOnError;

                    if (it->second.GetReferenceCount() > 1)
                    {
                        activeList.push_back(it->first);
                    }
                }

                // Try to delete the asset
                if (!it->second.DestroyAsset(flags))
                {
                    ++it;
                }
                else
                {
                    auto nextIt = std::next(it);
                    m_objectMap.erase(it);
                    it = nextIt;
                }
            }
        }

        if (activeList.size() > 0)
        {
            Log::Error("ERROR: %i objects were not properly cleaned up:", activeList.size());
            for (const auto& name : activeList)
            {
                Log::Error("  - %s", name);
            }
        }
    }
}