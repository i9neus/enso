﻿#include "GenericObjectContainer.cuh"

namespace Enso
{
    __host__ void Host::GenericObjectContainer::Finalise() const
    {
        Log::Debug("Finalising...\n");
        Log::Indent indent;

        Log::Debug("DAG map:\n");
        {
            Log::Indent indent;
            for (auto& object : m_dagMap)
            {
                Log::Debug("%s\n", object.first);
            }
        }
    }

    __host__ void Host::GenericObjectContainer::Emplace(AssetHandle<Host::GenericObject>& newObject, const bool requireDAGPath)
    {
        AssertMsgFmt(!Exists(newObject->GetAssetID()), "A render object with ID '%s' already exists in the object container.\n", newObject->GetAssetID().c_str());

        // Store a strong reference to the object in the object map
        m_objectMap[newObject->GetAssetID()] = newObject;        
        ++m_uniqueIdx;

        // If the object has a DAG path, add it to the map alongside its weak reference
        const std::string dagPath = newObject->GetAssetDAGPath();
        if (m_dagMap.find(dagPath) == m_dagMap.end())
        {
            m_dagMap[dagPath] = newObject.GetWeakHandle();
        }
        else
        {
            Log::Error("Internal error: object '%s' has the same DAG path (%s) as another object.\n", newObject->GetAssetID(), dagPath);
        }
    }

    __host__ void Host::GenericObjectContainer::Erase(const Host::GenericObject& obj)
    {
        Erase(obj.GetAssetID());
    }   

    __host__ void Host::GenericObjectContainer::Erase(const std::string& id)
    {
        // Get the handle to the object
        auto it = m_objectMap.find(id);
        AssertMsgFmt(it != m_objectMap.end(), "Render object '%s' is not in the container.", id.c_str());
        auto obj = it->second;

        // Erase the object from the DAG map
        m_dagMap.erase(obj->GetAssetDAGPath());

        // Erase the object from the main asset map
        AssertMsgFmt(m_objectMap.erase(id), "Internal error: object map and object list have gone out of sync with object '%s'", id.c_str());

        // Destroy the asset
        obj.DestroyAsset();
    } 

    __host__ void Host::GenericObjectContainer::Clear()
    {
        m_objectMap.clear();
    }

    __host__ void Host::GenericObjectContainer::Bind()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Bind();
        }
    }

    __host__ void Host::GenericObjectContainer::Synchronise(const uint flags)
    {
        for (auto& object : m_objectMap)
        {
            object.second->Synchronise(flags);
        }
    }

    __host__ Host::GenericObjectContainer::~GenericObjectContainer() noexcept
    {
        Log::Debug("Unloading scene graph...");
        
        constexpr int kMaxAttempts = 10;
        std::vector<std::string> activeList;
        for (int i = 0; !m_objectMap.empty() && i < kMaxAttempts; i++)
        {
            //Log::Indent indent(tfm::format("Pass %i...", i + 1));
            for (RenderObjectMap::iterator it = m_objectMap.begin(); it != m_objectMap.end();)
            {
                uint flags = kAssetCleanupPass;
                if (i == kMaxAttempts - 1)
                {
                    flags |= kAssetForceDestroy | kAssetAssertOnError;

                    if (it->second.GetReferenceCount() > 1)
                    {
                        activeList.push_back(it->first);
                    }
                }

                // Try to delete the asset
                if (!it->second.DestroyAsset(flags))
                {
                    ++it;
                }
                else
                {
                    auto nextIt = std::next(it);
                    m_objectMap.erase(it);
                    it = nextIt;
                }
            }
        }

        if (activeList.size() > 0)
        {
            Log::Error("ERROR: %i objects were not properly cleaned up:", activeList.size());
            for (const auto& name : activeList)
            {
                Log::Error("  - %s", name);
            }
        }
    }
}