#include "hip/hip_runtime.h"
#define CUDA_DEVICE_GLOBAL_ASSERTS

#include "PathTracer.cuh"
#include "core/math/ColourUtils.cuh"
#include "core/math/Hash.cuh"
#include "core/3d/Ctx.cuh"
#include "core/AssetContainer.cuh"
#include "core/3d/Transform.cuh"
#include "core/Vector.cuh"
#include "Geometry.cuh"
#include "core/3d/Cameras.cuh"
#include "Scene.cuh"
#include "Integrator.cuh"
//#include "../scene/SceneContainer.cuh"

#include "io/json/JsonUtils.h"
//#include "core/AccumulationBuffer.cuh"

namespace Enso
{        
    __host__ __device__ PathTracerParams::PathTracerParams()
    {
        viewport.dims = ivec2(0);  
        frameIdx = 0;
    }

    __host__ __device__ void PathTracerParams::Validate() const
    {
        CudaAssert(viewport.dims.x != 0 && viewport.dims.y != 0);
    }

    __host__ __device__ void PathTracerObjects::Validate() const
    {
        CudaAssert(transforms);
        CudaAssert(meanAccumBuffer);
        CudaAssert(varAccumBuffer); 
        CudaAssert(denoisedBuffer);
    }

    __device__ void Device::PathTracer::Synchronise(const PathTracerParams& params) 
    {
        m_params = params;   
        m_nlm.Initialise(10, 2, 2.f, 2.f);
     }
    
    __device__ void Device::PathTracer::Synchronise(const PathTracerObjects& objects) 
    { 
        objects.Validate(); 
        m_objects = objects; 
        m_nlm.Initialise(m_objects.meanAccumBuffer, m_objects.varAccumBuffer);
    }

    __device__ void Device::PathTracer::Render()
    {
        CudaAssertDebug(m_objects.transforms->Size() == 9);

        const ivec2 xyViewport = kKernelPos<ivec2>();
        if (xyViewport.x < 0 || xyViewport.x >= m_params.viewport.dims.x || xyViewport.y < 0 || xyViewport.y >= m_params.viewport.dims.y) { return; }        

        // Get pointers to the object transforms
        const auto& transforms = *m_objects.transforms;
        const auto& emitterTrans = transforms.Back();

        // Create a render context
        RenderCtx renderCtx;
        renderCtx.rng.Initialise(HashOf(m_params.frameIdx, xyViewport.x, xyViewport.y));
        renderCtx.qrng.Initialise(0x7a67bbfc, HashOf(xyViewport.x, xyViewport.y) + m_params.frameIdx);
        renderCtx.viewport.dims = m_params.viewport.dims;
        renderCtx.viewport.xy = xyViewport;
        renderCtx.frameIdx = m_params.frameIdx;

        // Transform into normalised sceen space
        const vec2 uvView = ScreenToNormalisedScreen(vec2(xyViewport) + renderCtx.Rand(0).xy, vec2(m_params.viewport.dims));
        Ray directRay, indirectRay;

        // Create the camera ray
        float cameraPhi = -kPi;
        //cameraPhi += kTwoPi * m_params.wallTime * 0.01f;
        vec3 cameraPos = vec3(cos(cameraPhi), 0.5, sin(cameraPhi)) * 2.;
        indirectRay = Cameras::CreatePinholeRay(uvView, cameraPos, vec3(0., -0., -0.), 50.);

        int genFlags = kGeneratedIndirect;
        HitCtx hit;
        vec3 L = kZero;
        //int renderMode = (xyViewport.x < m_params.viewport.dims.x * 0.5f) ? kModePathTraced : kModeNEE;
        const int renderMode = kModeNEE;

        constexpr int kMaxPathDepth = 5;
        constexpr int kMaxIterations = 10;
        for (int rayIdx = 0; rayIdx < kMaxIterations && genFlags != kGenerateNothing; ++rayIdx)
        {
            Ray ray;
            if ((genFlags & kGeneratedDirect) != 0) ray = directRay; else ray = indirectRay;

            if (ray.depth >= kMaxPathDepth) { continue; }

            if (Trace(ray, hit, transforms) == kMatInvalid && !ray.IsDirectSample())
            {
                //if(depth > 0)
                {
                    L += kOne * ray.weight * 0.2;
                    //L += kOne * 0.5 * luminance(texture(iChannel1, ray.od.d).xyz);
                }
                break;
            }

            //EvaluateMaterial(ray, hit);

            //L = hit.n * 0.5 + 0.5;
            //break;

            if ((genFlags & kGeneratedDirect) != 0)
            {
                ShadeDirectSample(ray, hit, L);
                genFlags &= ~kGeneratedDirect;
            }
            else if ((genFlags & kGeneratedIndirect) != 0)
            {
                genFlags = Shade(ray, indirectRay, directRay, hit, renderCtx, emitterTrans, renderMode, L);
            }
        }

        auto& meanL = m_objects.meanAccumBuffer->At(xyViewport);
        auto& varL = m_objects.varAccumBuffer->At(xyViewport);

        // Unstable running variance
        //varL += vec4(sqr(L), 1.);
        
        // Welford's online algorithm
        varL += vec4((L - meanL.xyz / fmaxf(1.f, meanL.w)) * 
                     (L - (L + meanL.xyz) / (1.f + meanL.w)), 1.0f);

        meanL += vec4(L, 1.);

    }
    DEFINE_KERNEL_PASSTHROUGH(Render);

    __device__ void Device::PathTracer::Denoise()
    {        
        const ivec2 xyViewport = kKernelPos<ivec2>();
        if (xyViewport.x < m_params.viewport.dims.x && xyViewport.y < m_params.viewport.dims.y)
        {
            m_objects.denoisedBuffer->At(xyViewport) = m_nlm.FilterPixel(xyViewport);
        }
    }
    DEFINE_KERNEL_PASSTHROUGH(Denoise);


    __device__ void Device::PathTracer::Composite(Device::ImageRGBA* deviceOutputImage)
    {
        CudaAssertDebug(deviceOutputImage);

        // TODO: Make alpha compositing a generic operation inside the Image class.
        const ivec2 xyAccum = kKernelPos<ivec2>();
        const ivec2 xyViewport = xyAccum + deviceOutputImage->Dimensions() / 2 - m_objects.meanAccumBuffer->Dimensions() / 2;
        BBox2i border(0, 0, m_params.viewport.dims.x, m_params.viewport.dims.y);
        /*if(border.PointOnPerimiter(xyAccum, 2))
        {
            deviceOutputImage->At(xyViewport) = vec4(1.0f);
        }*/
        if (xyAccum.x < m_params.viewport.dims.x && xyAccum.y < m_params.viewport.dims.y)
        {
            if (xyAccum.x < m_params.viewport.dims.x / 2)
            {
                //const vec4& varL = m_objects.varAccumBuffer->At(xyAccum);
                const vec4& meanL = m_objects.meanAccumBuffer->At(xyAccum);
                deviceOutputImage->At(xyViewport) = vec4(meanL.xyz / fmaxf(1.f, meanL.w), 1.0f);

                //deviceOutputImage->At(xyViewport) = vec4(varL.xyz / fmaxf(1.f, varL.w) - sqr(meanL.xyz / fmaxf(1.f, meanL.w)), 1.f);
                //deviceOutputImage->At(xyViewport) = vec4(varL.xyz / sqr(fmaxf(1.f, varL.w)), 1.f);
            }
            else
            {
                const vec3& denoisedL = m_objects.denoisedBuffer->At(xyAccum);
                deviceOutputImage->At(xyViewport) = vec4(denoisedL, 1.f);
            }
        }
    }
    DEFINE_KERNEL_PASSTHROUGH_ARGS(Composite);

    Host::PathTracer::PathTracer(const Asset::InitCtx& initCtx, /*const AssetHandle<const Host::SceneContainer>& scene, */const uint width, const uint height, hipStream_t renderStream):
        GenericObject(initCtx)
        //m_scene(scene)
    {                
        // Create some Cuda objects
        m_hostMeanAccumBuffer = AssetAllocator::CreateChildAsset<Host::ImageRGBW>(*this, "meanAccumBufferMean", width, height, renderStream);
        m_hostVarAccumBuffer = AssetAllocator::CreateChildAsset<Host::ImageRGBW>(*this, "meanAccumBufferVar", width, height, renderStream);
        m_hostDenoisedBuffer = AssetAllocator::CreateChildAsset<Host::ImageRGB>(*this, "denoisedBuffer", width, height, renderStream);
        m_hostTransforms = AssetAllocator::CreateChildAsset<Host::Vector<BidirectionalTransform>>(*this, "transforms", kVectorHostAlloc);

        m_deviceObjects.meanAccumBuffer = m_hostMeanAccumBuffer->GetDeviceInstance();
        m_deviceObjects.varAccumBuffer = m_hostVarAccumBuffer->GetDeviceInstance();
        m_deviceObjects.denoisedBuffer = m_hostDenoisedBuffer->GetDeviceInstance();
        m_deviceObjects.transforms = m_hostTransforms->GetDeviceInstance();
        //m_deviceObjects.scene = m_scene->GetDeviceInstance();

        cu_deviceInstance = AssetAllocator::InstantiateOnDevice<Device::PathTracer>(*this);

        m_params.viewport.dims = ivec2(width, height);
        m_params.frameIdx = 0;
        m_params.wallTime = 0.f;
        m_wallTime.Reset();

        CreateScene();
    }

    Host::PathTracer::~PathTracer() noexcept
    {
        m_hostMeanAccumBuffer.DestroyAsset();
        m_hostVarAccumBuffer.DestroyAsset();
        m_hostDenoisedBuffer.DestroyAsset();
        m_hostTransforms.DestroyAsset();

        AssetAllocator::DestroyOnDevice(*this, cu_deviceInstance);
    }

    __host__ void Host::PathTracer::CreateScene()
    {
        m_hostTransforms->Clear();

#define kNumSpheres 7        
        for (int sphereIdx = 0; sphereIdx < kNumSpheres; ++sphereIdx)
        {
            float phi = kTwoPi * (0.75f + float(sphereIdx) / float(kNumSpheres));
            m_hostTransforms->PushBack(BidirectionalTransform(vec3(cos(phi), 0.f, sin(phi)) * 0.7f, kZero, 0.2f));
        }

        m_hostTransforms->PushBack(BidirectionalTransform(vec3(0.f, -0.2f, 0.f), vec3(-kHalfPi, 0.f, 0.f), 2.f));   // Ground plane
        m_hostTransforms->PushBack(BidirectionalTransform(kEmitterPos, kEmitterRot, kEmitterSca));                  // Emitter plane

        m_hostTransforms->Synchronise(kVectorSyncUpload);

        Synchronise(kSyncObjects | kSyncParams);
    }

    __host__ void Host::PathTracer::Synchronise(const uint syncFlags)
    {
        if (syncFlags & kSyncObjects) { SynchroniseObjects<Device::PathTracer>(cu_deviceInstance, m_deviceObjects); }
        if (syncFlags & kSyncParams) { SynchroniseObjects<Device::PathTracer>(cu_deviceInstance, m_params); }
    }

    __host__ void Host::PathTracer::Render()
    {
        //KernelPrepare << <1, 1 >> > (cu_deviceInstance, m_dirtyFlags);

        //if (m_params.frameIdx > 10) return;

        dim3 blockSize, gridSize;
        KernelParamsFromImage(m_hostMeanAccumBuffer, blockSize, gridSize);

        // Accumulate the frame
        KernelRender << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceInstance);

        // Denoise if necessary
        if (m_params.frameIdx % 500 == 0)
        {
            KernelDenoise << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceInstance);
        }

        IsOk(hipDeviceSynchronize());

        if (m_renderTimer.Get() > 1.)
        {
            m_renderTimer.Reset();
            Log::Debug("Frame: %i", m_params.frameIdx);
        }
    }

    __host__ void Host::PathTracer::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage) const
    {
        dim3 blockSize, gridSize;
        KernelParamsFromImage(m_hostMeanAccumBuffer, blockSize, gridSize);

        KernelComposite << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceInstance, hostOutputImage->GetDeviceInstance());
        IsOk(hipDeviceSynchronize());
    }

    __host__ bool Host::PathTracer::Prepare()
    {
        m_params.frameIdx++;
        m_params.wallTime = m_wallTime.Get();

        // Upload to the device
        Synchronise(kSyncParams);
        return true;
    }

    __host__ void Host::PathTracer::Clear()
    {
        m_hostMeanAccumBuffer->Clear(vec4(0.f));

        m_params.frameIdx = 0;  
        Synchronise(kSyncParams);
    }
}