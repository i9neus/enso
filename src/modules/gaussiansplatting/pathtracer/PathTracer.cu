#include "hip/hip_runtime.h"
#define CUDA_DEVICE_GLOBAL_ASSERTS

#include "PathTracer.cuh"
#include "core/math/ColourUtils.cuh"
#include "core/math/Hash.cuh"
#include "core/assets/AssetContainer.cuh"
#include "core/3d/Transform.cuh"
#include "core/containers/Vector.cuh"
#include "core/3d/Cameras.cuh"
//#include "Scene.cuh"
//#include "Integrator.cuh"
#include "core/assets/GenericObjectContainer.cuh"
//#include "Geometry.cuh"
#include "core/math/samplers/MersenneTwister.cuh"
#include "core/math/samplers/Dither.cuh"

#include "../scene/SceneContainer.cuh"
#include "../scene/cameras/Camera.cuh"
#include "../scene/materials/Material.cuh"
#include "../scene/lights/LightSampler.cuh"
#include "../scene/lights/QuadLight.cuh"
#include "../scene/textures/Texture2D.cuh"
#include "../scene/tracables/Tracable.cuh"

#include "io/json/JsonUtils.h"
//#include "core/AccumulationBuffer.cuh"

namespace Enso
{        
    #define kMatInvalid -1

    #define kModePathTraced 0
    #define kModeNEE 1

    #define kGenerateNothing 0
    #define kGeneratedDirect 1
    #define kGeneratedIndirect 2
    
    __host__ __device__ PathTracerParams::PathTracerParams()
    {
        viewport.dims = ivec2(0);  
        hasValidScene = false;
    }

    __host__ __device__ void PathTracerParams::Validate() const
    {
        CudaAssert(viewport.dims.x != 0 && viewport.dims.y != 0);
    }   

    __host__ __device__ void Device::PathTracer::Synchronise(const PathTracerParams& params) 
    {
        m_params = params;   
        m_nlm.Initialise(10, 2, 2.f, 2.f);
     }
    
    __device__ void Device::PathTracer::Synchronise(const PathTracerObjects& objects) 
    { 
        objects.Validate(); 
        m_objects = objects; 
        m_nlm.Initialise(m_objects.meanAccumBuffer, m_objects.varAccumBuffer);
    }

    __device__ __forceinline__ float PowerHeuristic(float pdf1, float pdf2)
    {
        return saturatef(sqr(pdf1) / fmaxf(1e-10, sqr(pdf1) + sqr(pdf2)));
    }

    __device__ float Device::PathTracer::SampleEmitter(const Ray& incident, RayStack& extantStack, const HitCtx& hit, const Material& material, const LightSampler* lightSampler, const vec2& xi) const
    {
        auto& extant = extantStack.Push();
        float emitterPdf = lightSampler->Sample(incident, extant, hit, xi);
        if (emitterPdf <= 0.)
        {
            extantStack.Pop();
            return 0.;
        }
 
        //float bxdfPdf = material.Evaluate(-incident.od.d, extant.od.d, hit.n);
        vec3 materialWeight = kOne;
        float bxdfPdf = material.Evaluate(incident, extant, hit, materialWeight);

        // Lambert cosine factor
        bxdfPdf *= dot(extant.od.d, hit.n);

        // Apply power heuristic up-weighted by a factor to two to account for stochastic branching
        extant.weight *= 2. * bxdfPdf * materialWeight * PowerHeuristic(emitterPdf, bxdfPdf);

        return emitterPdf;
    }

    __device__ float Device::PathTracer::SampleBxDF(const Ray& incident, RayStack& extantStack, const HitCtx& hit, const Material& material, const LightSampler* lightSampler, const vec2& xi, const bool isDirectSample) const
    {
        vec3 o;
        vec3 kickoff = hit.n * 1e-4;
        
        // Sample the BxDF
        vec3 materialWeight = kOne;
        //float bxdfPdf = material.Sample(xi, -incident.od.d, hit.n, o, materialWeight);      
        float bxdfPdf = material.Sample(xi, incident, hit, o, materialWeight);

        // Create the ray
        auto& extant = extantStack.Push();
        extant.Construct(incident.Point(), o, kickoff, incident.weight * materialWeight, incident.depth + 1, incident.InheritedFlags());

        // If this isn't a perfect specular BxDF, flag the ray as scattered
        if (!material.IsPerfectSpecular()) { extant.flags |= kRayScattered; }

        // If this is a light samaple, compute the PDF of the emitter and apply the power heuristic
        if (isDirectSample)
        {
            CudaAssertDebug(lightSampler);
            const float emitterPdf = lightSampler->Evaluate(extant, hit);
            extant.weight *= 2. * PowerHeuristic(bxdfPdf, emitterPdf);
            extant.flags |= kRayDirectSampleBxDF;
        }

        return bxdfPdf;
    }

    __device__ void Device::PathTracer::Shade(const Ray& incidentRay, RayStack& extantStack, HitCtx& hit, RenderCtx& renderCtx, const Material& material, int renderMode, vec3& L) const
    {
        // Generate some random numbers
        //vec4 xi = Rand(renderCtx.rng);
        vec4 xi = renderCtx.Rand(1 + incidentRay.depth);
        float xiSplit = fract(OrderedDither(renderCtx.viewport.xy) + float(renderCtx.frameIdx) / 16.);

        // Perfect specular BxDFs don't need light sampling
        if (material.IsPerfectSpecular()) { renderMode = kModePathTraced; }

        int genFlags = 0;

        // Sample the BxDF for the indirect contribution
        SampleBxDF(incidentRay, extantStack, hit, material, nullptr, xi.zw, false);

        // If we're in next-event estimation mode, stochastically sample either the emitter or the BxDF
        // IMPORTANT: Direct rays must be the last on the stack to prevent it overflowing
        if (renderMode == kModeNEE)
        {
            const LightSampler* light = (*m_objects.scene.lightSamplers)[0];
            
            if (xiSplit < 0.5)
            {
                SampleBxDF(incidentRay, extantStack, hit, material, light, xi.xy, true);
            }
            else
            {
                SampleEmitter(incidentRay, extantStack, hit, material, light, xi.xy);
            }
        }
    }

    // Trace scene geometry and return a pointer to the closest hit object
    __device__ const Device::Tracable* Device::PathTracer::Trace(Ray& ray, HitCtx& hitCtx) const
    {
        const Tracable* hitTracable = nullptr;
        for (int idx = 0; idx < m_objects.scene.tracables->size(); ++idx)
        {
            const Tracable* testTracable = (*m_objects.scene.tracables)[idx];
            if (testTracable->IntersectRay(ray, hitCtx))
            {
                hitTracable = testTracable;
            }
        }

        return hitTracable;
    }

    __device__ void Device::PathTracer::Render()
    {        
        const ivec2 xyViewport = kKernelPos<ivec2>();
        if (xyViewport.x < 0 || xyViewport.x >= m_params.viewport.dims.x || xyViewport.y < 0 || xyViewport.y >= m_params.viewport.dims.y) { return; }        

        // Get pointers to the object transforms
        const auto& tracables = *m_objects.scene.tracables;
        const auto& textures = *m_objects.scene.textures;

        // Create a render context
        RenderCtx renderCtx;
        renderCtx.rng.Initialise(HashOf(RenderableObject::m_params.frameIdx, xyViewport.x, xyViewport.y));
        renderCtx.qrng.Initialise(0, HashOf(xyViewport.x, xyViewport.y) + RenderableObject::m_params.frameIdx);
        renderCtx.viewport.dims = m_params.viewport.dims;
        renderCtx.viewport.xy = xyViewport;
        renderCtx.frameIdx = RenderableObject::m_params.frameIdx;

        // Transform into normalised sceen space
        const vec4 xi = renderCtx.Rand(0);
        const vec2 uvView = PixelToNormalisedScreen(vec2(xyViewport) + xi.xy, vec2(m_params.viewport.dims));
        
        RayStack extantStack;
        m_objects.activeCamera->CreateRay(uvView, xi.zw, extantStack.Push());

        int genFlags = kGeneratedIndirect;
        HitCtx hitCtx;
        vec3 L = kZero;
        //int renderMode = (xyViewport.x < m_params.viewport.dims.x * 0.5f) ? kModePathTraced : kModeNEE;
        const int renderMode = kModePathTraced;

        constexpr int kMaxPathDepth = 5;
        constexpr int kMaxIterations = 10;
        for (int rayIdx = 0; rayIdx < kMaxIterations && !extantStack.IsEmpty(); ++rayIdx)
        {
            // Pop the stack
            Ray ray = extantStack.Pop();         

            // Trace the ray
            const auto* hitTracable = Trace(ray, hitCtx);
            if (!hitTracable)
            {
                // Only accumulate environment contribution on indirect samples
                if(!ray.IsDirectSample() && m_objects.scene.envTexture)
                {
                    //L += kOne * ray.weight * 0.2;
                    const float env = luminance(m_objects.scene.envTexture->Evaluate(DirToEquirect(ray.od.d)).xyz);
                    L += ray.weight * powf(clamp(env, 0.f, 10.0f), 1 / 1.8f);
                }
                continue;
            }

            // Evaluate a direct ray
            if (ray.IsDirectSample())
            {
                // If we're hit a light, 
                if (hitTracable->IsLight() && !ray.IsBackfacing())
                {
                    // If this sample is a light ray, all we need to know is whether or not it hit the light. 
                    // If it did, just accumulate the weight which contains the radiant energy from the light sample. 
                    L += ray.weight;
                }
            }
            else
            {
                // Emitters don't reflect light so we don't need to shade them. Simply accumulate the emitted radiance and we're done.
                if (hitTracable->IsLight())
                {
                    if (!ray.IsBackfacing())
                    {
                        L += hitTracable->GetRadiance() * ray.weight;
                    }
                }
                // If we're at depth, skip this evaluation
                else if (ray.depth < kMaxPathDepth) 
                {
                    const Material& material = *(*m_objects.scene.materials)[hitTracable->GetMaterialIdx()];
                    Shade(ray, extantStack, hitCtx, renderCtx, material, renderMode, L);
                }
            }
        }

        auto& meanL = m_objects.meanAccumBuffer->At(xyViewport);
        auto& varL = m_objects.varAccumBuffer->At(xyViewport);

        // Unstable running variance
        //varL += vec4(sqr(L), 1.);
        
        // Welford's online algorithm
        varL += vec4((L - meanL.xyz / fmaxf(1.f, meanL.w)) * 
                     (L - (L + meanL.xyz) / (1.f + meanL.w)), 1.0f);

        meanL += vec4(L, 1.);

    }
    DEFINE_KERNEL_PASSTHROUGH(Render);

    __device__ void Device::PathTracer::Denoise()
    {        
        const ivec2 xyViewport = kKernelPos<ivec2>();
        if (xyViewport.x < m_params.viewport.dims.x && xyViewport.y < m_params.viewport.dims.y)
        {
            m_objects.denoisedBuffer->At(xyViewport) = m_nlm.FilterPixel(xyViewport);
        }
    }
    DEFINE_KERNEL_PASSTHROUGH(Denoise);

    __device__ void Device::PathTracer::Composite(Device::ImageRGBA* deviceOutputImage, const bool isValidScene)
    {
        CudaAssertDebug(deviceOutputImage);

        // TODO: Make alpha compositing a generic operation inside the Image class.
        const ivec2 xyAccum = kKernelPos<ivec2>();
        const ivec2 xyViewport = xyAccum + deviceOutputImage->Dimensions() / 2 - m_objects.meanAccumBuffer->Dimensions() / 2;
        
        /*BBox2i border(0, 0, m_params.viewport.dims.x, m_params.viewport.dims.y);
        if(border.PointOnPerimiter(xyAccum, 2))
        {
            deviceOutputImage->At(xyViewport) = vec4(1.0f);
        }*/
        if (xyAccum.x < m_params.viewport.dims.x && xyAccum.y < m_params.viewport.dims.y)
        {
            //if (xyAccum.x < m_params.viewport.dims.x / 2)
            {
                //const vec4& varL = m_objects.varAccumBuffer->At(xyAccum);
                const vec4& texel = m_objects.meanAccumBuffer->At(xyAccum);
                vec3 L = texel.xyz / fmaxf(1.f, texel.w);
                L = pow(L, 0.7f);

                deviceOutputImage->At(xyViewport) = vec4(L, 1.0f);

                //deviceOutputImage->At(xyViewport) = vec4(varL.xyz / fmaxf(1.f, varL.w) - sqr(meanL.xyz / fmaxf(1.f, meanL.w)), 1.f);
                //deviceOutputImage->At(xyViewport) = vec4(varL.xyz / sqr(fmaxf(1.f, varL.w)), 1.f);
            }
            /*else
            {
                const vec3& denoisedL = m_objects.denoisedBuffer->At(xyAccum);
                deviceOutputImage->At(xyViewport) = vec4(denoisedL, 1.f);
            }*/
        }
    }
    DEFINE_KERNEL_PASSTHROUGH_ARGS(Composite);

    __host__ __device__ bool Device::PathTracer::IsClickablePoint(const UIViewCtx& viewCtx) const
    {
        return GetWorldBBox().Contains(viewCtx.mousePos);
    }

    __host__ __device__ vec4 Device::PathTracer::EvaluateOverlay(const vec2& pWorld, const UIViewCtx& viewCtx, const bool isMouseTest) const
    {
        if (!GetWorldBBox().Contains(pWorld)) { return vec4(0.0f); }

#ifdef __CUDA_ARCH__
        const vec2 pObject = ToObjectSpace(pWorld);
        const ivec2 pPixel = ivec2(vec2(m_params.viewport.dims) * (pObject - m_params.viewport.objectBounds.lower) / m_params.viewport.objectBounds.Dimensions());

        if (!m_params.hasValidScene)
        {
            const float hatch = step(0.8f, fract(0.05f * dot(pWorld / viewCtx.dPdXY, vec2(1.f))));
            return vec4(kOne * hatch * 0.1f, 1.f);
        }
        else if (pPixel.x >= 0 && pPixel.x < m_params.viewport.dims.x && pPixel.y >= 0 && pPixel.y < m_params.viewport.dims.y)
        {
            //if (pPixel.x < m_params.viewport.dims.x / 2)
            {
                const vec4& texel = m_objects.meanAccumBuffer->At(pPixel);
                vec3 L = texel.xyz / fmaxf(1.f, texel.w);
                L = pow(L, 0.7f);
                
                return vec4(L, 1.0f);

                //deviceOutputImage->At(xyViewport) = vec4(varL.xyz / fmaxf(1.f, varL.w) - sqr(meanL.xyz / fmaxf(1.f, meanL.w)), 1.f);
                //deviceOutputImage->At(xyViewport) = vec4(varL.xyz / sqr(fmaxf(1.f, varL.w)), 1.f);
            }
            /*else
            {
                const vec3& denoisedL = m_objects.denoisedBuffer->At(pPixel);
                return vec4(denoisedL, 1.f);
            }*/
        }
#else
        return vec4(1.);
#endif
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    __host__ AssetHandle<Host::GenericObject> Host::PathTracer::Instantiate(const std::string& id, const Host::Asset& parentAsset, const AssetHandle<const Host::GenericObjectContainer>& genericObjects)
    {
        return AssetAllocator::CreateChildAsset<Host::PathTracer>(parentAsset, id, genericObjects);
    }

    __host__ Host::PathTracer::PathTracer(const Asset::InitCtx& initCtx, const AssetHandle<const Host::GenericObjectContainer>& genericObjects) :
        DrawableObject(initCtx, &m_hostInstance),
        cu_deviceInstance(AssetAllocator::InstantiateOnDevice<Device::PathTracer>(*this))
    {        
        DrawableObject::SetDeviceInstance(AssetAllocator::StaticCastOnDevice<Device::DrawableObject>(cu_deviceInstance));
        RenderableObject::SetDeviceInstance(AssetAllocator::StaticCastOnDevice<Device::RenderableObject>(cu_deviceInstance));

        constexpr int kViewportWidth = 1200;
        constexpr int kViewportHeight = 675;

        // Create some Cuda objects
        m_hostMeanAccumBuffer = AssetAllocator::CreateChildAsset<Host::ImageRGBW>(*this, "meanAccumBufferMean", kViewportWidth, kViewportHeight, nullptr);
        m_hostVarAccumBuffer = AssetAllocator::CreateChildAsset<Host::ImageRGBW>(*this, "meanAccumBufferVar", kViewportWidth, kViewportHeight, nullptr);
        m_hostDenoisedBuffer = AssetAllocator::CreateChildAsset<Host::ImageRGB>(*this, "denoisedBuffer", kViewportWidth, kViewportHeight, nullptr);
        m_hostTransforms = AssetAllocator::CreateChildAsset<Host::Vector<BidirectionalTransform>>(*this, "transforms");

        m_deviceObjects.meanAccumBuffer = m_hostMeanAccumBuffer->GetDeviceInstance();
        m_deviceObjects.varAccumBuffer = m_hostVarAccumBuffer->GetDeviceInstance();
        m_deviceObjects.denoisedBuffer = m_hostDenoisedBuffer->GetDeviceInstance();
        m_deviceObjects.transforms = m_hostTransforms->GetDeviceInstance();

        const vec2 boundHalf = 0.25 * ((kViewportHeight > kViewportWidth) ?
            vec2(1.f, float(kViewportHeight) / float(kViewportWidth)) :
            vec2(float(kViewportWidth) / float(kViewportHeight), 1.f));

        m_params.viewport.dims = ivec2(kViewportWidth, kViewportHeight);
        m_params.viewport.objectBounds = BBox2f(-boundHalf, boundHalf);

        Cascade({ kDirtySceneObjectChanged });        
    }

    __host__ Host::PathTracer::~PathTracer() noexcept
    {
        m_hostMeanAccumBuffer.DestroyAsset();
        m_hostVarAccumBuffer.DestroyAsset();
        m_hostDenoisedBuffer.DestroyAsset();
        m_hostTransforms.DestroyAsset();

        AssetAllocator::DestroyOnDevice(*this, cu_deviceInstance);
    }

    __host__ void Host::PathTracer::OnSynchroniseDrawableObject(const uint syncFlags)
    {
        // Only sync the objects if a SceneContainer has been bound
        if (syncFlags & kSyncObjects)
        {
            SynchroniseObjects<Device::PathTracer>(cu_deviceInstance, m_deviceObjects);
        }
        if (syncFlags & kSyncParams)
        {
            SynchroniseObjects<Device::PathTracer>(cu_deviceInstance, m_params);
            m_hostInstance.Synchronise(m_params);
        }
    }

    __host__ void Host::PathTracer::Bind(GenericObjectContainer& objects)
    {
        m_hostSceneContainer = objects.FindFirstOfType<Host::SceneContainer>();
        if (!m_hostSceneContainer)
        {
            Log::Warning("Warning! Path tracer '%s' could not bind to a valid SceneContainer object.", GetAssetID());
            m_params.hasValidScene = false;
        }
        else
        {
            // Copy the structure containing the scene object pointers 
            m_deviceObjects.scene = m_hostSceneContainer->GetDeviceObjects();

            if (m_hostSceneContainer->Cameras().empty())
            {
                Log::Warning("Warning! Path tracer '%s' found no cameras in the scene.");
                m_hostActiveCamera = nullptr;
            }
            else
            {
                m_hostActiveCamera = m_hostSceneContainer->Cameras().back();
                m_deviceObjects.activeCamera = m_hostActiveCamera->GetDeviceInstance();
            }

            m_params.hasValidScene = true;
        }

        Synchronise(kSyncParams | kSyncObjects);
    }

    __host__ void Host::PathTracer::Render()
    {
        if (!m_hostSceneContainer || !m_hostActiveCamera) { return; }

        //if (!IsClean()) { Synchronise(kSyncParams); }

        if (IsDirty(kDirtySceneObjectChanged))
        {
            m_hostMeanAccumBuffer->Clear(vec4(0.f));
            SignalDirty(kDirtyViewportRedraw);
        }

        //KernelPrepare << <1, 1 >> > (cu_deviceInstance, m_dirtyFlags);

        //if (RenderableObject::m_params.frameIdx > 10) return;

        dim3 blockSize, gridSize;
        KernelParamsFromImage(m_hostMeanAccumBuffer, blockSize, gridSize);

        // Accumulate the frame
        KernelRender << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceInstance);

        // Denoise if necessary
        /*if (m_params.frameIdx % 500 == 0)
        {
            KernelDenoise << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceInstance);
        }*/

        IsOk(hipDeviceSynchronize());

        // If there's no user interaction, signal the viewport to update intermittently to save compute
        constexpr float kViewportUpdateInterval = 1. / 2.f;
        if (m_redrawTimer.Get() > kViewportUpdateInterval)
        {
            SignalDirty(kDirtyViewportRedraw);
            m_redrawTimer.Reset();
        }

        if (m_renderTimer.Get() > 1.)
        {
            //Log::Debug("Frame: %i", RenderableObject::m_params.frameIdx);
            m_renderTimer.Reset();
        }
    }

    __host__ void Host::PathTracer::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage) const
    {
        dim3 blockSize, gridSize;
        KernelParamsFromImage(m_hostMeanAccumBuffer, blockSize, gridSize);

        KernelComposite << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceInstance, hostOutputImage->GetDeviceInstance(), m_hostSceneContainer != nullptr);
        IsOk(hipDeviceSynchronize());
    }

    __host__ void Host::PathTracer::Clear()
    {
        m_hostMeanAccumBuffer->Clear(vec4(0.f));

        Synchronise(kSyncParams);
    }

    __host__ bool Host::PathTracer::OnCreateDrawableObject(const std::string& stateID, const UIViewCtx& viewCtx, const vec2& mousePosObject)
    {
        if (stateID == "kCreateDrawableObjectOpen" || stateID == "kCreateDrawableObjectHover")
        {
            m_isConstructed = true;
            m_isFinalised = true;
            if (stateID == "kCreateDrawableObjectOpen") { Log::Success("Opened path tracer %s", GetAssetID()); }

            return true;
        }
        else if (stateID == "kCreateDrawableObjectAppend")
        {
            m_isFinalised = true;
            return true;
        }

        return false;
    }

    __host__ bool Host::PathTracer::OnRebuildDrawableObject()
    {
        /*m_scene = m_componentContainer->GenericObjects().FindFirstOfType<Host::SceneContainer>();
        if (!m_scene)
        {
            Log::Warning("Warning: path tracer '%s' expected an initialised scene container but none was found.");
        }*/

        return true;
    }

    __host__ bool Host::PathTracer::IsClickablePoint(const UIViewCtx& viewCtx) const
    {
        return GetWorldSpaceBoundingBox().Contains(viewCtx.mousePos);
    }

    __host__ BBox2f Host::PathTracer::ComputeObjectSpaceBoundingBox()
    {
        return m_params.viewport.objectBounds;
    }

    __host__ bool Host::PathTracer::Serialise(Json::Node& node, const int flags) const
    {
        DrawableObject::Serialise(node, flags);

        Json::Node lookNode = node.AddChildObject("viewport");
        lookNode.AddVector("dims", m_params.viewport.dims);

        return true;
    }

    __host__ bool Host::PathTracer::Deserialise(const Json::Node& node, const int flags)
    {
        bool isDirty = DrawableObject::Deserialise(node, flags);
        
        Json::Node viewportNode = node.GetChildObject("viewport", flags);
        if (viewportNode)
        {
            isDirty |= viewportNode.GetVector("dims", m_params.viewport.dims, flags);
        }

        if (isDirty)
        {
            SignalDirty({ kDirtyParams });
        }

        return isDirty;
    }

    __host__ bool Host::PathTracer::OnDelegateAction(const std::string& stateID, const VirtualKeyMap& keyMap, const UIViewCtx& viewCtx)
    {        
        const auto& bBox = GetWorldSpaceBoundingBox();
        const vec2 mouseNorm = (viewCtx.mousePos - bBox.Centroid()) / bBox.Dimensions();

        const float cameraPhi = kTwoPi * mix(-1., 1., mouseNorm.x);
        const float cameraDist = mix(0.5, 3., mouseNorm.y);
        const vec3 cameraPos = vec3(cos(cameraPhi), 0.5, sin(cameraPhi)) * cameraDist;

        Log::Debug("%s", cameraPos.format());
        
        m_hostActiveCamera->SetPosition(cameraPos);
        
        return true;
    }
}