#include "hip/hip_runtime.h"
#include "SceneBuilder.cuh"
#include "SceneContainer.cuh"

#include "cameras/PinholeCamera.cuh"
#include "materials/Material.cuh"
#include "lights/QuadLight.cuh"
#include "textures/Texture2D.cuh"
#include "tracables/Primitives.cuh"

#include "core/Vector.cuh"

#define kEmitterPos vec3(0., 0.5, 0.5)
#define kEmitterRot vec3(kHalfPi * 1.5, 0., 0.)
#define kEmitterSca 1.
#define kEmitterPower 2.
#define kEmitterRadiance (kOne * kEmitterPower / sqr(kEmitterSca))

namespace Enso
{
    __host__ bool Host::SceneBuilder::Rebuild(AssetHandle<Host::SceneContainer>& scene)
    {
        Log::Write("Rebuilding scene '%s'...", scene->GetAssetID());
        
        scene->DestroyManagedObjects();
        
        auto& cameras = scene->Cameras();
        auto& tracables = scene->Tracables();

        // Create the primary camera
        const float cameraPhi = -kPi;
        const vec3 cameraPos = vec3(cos(cameraPhi), 0.5, sin(cameraPhi)) * 2.;
        const vec3 cameraLookAt = vec3(0., -0., -0.);
        cameras.push_back(AssetAllocator::CreateChildAsset<Host::PinholeCamera>(*scene, "pinholecamera", cameraPos, cameraLookAt, 40.));

        // Create some materials


        constexpr int kNumSpheres = 7;
        BidirectionalTransform transform;
        tracables.resize(7);
        for (int sphereIdx = 0; sphereIdx < kNumSpheres; ++sphereIdx)
        {
            float phi = kTwoPi * (0.75f + float(sphereIdx) / float(kNumSpheres));
            transform = BidirectionalTransform(vec3(cos(phi), 0.f, sin(phi)) * 0.7f, kZero, 0.2f);
            
            tracables[sphereIdx] = AssetAllocator::CreateChildAsset<Host::Primitive<UnitSphereParams>>(*scene, tfm::format("sphere%i", sphereIdx), transform, 5, UnitSphereParams{});
        }

        // Ground plane        
        transform = BidirectionalTransform(vec3(0.f, -0.2f, 0.f), vec3(-kHalfPi, 0.f, 0.f), 2.f);
        tracables.push_back(AssetAllocator::CreateChildAsset<Host::Primitive<PlaneParams>>(*scene, "groundplane", transform, 5, PlaneParams{ true }));
       
        // Emitter plane
        transform = BidirectionalTransform(kEmitterPos, kEmitterRot, kEmitterSca);
        tracables.push_back(AssetAllocator::CreateChildAsset<Host::Primitive<PlaneParams>>(*scene, "emitterplane", transform, 5, PlaneParams{ true }));

        for (auto& t : tracables) Log::Debug("  - %i", t.GetReferenceCount());


        // Synchronise the scene
        scene->Synchronise(kSyncObjects);
        return true;
    }    
}