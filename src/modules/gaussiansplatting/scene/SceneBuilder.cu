#include "hip/hip_runtime.h"
#include "SceneBuilder.cuh"
#include "SceneContainer.cuh"

#include "cameras/PinholeCamera.cuh"

#include "lights/QuadLight.cuh"

#include "textures/ProceduralTexture.cuh"
#include "textures/TextureMap.cuh"

#include "tracables/Primitives.cuh"

#include "materials/Diffuse.cuh"
#include "materials/GlossyConductor.cuh"
#include "materials/SpecularDielectric.cuh"

#include "core/containers/Vector.cuh"

#define kEmitterPos vec3(0., 0.5, 0.5)
#define kEmitterRot vec3(kHalfPi * 1.5, 0., 0.)
#define kEmitterSca 1.
#define kEmitterPower 2.
#define kEmitterRadiance (kOne * kEmitterPower / sqr(kEmitterSca))

namespace Enso
{
    __host__ Host::SceneBuilder::SceneBuilder()
    {
    }

    __host__ bool Host::SceneBuilder::Rebuild(AssetHandle<Host::SceneContainer>& scene)
    {
        Log::Write("Rebuilding scene '%s'...", scene->GetAssetID());
        
        scene->DestroyManagedObjects();
       
        // Create the primary camera
        const float cameraPhi = -kPi;
        const vec3 cameraLookAt = vec3(0., 0.1, -0.);
        const vec3 cameraPos = vec3(cos(cameraPhi), 0.5, sin(cameraPhi)) * 2. + cameraLookAt;
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::PinholeCamera>(*scene, "pinholecamera", cameraPos, cameraLookAt, 35.f));
        
        constexpr int kNumPrims = 7;

        // Create some textures
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::TextureMap>(*scene, "floortexture", "C:\\projects\\enso\\data\\Texture1.exr"));
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::TextureMap>(*scene, "grace", "C:\\projects\\enso\\data\\Grace.exr"));

        // Create some materials
        for (int primIdx = 0; primIdx < kNumPrims; ++primIdx)
        {
            vec3 colour = Hue((0.5f + float(primIdx)) / float(kNumPrims));
            if (primIdx >= kNumPrims - 2)
            {
                scene->Emplace(AssetAllocator::CreateChildAsset<Host::SpecularDielectric>(*scene, tfm::format("primmaterial%i", primIdx), scene, SpecularDielectricParams(1.5f, 5.f, kOne - colour)));
            }
            else
            {
                const float alpha = mix(0.01f, 0.5f, sqr(float(primIdx) / float(kNumPrims - 2)));
                scene->Emplace(AssetAllocator::CreateChildAsset<Host::GlossyConductor>(*scene, tfm::format("primmaterial%i", primIdx), scene, GlossyConductorParams(colour, alpha)));
            }

        }
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::GlossyConductor>(*scene, "floormaterial", scene, GlossyConductorParams(vec3(0.5f), 0.5f, vec2(0.01, 0.5f), kInvalidMaterial, scene->FindAssetIdx("floortexture"))));

        BidirectionalTransform transform;
        for (int primIdx = 0; primIdx < kNumPrims; ++primIdx)
        {
            float phi = kTwoPi * (0.75f + float(primIdx) / float(kNumPrims));
            transform = BidirectionalTransform(vec3(cos(phi), 0.f, sin(phi)) * 0.7f, kZero, 0.2f);
            
            switch (primIdx % 3)
            {
            case 0:
                scene->Emplace(AssetAllocator::CreateChildAsset<Host::Primitive<UnitSphereParams>>(*scene, tfm::format("ring%i", primIdx), transform, primIdx, UnitSphereParams()));
                break;
            case 1:
                scene->Emplace(AssetAllocator::CreateChildAsset<Host::Primitive<BoxParams>>(*scene, tfm::format("ring%i", primIdx), transform, primIdx, BoxParams(vec3(1.0f))));
                break;
            case 2:
                scene->Emplace(AssetAllocator::CreateChildAsset<Host::Primitive<CylinderParams>>(*scene, tfm::format("ring%i", primIdx), transform, primIdx, CylinderParams(1.f)));
                break;
            }
        }

        // Ground plane        
        transform = BidirectionalTransform(vec3(0.f, -0.2f, 0.f), vec3(-kHalfPi, 0.f, 0.f), 2.f);
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::Primitive<PlaneParams>>(*scene, "groundplane", transform, kNumPrims, PlaneParams(true, false)));

        // Emitter plane
        transform = BidirectionalTransform(kEmitterPos, kEmitterRot, kEmitterSca);
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::Primitive<PlaneParams>>(*scene, "emitterplane", transform, kInvalidMaterial, PlaneParams(true, true)));

        // Light sampler
        auto emitterTracable = scene->Find<Host::Tracable>("emitterplane");
        scene->Emplace(AssetAllocator::CreateChildAsset<Host::QuadLight>(*scene, "emittersampler", kOne, emitterTracable));

        // Set the environment light
        scene->SetEnvironmentTexture("grace");

        // Finalise the scene
        //scene->Finalise();         

        // Synchronise the newly created scene objects
        scene->Synchronise(kSyncObjects);
        return true;
    }   
}