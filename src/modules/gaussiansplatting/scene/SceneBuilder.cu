#include "hip/hip_runtime.h"
#include "SceneBuilder.cuh"
#include "SceneContainer.cuh"

#include "cameras/PinholeCamera.cuh"
#include "materials/Material.cuh"
#include "lights/QuadLight.cuh"
#include "textures/TextureMap.cuh"
#include "tracables/Primitives.cuh"

#include "core/containers/Vector.cuh"

#define kEmitterPos vec3(0., 0.5, 0.5)
#define kEmitterRot vec3(kHalfPi * 1.5, 0., 0.)
#define kEmitterSca 1.
#define kEmitterPower 2.
#define kEmitterRadiance (kOne * kEmitterPower / sqr(kEmitterSca))

namespace Enso
{
    __host__ Host::SceneBuilder::SceneBuilder()
    {
    }

    __host__ bool Host::SceneBuilder::Rebuild(AssetHandle<Host::SceneContainer>& scene)
    {
        Log::Write("Rebuilding scene '%s'...", scene->GetAssetID());
        
        scene->DestroyManagedObjects();
        
        auto& cameras = scene->Cameras();
        auto& tracables = scene->Tracables();
        auto& textures = scene->Textures();

        // Create the primary camera
        const float cameraPhi = -kPi;
        const vec3 cameraLookAt = vec3(0., 0.1, -0.);
        const vec3 cameraPos = vec3(cos(cameraPhi), 0.5, sin(cameraPhi)) * 2. + cameraLookAt;
        cameras.push_back(AssetAllocator::CreateChildAsset<Host::PinholeCamera>(*scene, "pinholecamera", cameraPos, cameraLookAt, 35.f));

        // Create some textures
        textures.push_back(AssetAllocator::CreateChildAsset<Host::TextureMap>(*scene, "floortexture", "C:\\projects\\enso\\data\\Texture1.exr"));
        textures.push_back(AssetAllocator::CreateChildAsset<Host::TextureMap>(*scene, "grace", "C:\\projects\\enso\\data\\Grace.exr"));

        constexpr int kNumPrims = 7;
        BidirectionalTransform transform;
        tracables.resize(7);
        for (int primIdx = 0; primIdx < kNumPrims; ++primIdx)
        {
            float phi = kTwoPi * (0.75f + float(primIdx) / float(kNumPrims));
            transform = BidirectionalTransform(vec3(cos(phi), 0.f, sin(phi)) * 0.7f, kZero, 0.2f);
            
            switch (primIdx % 3)
            {
            case 0:
                tracables[primIdx] = AssetAllocator::CreateChildAsset<Host::Primitive<UnitSphereParams>>(*scene, tfm::format("sphere%i", primIdx), transform, 5, UnitSphereParams());
                break;
            case 1:
                tracables[primIdx] = AssetAllocator::CreateChildAsset<Host::Primitive<BoxParams>>(*scene, tfm::format("box%i", primIdx), transform, 5, BoxParams(vec3(1.0f)));
                break;
            case 2:
                tracables[primIdx] = AssetAllocator::CreateChildAsset<Host::Primitive<CylinderParams>>(*scene, tfm::format("cylinder%i", primIdx), transform, 5, CylinderParams(1.f));
                break;
            }       
        }

        // Ground plane        
        transform = BidirectionalTransform(vec3(0.f, -0.2f, 0.f), vec3(-kHalfPi, 0.f, 0.f), 2.f);
        tracables.push_back(AssetAllocator::CreateChildAsset<Host::Primitive<PlaneParams>>(*scene, "groundplane", transform, 5, PlaneParams{ true }));
       
        // Emitter plane
        transform = BidirectionalTransform(kEmitterPos, kEmitterRot, kEmitterSca);
        tracables.push_back(AssetAllocator::CreateChildAsset<Host::Primitive<PlaneParams>>(*scene, "emitterplane", transform, 5, PlaneParams{ true }));

        for (auto& t : tracables) Log::Debug("  - %i", t.GetReferenceCount());


        // Synchronise the scene
        scene->Synchronise(kSyncObjects);
        return true;
    }    
}