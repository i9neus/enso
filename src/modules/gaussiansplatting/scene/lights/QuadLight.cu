#include "hip/hip_runtime.h"
#include "QuadLight.cuh"

namespace Enso
{    
    __device__ float Device::QuadLight::Sample(const Ray& incident, Ray& extant, const HitCtx& hit, const vec2& xi)
    {
        // Sample a point on the light 
        vec3 hitPos = incident.Point();

        //vec2 xi = vec2(0.0);
        //uint hash = HashOf(uint(gFragCoord.x), uint(gFragCoord.y));
        //vec2 xi = vec2(HaltonBase2(hash + uint(sampleIdx)), HaltonBase3(hash + uint(sampleIdx))) - 0.5;

        vec3 lightPos = Tracable::m_params.transform.inv * vec3(xi - 0.5f, 0.f) * Tracable::m_params.transform.sca + Tracable::m_params.transform.trans;
        //lightPos = Tracable::m_params.transform.trans;

        // Compute the normalised extant direction based on the light position local to the shading point
        vec3 outgoing = lightPos - hitPos;
        float lightDist = length(outgoing);
        outgoing /= lightDist;

        // Test if the emitter is behind the shading point
        if (dot(outgoing, hit.n) <= 0.f) { return 0.0f; }

        vec3 lightNormal = normalize(Tracable::m_params.transform.inv * vec3(0.0f, 0.0f, 1.0f));
        float cosPhi = dot(normalize(hitPos - lightPos), lightNormal);

        // Test if the emitter is rotated away from the shading point
        if (cosPhi < 0.f) { return 0.0f; }

        // Compute the projected solid angle of the light        
        float solidAngle = cosPhi * sqr(Tracable::m_params.transform.sca) / fmaxf(1e-10f, sqr(lightDist));

        // Create the ray from the sampled BRDF direction
        extant.Construct(hitPos,
            outgoing,
            //(IsBackfacing(ray) ? hit.n : hit.n) * hit.kickoff,
            hit.n * 1e-4f,
            incident.weight * Light::m_params.radiance * solidAngle,
            incident.depth + 1,
            kRayDirectSampleLight);

        return 1.0f / fmaxf(1e-10f, solidAngle);
    }

    __device__  float Device::QuadLight::Evaluate(Ray& extant, const HitCtx& hit)
    {
        RayBasic localRay = Tracable::m_params.transform.RayToObjectSpace(extant.od);
        if (fabsf(localRay.d.z) < 1e-10f) { return 0.0f; }

        float t = localRay.o.z / -localRay.d.z;

        const vec2 uv = (localRay.o.xy + localRay.d.xy * t) + 0.5f;
        if (cwiseMin(uv) < 0.0 || cwiseMax(uv) > 1.0) { return 0.0f; }

        const vec3 lightNormal = normalize(Tracable::m_params.transform.inv * vec3(0.0f, 0.0f, 1.0f));
        const vec3 lightPos = extant.PointAt(t);

        const float cosPhi = dot(normalize(extant.od.o - lightPos), lightNormal);

        // Test if the emitter is rotated away from the shading point
        if (cosPhi < 0.f) { return 0.0f; }

        float solidAngle = cosPhi * sqr(Tracable::m_params.transform.sca) / fmaxf(1e-10f, sqr(t));

        //if(!IsVolumetricBxDF(hit))
        {
            const float cosTheta = dot(hit.n, extant.od.d);
            if (cosTheta < 0.0f) { return 0.0f; }

            solidAngle *= cosTheta;
        }

        extant.weight *= Light::m_params.radiance;
        return 1.0f / fmaxf(1e-10f, solidAngle);
    }
    
    __host__ Host::QuadLight::QuadLight(const Asset::InitCtx& initCtx) :
        Host::Light(initCtx),
        cu_deviceInstance(AssetAllocator::InstantiateOnDevice<Device::QuadLight>(*this))
    {
        Light::SetDeviceInstance(AssetAllocator::StaticCastOnDevice<Device::Light>(cu_deviceInstance));

        Synchronise(kSyncObjects);
    }
}