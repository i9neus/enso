#include "hip/hip_runtime.h"
#pragma once

#include "ModuleInterface.cuh"
#include "io/CommandQueue.h"

namespace Enso
{
	ModuleInterface::ModuleInterface(std::shared_ptr<CommandQueue> outQueue) :
		m_frameTimes(20),
		m_mouseWheelAngle(0.0f),
		m_clientWidth(1.0f),
		m_clientHeight(1.0f),
		m_uiGraph(m_keyCodes),
		m_renderSemaphore(kRenderManagerD3DBlitFinished),
		m_outboundCmdQueue(outQueue)
	{
		m_mouse.pos = std::numeric_limits<int>::min();
		m_mouse.prevPos = std::numeric_limits<int>::min();
		m_mouse.delta = 0.0f;

		m_uiEventQueue.maxEvents = 100;
		m_uiEventQueue.autoFlushAfterEvents = -1;
	}

	ModuleInterface::~ModuleInterface()
	{

	}

	void ModuleInterface::SetCudaObjects(AssetHandle<Host::ImageRGBA>& compositeImage, hipStream_t renderStream)
	{
		m_compositeImage = compositeImage;
		m_renderStream = renderStream;
	}

	void ModuleInterface::Initialise(const UINT clientWidth, const UINT clientHeight)
	{
		SetClientSize(clientWidth, clientHeight);

		OnInitialise();
	}

	void ModuleInterface::Destroy()
	{
		// Stop and clean up the renderer object
		Stop();
		OnDestroy();
	}

	void ModuleInterface::Start()
	{
		Log::Write("Starting %s...\b", GetRendererName());

		m_threadSignal = kRenderManagerRun;
		m_managerThread = std::thread(std::bind(&ModuleInterface::RunThread, this));

		m_renderStartTime = std::chrono::high_resolution_clock::now();

		Assert(m_managerThread.joinable());

		Log::Success("Okay!");
	}

	void ModuleInterface::Stop()
	{
		if (!m_managerThread.joinable() || m_threadSignal != kRenderManagerRun) { return; }

		Log::Indent indent(tfm::format("Halting %s...\r", GetRendererName()));

		m_threadSignal.store(kRenderManagerHalt);
		m_managerThread.join();

		Log::Success("Successfully halted '%s'!", GetRendererName());
	}

	void ModuleInterface::RunThread()
	{
		checkCudaErrors(hipStreamSynchronize(m_renderStream));

		// Notify the inheriting class that the render is about to start
		{
			//std::lock_guard<std::mutex> lock(m_resourceMutex);
			OnPreRender();
		}

		m_frameIdx = 0;
		//#define DISABLE_EXCEPTION_HANDLING
#ifndef DISABLE_EXCEPTION_HANDLING
		try
		{
#endif
			while (m_threadSignal.load() == kRenderManagerRun)
			{
				HighResolutionTimer timer;

				if (m_inboundCmdQueue && !m_inboundCmdQueue->IsEmpty())
				{
					OnCommandsWaiting(*m_inboundCmdQueue);
				}

				// Notify that a render "tick" has begun
				{
					//std::lock_guard<std::mutex> lock(m_resourceMutex);
					OnRender();
				}

				// Compute some stats on the framerate
				m_frameIdx++;
				m_lastFrameTime = timer.Get();
				m_frameTimes[m_frameIdx % m_frameTimes.size()] = m_lastFrameTime;
				m_meanFrameTime = 0.0f;
				for (const auto& ft : m_frameTimes)
				{
					m_meanFrameTime += ft;
				}
				m_meanFrameTime /= min(m_frameIdx, int(m_frameTimes.size()));
			}

#ifndef DISABLE_EXCEPTION_HANDLING
		}
		catch (const std::runtime_error& err)
		{
			Log::Error("Runtime error: %s\n", err.what());
			StackBacktrace::Print();
		}
		catch (...)
		{
			Log::Error("Unhandled error");
			StackBacktrace::Print();
		}
#endif

		// Notify that the render has completed
		{
			//std::lock_guard<std::mutex> lock(m_resourceMutex);
			OnPostRender();
		}

		// Signal that the renderer has finished
		m_threadSignal.store(kRenderManagerIdle);
	}

	bool ModuleInterface::Poll(Json::Document& stateJson)
	{
		stateJson.Clear();

		// Add some generic data about the renderer that's exported each time the state is polled
		Json::Node managerJson = stateJson.AddChildObject("renderer");
		managerJson.AddValue("frameIdx", m_frameIdx);
		managerJson.AddValue("smoothedFrameTime", m_meanFrameTime);
		managerJson.AddValue("smoothedFPS", 1.0f / m_meanFrameTime);
		managerJson.AddValue("lastFrameTime", m_lastFrameTime);
		managerJson.AddValue("lastFPS", 1.0f / m_lastFrameTime);
		const int threadSignal = m_threadSignal;
		managerJson.AddValue("rendererStatus", threadSignal);

		return true;
	}

	template<typename T>
	T ModuleInterface::PopUIEventQueue(std::deque<T>& queue)
	{
		Assert(!queue.empty());
		const T item = queue.front();
		queue.pop_front();
		return item;
	}

	template<typename T>
	void ModuleInterface::PushUIEventQueue(const int event, std::deque<T>& queue, const T& newItem)
	{
		m_controlQueueMutex.lock();

		// If the control queue hasn't been purged, just replace the most recent event
		if (m_uiEventQueue.events.size() >= m_uiEventQueue.maxEvents)
		{
			Log::Debug("Warning: UI control queue exceeded max size of %i events", m_uiEventQueue.maxEvents);

			m_uiEventQueue.events.back() = event;
			queue.back() = newItem;
		}
		// Otherwise, push the event to the deque
		else
		{
			m_uiEventQueue.events.push_back(event);
			queue.push_back(newItem);
		}

		m_controlQueueMutex.unlock();

		// If the queue is full and auto-flush is enabled, flush everything now. 
		if (m_uiEventQueue.autoFlushAfterEvents >= 0 && m_uiEventQueue.events.size() >= m_uiEventQueue.autoFlushAfterEvents)
		{
			FlushUIEventQueue();
		}
	}

	void ModuleInterface::FlushUIEventQueue()
	{
		if (m_uiEventQueue.events.empty()) { return; }

		std::lock_guard<std::mutex> lock(m_controlQueueMutex);

		// Dispatch queued events in the order that they were posted
		while (!m_uiEventQueue.events.empty())
		{
			const int event = m_uiEventQueue.events.front();
			m_uiEventQueue.events.pop_front();

			switch (event)
			{
			case kControlEventKeyboard:
			{
				const auto keyButton = PopUIEventQueue(m_uiEventQueue.keyButton);

				m_keyCodes.Update(keyButton.first, keyButton.second);

				// Notify the superclass that a key state has changed
				OnKey(keyButton.first, false, keyButton.second);

				if (keyButton.first == VK_ESCAPE)
				{
					m_uiGraph.Reset();
				}
				else
				{
					m_uiGraph.OnTriggerTransition(kUITriggerOnKeyboard);
				}
			}
			break;

			case kControlEventMouseMove:
			{
				const auto mousePos = PopUIEventQueue(m_uiEventQueue.mouseMove);

				// Update the mouse position information
				m_mouse.prevPos = (m_mouse.pos.x == std::numeric_limits<int>::min()) ? ivec2(mousePos.x, m_clientHeight - 1 - mousePos.y) : m_mouse.pos;
				m_mouse.pos = ivec2(mousePos.x, m_clientHeight - 1 - mousePos.y);
				m_mouse.delta = m_mouse.pos - m_mouse.prevPos;

				// Notify the superclass that a mouse state has changed
				OnMouseMove();

				m_keyCodes.Update();
				m_uiGraph.OnTriggerTransition(kUITriggerOnMouseMove);
			}
			break;

			case kControlEventMouseButton:
			{
				const auto mouseButton = PopUIEventQueue(m_uiEventQueue.mouseButton);

				// Notify the superclass that a mouse state has changed
				OnMouseButton(mouseButton.first, mouseButton.second);

				// TODO: Calling Update() here feels messy and brittle. Should the UI graph have ownership of the codes?
				m_keyCodes.Update(mouseButton.first, mouseButton.second);
				m_uiGraph.OnTriggerTransition(kUITriggerOnMouseButton);
			}
			break;

			case kControlEventMouseWheel:
			{
				m_mouseWheelAngle = PopUIEventQueue(m_uiEventQueue.mouseWheel);

				// Notify the superclass that a mouse wheel state has changed
				OnMouseWheel();

				m_keyCodes.Update();
				m_uiGraph.OnTriggerTransition(kUITriggerOnMouseWheel);
			}
			break;

			default:
				AssertMsgFmt(false, "Unrecognised UI control event %i", event);
			}
		}

		// Sanity check
		Assert(m_uiEventQueue.keyButton.empty());
		Assert(m_uiEventQueue.mouseButton.empty());
		Assert(m_uiEventQueue.mouseMove.empty());
		Assert(m_uiEventQueue.mouseWheel.empty());

		/*m_uiEventQueue.keyButton.clear();
		m_uiEventQueue.mouseButton.clear();
		m_uiEventQueue.mouseMove.clear();
		m_uiEventQueue.mouseWheel.clear();*/
	}

	void ModuleInterface::SetKey(const uint code, const bool isSysKey, const bool isDown)
	{
		PushUIEventQueue(kControlEventKeyboard, m_uiEventQueue.keyButton, std::make_pair(code, isDown));
	}

	void ModuleInterface::SetMouseButton(const uint code, const bool isDown)
	{
		PushUIEventQueue(kControlEventMouseButton, m_uiEventQueue.mouseButton, std::make_pair(code, isDown));
	}

	void ModuleInterface::SetMousePos(const int mouseX, const int mouseY, const WPARAM flags)
	{
		PushUIEventQueue(kControlEventMouseMove, m_uiEventQueue.mouseMove, ivec2(mouseX, mouseY));
	}

	void ModuleInterface::SetMouseWheel(const float angle)
	{
		PushUIEventQueue(kControlEventMouseWheel, m_uiEventQueue.mouseWheel, angle);
	}

	void ModuleInterface::SetClientSize(const int width, const int height)
	{
		m_clientWidth = width;
		m_clientHeight = height;

		m_clientToNormMatrix = mat3::Identity();
		m_clientToNormMatrix.i00 = 1.0f / height;
		m_clientToNormMatrix.i11 = 1.0f / height;
		m_clientToNormMatrix.i02 = -0.5f * float(width) / float(height);
		m_clientToNormMatrix.i12 = -0.5f;

		OnResizeClient();
	}

	void ModuleInterface::FocusChange(const bool isSet)
	{	
		// Notify the deriving class that the focus has changed so it can do clean-up
		OnFocusChange(isSet);

		// Reset the state and UI graph
		m_keyCodes.Clear();
		m_uiGraph.Reset();
		Log::Debug(isSet ? "Focus set" : "Focus lost");
	}

	void ModuleInterface::OnCommandsWaiting(CommandQueue& inbound) 
	{ 
		inbound.Clear(); 
	}
}
