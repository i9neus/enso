#include "hip/hip_runtime.h"
#include "GI2DRenderer.cuh"

#include "core/math/Math.cuh"
#include "core/math/ColourUtils.cuh"
#include "core/GenericObjectContainer.cuh"
#include "core/Vector.cuh"
#include "core/Tuple.cuh"

#include "tracables/Tracable.cuh"
#include "tracables/Curve.cuh"
#include "lights/OmniLight.cuh"
#include "widgets/UIInspector.cuh" 
#include "SceneDescription.cuh"
#include "integrators/VoxelProxyGrid.cuh"
#include "layers/OverlayLayer.cuh"
#include "layers/PathTracerLayer.cuh"
#include "layers/VoxelProxyGridLayer.cuh"

#include "io/SerialisableObjectSchema.h"

//#include "kernels/gi2d/ObjectDebugger.cuh"

namespace Enso
{

    __host__ GI2DRenderer::GI2DRenderer(std::shared_ptr<CommandQueue> outQueue) :
        ModuleInterface(outQueue),
        m_isRunning(true)
    {
        // Load the object schema
        SerialisableObjectSchemaContainer::Load("schema.json");

        // Register the outbound commands
        m_outboundCmdQueue->RegisterCommand("OnCreateObject");
        m_outboundCmdQueue->RegisterCommand("OnUpdateObject");
        m_outboundCmdQueue->RegisterCommand("OnDeleteObject");

        // Register the inbound command handlers
        m_commandManager.RegisterEventHandler("OnUpdateObject", this, &GI2DRenderer::OnInboundUpdateObject);

        m_sceneObjectFactory.RegisterInstantiator<Host::Curve>(VirtualKeyMap({ {'Q', kOnButtonDepressed}, {VK_CONTROL, kButtonDown} }).HashOf());
        m_sceneObjectFactory.RegisterInstantiator<Host::OmniLight>(VirtualKeyMap({ {'W', kOnButtonDepressed}, {VK_CONTROL, kButtonDown} }).HashOf());

        m_uiGraph.DeclareState("kIdleState", this, &GI2DRenderer::OnIdleState);

        // Create scene object
        m_uiGraph.DeclareState("kCreateSceneObjectOpen", this, &GI2DRenderer::OnCreateSceneObject);
        m_uiGraph.DeclareState("kCreateSceneObjectHover", this, &GI2DRenderer::OnCreateSceneObject);
        m_uiGraph.DeclareState("kCreateSceneObjectAppend", this, &GI2DRenderer::OnCreateSceneObject);
        m_uiGraph.DeclareState("kCreateSceneObjectClose", this, &GI2DRenderer::OnCreateSceneObject);
        m_uiGraph.DeclareDeterministicTransition("kIdleState", "kCreateSceneObjectOpen", VirtualKeyMap({ {'Q', kOnButtonDepressed}, {VK_CONTROL, kButtonDown} }), 0);
        m_uiGraph.DeclareDeterministicTransition("kIdleState", "kCreateSceneObjectOpen", VirtualKeyMap({ {'W', kOnButtonDepressed}, {VK_CONTROL, kButtonDown} }), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kCreateSceneObjectOpen", "kCreateSceneObjectHover");
        m_uiGraph.DeclareDeterministicTransition("kCreateSceneObjectHover", "kCreateSceneObjectHover", nullptr, kUITriggerOnMouseMove);
        m_uiGraph.DeclareDeterministicTransition("kCreateSceneObjectHover", "kCreateSceneObjectAppend", VirtualKeyMap(VK_LBUTTON, kOnButtonDepressed), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kCreateSceneObjectAppend", "kCreateSceneObjectHover");
        m_uiGraph.DeclareDeterministicTransition("kCreateSceneObjectHover", "kCreateSceneObjectClose", VirtualKeyMap(VK_RBUTTON, kOnButtonDepressed), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kCreateSceneObjectClose", "kIdleState");

        // Select/deselect scene object
        m_uiGraph.DeclareState("kSelectSceneObjectDragging", this, &GI2DRenderer::OnSelectSceneObjects);
        m_uiGraph.DeclareState("kSelectSceneObjectEnd", this, &GI2DRenderer::OnSelectSceneObjects);
        m_uiGraph.DeclareState("kDeselectSceneObject", this, &GI2DRenderer::OnSelectSceneObjects);
        m_uiGraph.DeclareNonDeterministicTransition("kIdleState", VirtualKeyMap(VK_LBUTTON, kOnButtonDepressed), 0, this, &GI2DRenderer::DecideOnClickState);
        m_uiGraph.DeclareDeterministicTransition("kSelectSceneObjectDragging", "kSelectSceneObjectDragging", VirtualKeyMap(VK_LBUTTON, kButtonDown), kUITriggerOnMouseMove);
        m_uiGraph.DeclareDeterministicTransition("kSelectSceneObjectDragging", "kSelectSceneObjectEnd", VirtualKeyMap(VK_LBUTTON, kOnButtonReleased), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kSelectSceneObjectEnd", "kIdleState");
        m_uiGraph.DeclareDeterministicTransition("kIdleState", "kDeselectSceneObject", VirtualKeyMap(VK_RBUTTON, kOnButtonDepressed), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kDeselectSceneObject", "kIdleState");

        // Move scene object
        m_uiGraph.DeclareState("kMoveSceneObjectBegin", this, &GI2DRenderer::OnMoveSceneObject);
        m_uiGraph.DeclareState("kMoveSceneObjectDragging", this, &GI2DRenderer::OnMoveSceneObject);
        m_uiGraph.DeclareState("kMoveSceneObjectEnd", this, &GI2DRenderer::OnMoveSceneObject);
        //m_uiGraph.DeclareNonDeterministicTransition("kIdleState", nullptr, MouseButtonMap(VK_LBUTTON, kOnButtonDepressed), 0, this, &GI2DRenderer::DecideOnClickState);
        m_uiGraph.DeclareDeterministicAutoTransition("kMoveSceneObjectBegin", "kMoveSceneObjectDragging");
        m_uiGraph.DeclareDeterministicTransition("kMoveSceneObjectDragging", "kMoveSceneObjectDragging", VirtualKeyMap(VK_LBUTTON, kButtonDown), kUITriggerOnMouseMove);
        m_uiGraph.DeclareDeterministicTransition("kMoveSceneObjectDragging", "kMoveSceneObjectEnd", VirtualKeyMap(VK_LBUTTON, kOnButtonReleased), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kMoveSceneObjectEnd", "kIdleState");

        // Delete scene object
        m_uiGraph.DeclareState("kDeleteSceneObjects", this, &GI2DRenderer::OnDeleteSceneObject);
        m_uiGraph.DeclareDeterministicTransition("kIdleState", "kDeleteSceneObjects", VirtualKeyMap({ {VK_DELETE, kOnButtonDepressed} }), 0);
        m_uiGraph.DeclareDeterministicTransition("kIdleState", "kDeleteSceneObjects", VirtualKeyMap({ {VK_BACK, kOnButtonDepressed} }), 0);
        m_uiGraph.DeclareDeterministicAutoTransition("kDeleteSceneObjects", "kIdleState");

        // Utils
        m_uiGraph.DeclareState("kToggleRun", this, &GI2DRenderer::OnToggleRun);
        m_uiGraph.DeclareDeterministicTransition("kIdleState", "kToggleRun", VirtualKeyMap({ {VK_SPACE, kOnButtonDepressed} }), 0);

        m_uiGraph.Finalise();
    }

    __host__ GI2DRenderer::~GI2DRenderer()
    {
        Destroy();
    }

    std::shared_ptr<ModuleInterface> GI2DRenderer::Instantiate(std::shared_ptr<CommandQueue> outQueue)
    {
        return std::make_shared<GI2DRenderer>(outQueue);
    }

    __host__ void GI2DRenderer::OnInitialise()
    {
        m_viewCtx.transform = ViewTransform2D(m_clientToNormMatrix, vec2(0.f), 0.f, 1.0f);
        m_viewCtx.dPdXY = length(vec2(m_viewCtx.transform.matrix.i00, m_viewCtx.transform.matrix.i10));
        m_viewCtx.zoomSpeed = 10.0f;
        m_viewCtx.sceneBounds = BBox2f(vec2(-0.5f), vec2(0.5f));

        //m_primitiveContainer.Create(m_renderStream);

        m_sceneObjects = CreateAsset<GenericObjectContainer>(":gi2d/renderObjects");
        m_scene = CreateAsset<Host::SceneDescription>(":gi2d/sceneDescription");

        m_overlayRenderer = CreateAsset<Host::OverlayLayer>(":gi2d/overlay", m_scene, m_clientWidth, m_clientHeight, m_renderStream);
        m_pathTracerLayer = CreateAsset<Host::PathTracerLayer>(":gi2d/pathTracerLayer", m_scene, m_clientWidth, m_clientHeight, 2, m_renderStream);
        m_voxelProxyGridLayer = CreateAsset<Host::VoxelProxyGridLayer>(":gi2d/voxelProxyGridLayer", m_scene, 100, 100);

        //m_isosurfaceExplorer = CreateAsset<Host::IsosurfaceExplorer>(":gi2d/isosurfaceExplorer", m_scene, m_clientWidth, m_clientHeight, 1, m_renderStream);

        SetDirtyFlags(kDirtyAll);

        if (m_dirtyFlags)
        {
            Rebuild();
        }
    }

    __host__ void GI2DRenderer::OnDestroy()
    {
        m_overlayRenderer.DestroyAsset();
        m_pathTracerLayer.DestroyAsset();
        m_voxelProxyGridLayer.DestroyAsset();
        //m_isosurfaceExplorer.DestroyAsset();

        //m_scene->voxelProxy.DestroyAsset();

        m_scene.DestroyAsset();
        m_sceneObjects.DestroyAsset();
    }

    __host__ void GI2DRenderer::Rebuild()
    {
        std::lock_guard<std::mutex> lock(m_resourceMutex);

        if (!m_dirtyFlags) { return; }

        m_scene->Rebuild(m_sceneObjects, m_viewCtx, m_dirtyFlags);

        // View has changed
        m_overlayRenderer->Rebuild(m_dirtyFlags, m_viewCtx, m_selectionCtx);
        m_pathTracerLayer->Rebuild(m_dirtyFlags, m_viewCtx, m_selectionCtx);
        m_voxelProxyGridLayer->Rebuild(m_dirtyFlags, m_viewCtx, m_selectionCtx);
        //m_isosurfaceExplorer->Rebuild(m_dirtyFlags, m_viewCtx, m_selectionCtx);

        //m_scene->voxelProxy->Rebuild(m_dirtyFlags, m_viewCtx);

        SetDirtyFlags(kDirtyAll, false);
    }

    __host__ void GI2DRenderer::OnInboundUpdateObject(const Json::Node& node)
    {
        for (Json::Node::ConstIterator nodeIt = node.begin(); nodeIt != node.end(); ++nodeIt)
        {
            const std::string& objId = nodeIt.Name();
            auto objectHandle = m_sceneObjects->FindByID(objId);

            if (!objectHandle)
            {
                Log::Warning("Error: '%s' is not a valid scene object.", objId);
                continue;
            }

            const uint dirtyFlags = objectHandle->Deserialise(*nodeIt, Json::kRequiredWarn);
            SetDirtyFlags(dirtyFlags);
        }
    }

    __host__ void GI2DRenderer::EnqueueObjects(const std::string& eventId, const int flags, const AssetHandle<Host::SceneObject> asset)
    {
        if (!m_outboundCmdQueue->IsRegistered(eventId)) { return; }

        // Lambda to do the actual serialisation
        auto SerialiseImpl = [&](Json::Node& node, const AssetHandle<Host::SceneObject>& obj) -> void
        {
            // Create a new child object and add its class ID for the schema
            Json::Node childNode = node.AddChildObject(obj->GetAssetID());
            const std::string assetClass = obj->GetAssetClass();
            AssertMsgFmt(!assetClass.empty(), "Error: asset '%s' has no defined class", obj->GetAssetClass());
            childNode.AddValue("class", assetClass);

            // Deleted objects don't need their full attribute list serialised
            if (!(flags & kEnqueueIdOnly))
            {
                m_onCreate.newObject->Serialise(childNode, kSerialiseExposedOnly);
            }
        };

        Json::Node node = m_outboundCmdQueue->Create(eventId);
        if (flags & kEnqueueAll)
        {
            for (auto& obj : *m_sceneObjects) { SerialiseImpl(node, obj.DynamicCast<Host::SceneObject>()); }
        }
        else if (flags & kEnqueueSelected)
        {
            for (auto& obj : m_selectedTracables) { SerialiseImpl(node, obj.DynamicCast<Host::SceneObject>()); }
        }
        else if (flags & kEnqueueOne)
        {
            SerialiseImpl(node, asset);
        }

        m_outboundCmdQueue->Enqueue();  // Enqueue the staged command
    }

    __host__ uint GI2DRenderer::OnToggleRun(const uint& sourceStateIdx, const uint& targetStateIdx, const VirtualKeyMap& keyMap)
    {
        m_isRunning = !m_isRunning;
        Log::Warning(m_isRunning ? "Running" : "Paused");

        m_uiGraph.SetState("kIdleState");
        return kUIStateOkay;
    }

    __host__ uint GI2DRenderer::OnIdleState(const uint& sourceStateIdx, const uint& targetStateIdx, const VirtualKeyMap& keyMap)
    {
        //Log::Success("Back home!");
        return kUIStateOkay;
    }

    __host__ uint GI2DRenderer::OnDeleteSceneObject(const uint& sourceStateIdx, const uint& targetStateIdx, const VirtualKeyMap& keyMap)
    {
        if (m_selectionCtx.numSelected == 0) { return kUIStateOkay; }

        std::lock_guard <std::mutex> lock(m_resourceMutex);

        auto& tracables = m_scene->Tracables();
        int emptyIdx = -1;
        int numDeleted = 0;
        for (int primIdx = 0; primIdx < tracables.Size(); ++primIdx)
        {
            if (tracables[primIdx]->IsSelected())
            {
                // Erase the object from the container
                m_sceneObjects->Erase(tracables[primIdx]->GetSceneObject().GetAssetID());

                ++numDeleted;
                if (emptyIdx == -1) { emptyIdx = primIdx; }
            }
            else if (emptyIdx >= 0)
            {
                tracables[emptyIdx++] = tracables[primIdx];
            }
        }

        Assert(numDeleted <= tracables.Size());
        tracables.Resize(tracables.Size() - numDeleted);
        Log::Error("Delete!");

        EnqueueObjects("OnDeleteObject", kEnqueueSelected | kEnqueueIdOnly);

        // Clear the tracables list
        m_selectedTracables.clear();
        m_selectionCtx.numSelected = 0;

        SetDirtyFlags(kDirtyObjectBounds);

        return kUIStateOkay;
    }

    __host__ uint GI2DRenderer::OnMoveSceneObject(const uint& sourceStateIdx, const uint& targetStateIdx, const VirtualKeyMap& keyMap)
    {
        const std::string stateID = m_uiGraph.GetStateID(targetStateIdx);
        if (stateID == "kMoveSceneObjectBegin")
        {
            m_onMove.dragAnchor = m_viewCtx.mousePos;
        }
        else if (stateID == "kMoveSceneObjectDragging")
        {
            // Update the selection overlay
            m_selectionCtx.selectedBBox += m_viewCtx.mousePos - m_onMove.dragAnchor;
            m_onMove.dragAnchor = m_viewCtx.mousePos;
            SetDirtyFlags(kDirtyUI);
        }

        // Notify the scene objects of the move operation  
        std::lock_guard <std::mutex> lock(m_resourceMutex);
        uint tracableDirtyFlags = 0u;
        for (auto& obj : m_selectedTracables)
        {
            Assert(obj->IsSelected());

            // If the object has moved, trigger a rebuild of the BVH
            const uint objDirty = obj->OnMove(stateID, m_viewCtx);
            SetDirtyFlags(objDirty & (kDirtyObjectBounds | kDirtyObjectBVH));
        }

        // Enqueue the list of selected tracables
        EnqueueObjects("OnUpdateObject", kEnqueueSelected);

        return kUIStateOkay;
    }

    __host__ void GI2DRenderer::DeselectAll()
    {
        std::lock_guard <std::mutex> lock(m_resourceMutex);

        for (auto obj : m_scene->Tracables())
        {
            obj->OnSelect(false);
        }

        m_selectedTracables.clear();
        m_selectionCtx.numSelected = 0;

        SetDirtyFlags(kDirtyUI);
    }

    __host__ std::string GI2DRenderer::DecideOnClickState(const uint& sourceStateIdx)
    {
        // If there are no paths selected, enter selection state. Otherwise, enter moving state.
        if (m_selectionCtx.numSelected == 0)
        {
            return "kSelectSceneObjectDragging";
        }
        else
        {
            //Assert(selection.selectedBBox.HasValidArea());
            if (Grow(m_selectionCtx.selectedBBox, m_viewCtx.dPdXY * 2.f).Contains(m_viewCtx.mousePos))
            {
                return "kMoveSceneObjectBegin";
            }
            else
            {
                // Deselect everything
                DeselectAll();
                return "kSelectSceneObjectDragging";
            }
        }
        return "kSelectSceneObjectDragging";
    }

    __host__ uint GI2DRenderer::OnSelectSceneObjects(const uint& sourceStateIdx, const uint& targetStateIdx, const VirtualKeyMap& keyMap)
    {
        const std::string stateID = m_uiGraph.GetStateID(targetStateIdx);
        if (stateID == "kSelectSceneObjectDragging")
        {
            auto& tracables = m_scene->Tracables();
            const bool wasLassoing = m_selectionCtx.isLassoing;

            if (!m_selectionCtx.isLassoing)
            {
                // Deselect all the path segments
                DeselectAll();

                m_selectionCtx.mouseBBox = BBox2f(m_viewCtx.mousePos);
                m_selectionCtx.isLassoing = true;
            }

            m_selectionCtx.mouseBBox.upper = m_viewCtx.mousePos;
            m_selectionCtx.lassoBBox = Grow(Rectify(m_selectionCtx.mouseBBox), m_viewCtx.dPdXY * 2.);
            m_selectionCtx.selectedBBox = BBox2f::MakeInvalid();
            m_selectedTracables.clear();

            std::lock_guard <std::mutex> lock(m_resourceMutex);
            if (m_scene->TracableBIH().IsConstructed())
            {
                const uint lastNumSelected = m_selectionCtx.numSelected;

                auto onIntersectPrim = [&tracables, this](const uint* primRange, const bool isInnerNode)
                {
                    // Inner nodes are tested when the bounding box envelops them completely. Hence, there's no need to do a bbox checks.
                    if (isInnerNode)
                    {
                        for (int idx = primRange[0]; idx < primRange[1]; ++idx)
                        {
                            m_selectedTracables.emplace_back(tracables[idx]);
                            tracables[idx]->OnSelect(true);
                        }
                        m_selectionCtx.numSelected += primRange[1] - primRange[0];
                    }
                    else
                    {
                        for (int idx = primRange[0]; idx < primRange[1]; ++idx)
                        {
                            const auto& bBoxWorld = tracables[idx]->GetWorldSpaceBoundingBox();
                            const bool isCaptured = bBoxWorld.Intersects(m_selectionCtx.lassoBBox);
                            if (isCaptured)
                            {
                                m_selectedTracables.emplace_back(tracables[idx]);
                                m_selectionCtx.selectedBBox = Union(m_selectionCtx.selectedBBox, bBoxWorld);
                                ++m_selectionCtx.numSelected;
                            }
                            tracables[idx]->OnSelect(isCaptured);
                        }
                    }
                };
                m_scene->TracableBIH().TestBBox(m_selectionCtx.lassoBBox, onIntersectPrim);

                // Only if the number of selected primitives has changed
                if (lastNumSelected != m_selectionCtx.numSelected)
                {
                    if (m_selectionCtx.numSelected > 0 && !wasLassoing)
                    {
                        m_selectionCtx.isLassoing = false;
                        m_uiGraph.SetState("kMoveSceneObjectBegin");
                    }
                }
            }

            SetDirtyFlags(kDirtyUI);
            //Log::Success("Selecting!");
        }
        else if (stateID == "kSelectSceneObjectEnd")
        {
            m_selectionCtx.isLassoing = false;
            SetDirtyFlags(kDirtyUI);

            //Log::Success("Finished!");
        }
        else if (stateID == "kDeselectSceneObject")
        {
            DeselectAll();
            SetDirtyFlags(kDirtyUI);

            //Log::Success("Finished!");
        }
        else
        {
            return kUIStateError;
        }

        return kUIStateOkay;
    }

    __host__ uint GI2DRenderer::OnCreateSceneObject(const uint& sourceStateIdx, const uint& targetStateIdx, const VirtualKeyMap& trigger)
    {
        std::lock_guard <std::mutex> lock(m_resourceMutex);

        const std::string stateID = m_uiGraph.GetStateID(targetStateIdx);
        if (stateID == "kCreateSceneObjectOpen")
        {
            // Try and instante the objerct
            auto newObject = m_sceneObjectFactory.InstantiateFromHash(trigger.HashOf(), m_sceneObjects);
            m_onCreate.newObject = newObject.DynamicCast<Host::SceneObject>();
            Assert(m_onCreate.newObject);
        }

        // Invoke the event handler of the new object
        SetDirtyFlags(m_onCreate.newObject->OnCreate(stateID, m_viewCtx));

        // Some objects will automatically finalise themselves. If this happens, we're done.
        if (m_onCreate.newObject->IsFinalised())
        {
            EnqueueObjects("OnCreateObject", kEnqueueOne, m_onCreate.newObject);
            m_uiGraph.SetState("kIdleState");
            return kUIStateOkay;
        }

        if (stateID == "kCreateSceneObjectClose")
        {
            Assert(m_onCreate.newObject);

            // If the new object can't be finalised, delete it
            if (!m_onCreate.newObject->Finalise())
            {
                m_sceneObjects->Erase(m_onCreate.newObject->GetSceneObject().GetAssetID());
                SetDirtyFlags(kDirtyObjectBounds);

                Log::Success("Destroyed unfinalised tracable '%s'", m_onCreate.newObject->GetSceneObject().GetAssetID());
            }

            // Serialise the new object to the outbound queue
            EnqueueObjects("OnCreateObject", kEnqueueOne, m_onCreate.newObject);

            return kUIStateOkay;
        }

        return kUIStateOkay;
    }

    __host__ void GI2DRenderer::OnCommandsWaiting(CommandQueue& inbound)
    {
        m_commandManager.Flush(inbound, true);
    }

    __host__ void GI2DRenderer::OnRender()
    {
        if (m_dirtyFlags)
        {
            Rebuild();
        }

        //m_scene->voxelProxy->Render();

        // Render the pass
        //m_pathTracerLayer->Render();
        if (m_isRunning)
        {
            //if (m_renderTimer.Get() > 0.1f)
            {
                m_voxelProxyGridLayer->Render();
                //m_renderTimer.Reset();
                //Log::Write("-----");
            }
        }
        //m_isosurfaceExplorer->Render();
        m_overlayRenderer->Render();

        // If a blit is in progress, skip the composite step entirely.
        // TODO: Make this respond intelligently to frame rate. If the CUDA renderer is running at a lower FPS than the D3D renderer then it should wait rather than
        // than skipping frames like this.
        //m_renderSemaphore.Wait(kRenderManagerD3DBlitFinished, kRenderManagerCompInProgress);
        if (!m_renderSemaphore.Try(kRenderManagerD3DBlitFinished, kRenderManagerCompInProgress, false)) { return; }

        //m_compositeImage->Clear(vec4(kZero, 1.0f));
        //m_pathTracerLayer->Composite(m_compositeImage);
        m_voxelProxyGridLayer->Composite(m_compositeImage);
        //m_isosurfaceExplorer->Composite(m_compositeImage);    
        m_overlayRenderer->Composite(m_compositeImage);

        m_renderSemaphore.Try(kRenderManagerCompInProgress, kRenderManagerCompFinished, true);
    }

    __host__ void GI2DRenderer::OnKey(const uint code, const bool isSysKey, const bool isDown)
    {

    }

    __host__ void GI2DRenderer::OnMouseButton(const uint code, const bool isDown)
    {
        // Is the view being changed? 
        if (code == VK_MBUTTON)
        {
            m_viewCtx.dragAnchor = vec2(m_mouse.pos);
            m_viewCtx.rotAxis = normalize(m_viewCtx.dragAnchor - vec2(m_clientWidth, m_clientHeight) * 0.5f);
            m_viewCtx.transAnchor = m_viewCtx.transform.trans;
            m_viewCtx.scaleAnchor = m_viewCtx.transform.scale;
            m_viewCtx.rotAnchor = m_viewCtx.transform.rotate;
        }
    }

    __host__ void GI2DRenderer::OnMouseMove()
    {
        // Dragging?
        if (IsMouseButtonDown(VK_MBUTTON))
        {
            OnViewChange();
        }

        {
            std::lock_guard <std::mutex> lock(m_resourceMutex);
            m_viewCtx.mousePos = m_viewCtx.transform.matrix * vec2(m_mouse.pos);
        }
    }

    __host__ void GI2DRenderer::OnViewChange()
    {
        auto& transform = m_viewCtx.transform;

        // Zooming?
        if (IsKeyDown(VK_CONTROL))
        {
            float logScaleAnchor = std::log2(std::max(1e-10f, m_viewCtx.scaleAnchor));
            logScaleAnchor += m_viewCtx.zoomSpeed * float(m_mouse.pos.y - m_viewCtx.dragAnchor.y) / m_clientHeight;
            transform.scale = std::pow(2.0, logScaleAnchor);

            //Log::Write("Scale: %f", transform.scale);
        }
        // Rotating?
        else if (IsKeyDown(VK_SHIFT))
        {
            const vec2 delta = normalize(vec2(m_mouse.pos) - vec2(m_clientWidth, m_clientHeight) * 0.5f);
            const float theta = std::acos(dot(delta, m_viewCtx.rotAxis)) * (float(dot(delta, vec2(m_viewCtx.rotAxis.y, -m_viewCtx.rotAxis.x)) < 0.0f) * 2.0 - 1.0f);
            transform.rotate = m_viewCtx.rotAnchor + theta;

            if (std::abs(std::fmod(transform.rotate, kHalfPi)) < 0.05f) { transform.rotate = std::round(transform.rotate / kHalfPi) * kHalfPi; }

            //Log::Write("Theta: %f", transform.rotate);
        }
        // Translating
        else
        {
            // Update the transformation
            const mat3 newMat = ConstructViewMatrix(m_viewCtx.transAnchor, transform.rotate, transform.scale) * m_clientToNormMatrix;
            const vec2 dragDelta = (newMat * vec2(m_viewCtx.dragAnchor)) - (newMat * vec2(m_mouse.pos));
            transform.trans = m_viewCtx.transAnchor + dragDelta;

            //Log::Write("Trans: %s", m_viewCtx.trans.format());
        }

        // Update the parameters in the overlay renderer
        {
            std::lock_guard <std::mutex> lock(m_resourceMutex);
            transform.matrix = ConstructViewMatrix(transform.trans, transform.rotate, transform.scale) * m_clientToNormMatrix;
            m_viewCtx.Prepare();

            // Mark the scene as dirty
            SetDirtyFlags(kDirtyView);
        }
    }

    __host__ void GI2DRenderer::OnMouseWheel()
    {

    }

    __host__ void GI2DRenderer::OnResizeClient()
    {
    } 

    __host__ bool GI2DRenderer::Serialise(Json::Document& json, const int flags)
    {
        return true;
    }
}