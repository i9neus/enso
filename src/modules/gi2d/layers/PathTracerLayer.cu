#include "hip/hip_runtime.h"
#include "PathTracerLayer.cuh"
#include "core/math/ColourUtils.cuh"
#include "core/Hash.h"

#include "../RenderCtx.cuh"
#include "../SceneDescription.cuh"
#include "../integrators/VoxelProxyGrid.cuh"

namespace Enso
{        
    __host__ __device__ PathTracerLayerParams::PathTracerLayerParams()
    {
        accum.downsample = 1;
    }

    __device__ void Device::PathTracerLayer::OnSynchronise(const int syncFlags)
    {
        m_frameIdx = 0;      
    }

    __device__ void Device::PathTracerLayer::Accumulate(const vec4& L, const RenderCtx& ctx)
    {
        m_objects.accumBuffer->At(kKernelPos<ivec2>()) += L;
    }

    __device__ bool Device::PathTracerLayer::CreateRay(Ray2D& ray, HitCtx2D& hit, RenderCtx& renderCtx) const
    {
        // Transform from screen space to view space
        ray.o = UILayer::m_params.viewCtx.transform.matrix * vec2(kKernelPos<ivec2>() * m_params.accum.downsample);
        if (!UILayer::m_params.viewCtx.sceneBounds.Contains(ray.o)) { return false; }

        // Randomly scatter
        const float theta = renderCtx.rng.Rand<0>() * kTwoPi;
        ray.d = vec2(cosf(theta), sinf(theta));

        return true;
    }

    __device__ void Device::PathTracerLayer::Render()
    {        
        return;
        
        const ivec2 xyScreen = kKernelPos<ivec2>();
        if (xyScreen.x < 0 || xyScreen.x >= m_objects.accumBuffer->Width() || xyScreen.y < 0 || xyScreen.y >= m_objects.accumBuffer->Height()) { return; }

        RenderCtx renderCtx(kKernelY * kKernelWidth + kKernelX, uint(m_frameIdx), 0, *this);

        m_overlayTracer.Integrate(renderCtx);       

    }
    DEFINE_KERNEL_PASSTHROUGH(Render);

    __device__ void Device::PathTracerLayer::Prepare(const uint dirtyFlags)
    {
        m_frameIdx++;

        // Save ourselves a deference here by caching the scene pointers
        assert(m_objects.scenePtr);
        m_scene = *m_objects.scenePtr;
    }
    DEFINE_KERNEL_PASSTHROUGH_ARGS(Prepare);

    __device__ void Device::PathTracerLayer::Composite(Device::ImageRGBA* deviceOutputImage)
    {        
        assert(deviceOutputImage);

        const ivec2 xyScreen = kKernelPos<ivec2>();
        if (xyScreen.x < 0 || xyScreen.x >= deviceOutputImage->Width() || xyScreen.y < 0 || xyScreen.y >= deviceOutputImage->Height()) { return; }

        // Transform from screen space to view space
        const vec2 xyView = UILayer::m_params.viewCtx.transform.matrix * vec2(xyScreen);

        if (!UILayer::m_params.viewCtx.sceneBounds.Contains(xyView))
        { 
            deviceOutputImage->At(xyScreen) = vec4(0.1f, 0.1f, 0.1f, 1.0f);
            return; 
        }

        vec4 L(0.0f);

        const vec2 uv = vec2(xyScreen) * vec2(m_objects.accumBuffer->Dimensions()) / vec2(deviceOutputImage->Dimensions());
        L = m_objects.accumBuffer->Lerp(uv);
        L.xyz /= fmaxf(L.w, 1.0f);

        //L.xyz += m_scene.voxelProxy->Evaluate(xyView);

        deviceOutputImage->At(xyScreen) = vec4(L.xyz, 1.0f);
    }
    DEFINE_KERNEL_PASSTHROUGH_ARGS(Composite);

    Host::PathTracerLayer::PathTracerLayer(const std::string& id, const AssetHandle<Host::SceneDescription>& scene, const uint width, const uint height, const uint downsample, hipStream_t renderStream) :
        UILayer(id, scene)
    {
        // Create some Cuda objects
        m_hostAccumBuffer = CreateChildAsset<Host::ImageRGBW>("id_2dgiAccumBuffer", width / downsample, height / downsample, renderStream);

        m_deviceObjects.scenePtr = scene->GetDeviceInstance(); 
        m_deviceObjects.accumBuffer = m_hostAccumBuffer->GetDeviceInstance();

        m_params.accum.downsample = downsample;

        cu_deviceData = InstantiateOnDevice<Device::PathTracerLayer>();

        Synchronise(kSyncObjects);
    }

    Host::PathTracerLayer::~PathTracerLayer()
    {
        OnDestroyAsset();
    }

    __host__ void Host::PathTracerLayer::Rebuild(const uint dirtyFlags, const UIViewCtx& viewCtx, const UISelectionCtx& selectionCtx)
    {
        m_dirtyFlags = dirtyFlags;
        
        UILayer::Rebuild(dirtyFlags, viewCtx, selectionCtx);

        Synchronise(kSyncParams);
    }

    __host__ void Host::PathTracerLayer::Synchronise(const int syncType)
    {
        UILayer::Synchronise(cu_deviceData, syncType);

        if (syncType & kSyncObjects) { SynchroniseObjects<Device::PathTracerLayer>(cu_deviceData, m_deviceObjects); }
        if (syncType & kSyncParams) { SynchroniseObjects<Device::PathTracerLayer>(cu_deviceData, m_params); }
    }

    __host__ void Host::PathTracerLayer::OnDestroyAsset()
    {
        DestroyOnDevice(cu_deviceData);
        m_hostAccumBuffer.DestroyAsset();
    }

    __host__ void Host::PathTracerLayer::Render()
    {
        // Advance the frame counter
        KernelPrepare << <1, 1 >> > (cu_deviceData, m_dirtyFlags);
        
        if (m_dirtyFlags & (kDirtyMaterials | kDirtyObjectBounds | kDirtyObjectBVH))
        {
            m_hostAccumBuffer->Clear(vec4(0.0f));
        }        
        m_dirtyFlags = 0;

        dim3 blockSize, gridSize;
        KernelParamsFromImage(m_hostAccumBuffer, blockSize, gridSize);

        // Render the frame
        KernelRender << < gridSize, blockSize, 0 >> > (cu_deviceData);
        IsOk(hipDeviceSynchronize());
    }

    __host__ void Host::PathTracerLayer::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage) const
    {
        dim3 blockSize, gridSize;
        KernelParamsFromImage(hostOutputImage, blockSize, gridSize);

        KernelComposite << < gridSize, blockSize, 0 >> > (cu_deviceData, hostOutputImage->GetDeviceInstance());
        IsOk(hipDeviceSynchronize());
    }
}