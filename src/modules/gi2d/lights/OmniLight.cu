#include "hip/hip_runtime.h"
#include "OmniLight.cuh"

#include "../primitives/Ellipse.cuh"
#include "../primitives/GenericIntersector.cuh"
#include "io/json/JsonUtils.h"

namespace Enso
{
    __host__ __device__ vec4 Device::OmniLight::EvaluateOverlay(const vec2& pWorld, const UIViewCtx& viewCtx, const bool isMouseTest) const
    {
        if (!GetWorldBBox().Contains(pWorld)) { return vec4(0.f); }

        return m_primitive.EvaluateOverlay(pWorld - GetTransform().trans, OverlayCtx::MakeStroke(viewCtx, vec4(1.), 3.f));
    }

    __host__ __device__ uint Device::OmniLight::OnMouseClick(const UIViewCtx& viewCtx) const
    {       
        return (m_primitive.Contains(viewCtx.mousePos - GetTransform().trans, viewCtx.dPdXY) > 0.0f) ? kSceneObjectPrecisionDrag : kSceneObjectInvalidSelect;
    }

    __host__ __device__ bool Device::OmniLight::IntersectRay(const Ray2D& rayWorld, HitCtx2D& hitWorld) const
    {
        RayRange2D range;
        if (!IntersectRayBBox(rayWorld, GetWorldBBox(), range) || range.tNear > hitWorld.tFar) { return false; }

        // TODO: Untransform normal
        return m_primitive.IntersectRay(RayBasic2D(rayWorld.o - GetTransform().trans, rayWorld.d), hitWorld);
    }

    __device__ bool Device::OmniLight::Sample(const Ray2D& parentRay, const HitCtx2D& hit, float xi, vec2& extant, vec3& L, float& pdf) const
    {
        const vec2 lightLocal = GetTransform().trans - hit.p;
        extant = normalize(vec2(lightLocal.y, -lightLocal.x)) * m_params.lightRadius * (xi - 0.5f) + lightLocal;
        float lightDist = length(extant);
        extant /= lightDist;

        float cosTheta = (hit.flags & kHit2DIsVolume) ? (1.0 / kTwoPi) : dot(extant, hit.n);
        if (cosTheta <= 0.0) { return false; }

        const float lightSolidAngle = 2.0f * ((lightDist >= m_params.lightRadius) ? asin(m_params.lightRadius / lightDist) : kHalfPi);

        L = m_params.lightColour * powf(2.0f, m_params.lightIntensity) * lightSolidAngle * cosTheta / (2.0 * m_params.lightRadius);
        pdf = 1.0 / lightSolidAngle;

        return true;
    }

    __device__ bool Device::OmniLight::Evaluate(const Ray2D& parentRay, const HitCtx2D& hit, vec3& L, float& pdfLight) const
    {

    }

    __device__ float Device::OmniLight::Estimate(const Ray2D& parentRay, const HitCtx2D& hit) const
    {
        return length(GetTransform().trans - hit.p) * powf(2.0f, m_params.lightIntensity);
    }

    __device__ void Device::OmniLight::OnSynchronise(const int syncFlags)
    {
        if (syncFlags == kSyncParams)
        {
            m_primitive = Ellipse(vec2(0.f), m_params.lightRadius);
        }
    }

    __host__ AssetHandle<Host::GenericObject> Host::OmniLight::Instantiate(const std::string& id, const Json::Node&, const AssetHandle<const Host::SceneDescription>&)
    {
        return CreateAsset<Host::OmniLight>(id);
    }

    __host__ Host::OmniLight::OmniLight(const std::string& id) :
        Host::Light(id, m_hostInstance),
        cu_deviceInstance(m_allocator.InstantiateOnDevice<Device::OmniLight>())
    {
        Light::SetDeviceInstance(m_allocator.StaticCastOnDevice<Device::Light>(cu_deviceInstance));

        Synchronise(kSyncObjects);
    }

    __host__ Host::OmniLight::~OmniLight()
    {
        BEGIN_EXCEPTION_FENCE

            OnDestroyAsset();

        END_EXCEPTION_FENCE
    }

    __host__ void Host::OmniLight::OnDestroyAsset()
    {
        m_allocator.DestroyOnDevice(cu_deviceInstance);
    }

    __host__ void Host::OmniLight::Synchronise(const uint syncFlags)
    {
        Light::Synchronise(syncFlags);

        if (syncFlags & kSyncParams) 
        { 
            SynchroniseObjects<>(cu_deviceInstance, m_hostInstance.m_params); 
            m_hostInstance.OnSynchronise(syncFlags);
        }
    }

    /*__host__ uint Host::OmniLight::OnMove(const std::string& stateID, const UIViewCtx& viewCtx)
    {
        if (stateID != "kMoveSceneObjectDragging") { return 0; }

        GetTransform().trans = viewCtx.mousePos;
        SceneObject::m_params.worldBBox = BBox2f(GetTransform().trans - vec2(m_params.lightRadius), GetTransform().trans + vec2(m_params.lightRadius));

        SetDirtyFlags(kDirtyObjectBounds);
        return m_dirtyFlags;
    }*/

    __host__ uint Host::OmniLight::OnCreate(const std::string& stateID, const UIViewCtx& viewCtx)
    {
        //AssertInThread("kMainThread");

        Log::Warning(stateID);

        if (stateID == "kCreateSceneObjectOpen")
        {
            // Set the origin of the 
            m_onCreate.isCentroidSet = false;
            m_isConstructed = true;
            m_hostInstance.GetTransform().trans = viewCtx.mousePos;
            m_hostInstance.m_params.lightRadius = viewCtx.dPdXY;
        }
        else if (stateID == "kCreateSceneObjectHover")
        {
            if (m_onCreate.isCentroidSet)
            {
                m_hostInstance.m_params.lightRadius = length(m_hostInstance.GetTransform().trans - viewCtx.mousePos);
            }
            else
            {
                m_hostInstance.GetTransform().trans = viewCtx.mousePos;
            }
        }
        else if (stateID == "kCreateSceneObjectAppend")
        {
            if (!m_onCreate.isCentroidSet)
            {
                m_hostInstance.GetTransform().trans = viewCtx.mousePos;
                m_hostInstance.m_params.lightRadius = viewCtx.dPdXY;
                m_onCreate.isCentroidSet = true;
            }
            else
            {
                m_isFinalised = true;
            }
        }
        else
        {
            return m_dirtyFlags;
        }

        // If the object is dirty, recompute the bounding box
        SetDirtyFlags(kDirtyObjectBounds);
        return m_dirtyFlags;
    }

    __host__ bool Host::OmniLight::Rebuild(const uint parentFlags, const UIViewCtx& viewCtx)
    {
        //AssertInThread("kRenderThread");

        if (!m_dirtyFlags) { return IsConstructed(); }
        
        if (m_dirtyFlags & kDirtyObjectBounds)
        {
            RecomputeBoundingBoxes();
        }

        Synchronise(kSyncParams);
        ClearDirtyFlags();
        return IsConstructed();
    }

    __host__ bool Host::OmniLight::Serialise(Json::Node& node, const int flags) const
    {
        Tracable::Serialise(node, flags);

        node.AddValue("radius", m_hostInstance.m_params.lightRadius);
        node.AddVector("colour", m_hostInstance.m_params.lightColour);
        node.AddValue("intensity", m_hostInstance.m_params.lightIntensity);
        return true;
    }

    __host__ uint Host::OmniLight::Deserialise(const Json::Node& node, const int flags)
    {
        Tracable::Deserialise(node, flags);

        if (node.GetValue("radius", m_hostInstance.m_params.lightRadius, flags)) { SetDirtyFlags(kDirtyObjectBounds); }
        if (node.GetVector("colour", m_hostInstance.m_params.lightColour, flags)) { SetDirtyFlags(kDirtyMaterials); }
        if (node.GetValue("intensity", m_hostInstance.m_params.lightIntensity, flags)) { SetDirtyFlags(kDirtyMaterials); }

        return m_dirtyFlags;
    }

    __host__ uint Host::OmniLight::OnMouseClick(const UIViewCtx& viewCtx) const
    {
        return m_hostInstance.OnMouseClick(viewCtx);
    }

    __host__ BBox2f Host::OmniLight::RecomputeObjectSpaceBoundingBox()
    {
        return BBox2f(-vec2(m_hostInstance.m_params.lightRadius), vec2(m_hostInstance.m_params.lightRadius));
    }
}
