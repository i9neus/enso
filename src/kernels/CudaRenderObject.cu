﻿#include "CudaRenderObject.cuh"
#include "generic/JsonUtils.h"
#include "generic/FilesystemUtils.h"

namespace Cuda
{
    __host__ __device__ RenderObjectParams::RenderObjectParams() :
        flags(0, 2) {}

    __host__ void RenderObjectParams::ToJson(::Json::Node& node) const
    {
        flags.ToJson("objectFlags", node);
    }

    __host__ void RenderObjectParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        this->flags.FromJson("objectFlags", node, flags);
    }

    __host__ void RenderObjectParams::Randomise(const vec2& range)
    {
        flags.Update(kJitterRandomise);
    }
    
    __host__ void Host::RenderObject::UpdateDAGPath(const ::Json::Node& node)
    {
        if (!node.HasDAGPath())
        {
            Log::Error("Internal error: JSON node for '%s' has no DAG path.\n", GetAssetID());
            return;
        }

        SetDAGPath(node.GetDAGPath());
    }

    __host__ void RenderObjectContainer::Finalise() const
    {
        Log::Debug("Finalising...\n");
        Log::Indent indent;

        Log::Debug("DAG map:\n");
        {
            Log::Indent indent;
            for (auto& object : m_dagMap)
            {
                Log::Debug("%s\n", object.first);
            }
        }
    }

    __host__ void RenderObjectContainer::Emplace(AssetHandle<Host::RenderObject>& newObject)
    {
        AssertMsgFmt(!Exists(newObject->GetAssetID()), "A render object with ID '%s' already exists in the object container.\n", newObject->GetAssetID().c_str());

        m_objectMap[newObject->GetAssetID()] = newObject;

        if (newObject->HasDAGPath())
        {
            if (m_dagMap.find(newObject->GetDAGPath()) == m_dagMap.end())
            {
                m_dagMap[newObject->GetDAGPath()] = newObject;
            }
            else
            {
                Log::Error("Internal error: object '%s' has the same DAG path (%s) as another object.\n", newObject->GetAssetID(), newObject->GetDAGPath());
            }
        }
        else if(!newObject->IsChildObject())
        {
            Log::Error("Error: instantiated object '%s' does not have a valid DAG path. (Did you forget to call UpdateDAGPath() during FromJson()?)\n", newObject->GetAssetID());
            return;
        }
    }

    __host__ void RenderObjectContainer::Bind()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Bind(*this);
        }
    }

    __host__ void RenderObjectContainer::Synchronise()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Synchronise();
        }
    }

    __host__ void RenderObjectContainer::OnDestroyAsset()
    {
        constexpr int kMaxAttempts = 10;
        for (int i = 0; !m_objectMap.empty() && i < kMaxAttempts; i++)
        {
            for (RenderObjectMap::iterator it = m_objectMap.begin(); it != m_objectMap.end();)
            {
                uint flags = 0;
                if (i == kMaxAttempts - 1)
                {
                    flags |= kAssetForceDestroy | kAssetAssertOnError;
                }

                // Try to delete the asset
                if (!it->second.DestroyAsset(flags))
                {
                    ++it;
                }
                else
                {
                    auto nextIt = std::next(it);
                    m_objectMap.erase(it);
                    it = nextIt;
                }
            }
        }
    }
}