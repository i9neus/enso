#include "GI2DIsosurfaceExplorer.cuh"
#include "kernels/math/CudaColourUtils.cuh"
#include "kernels/math/Complex.cuh"
#include "generic/Hash.h"

#include "../RenderCtx.cuh"

using namespace Cuda;

namespace GI2D
{
    __host__ __device__ IsosurfaceExplorerParams::IsosurfaceExplorerParams()
    {
        m_accum.width = 0;
        m_accum.height = 0;
        m_accum.downsample = 1;
        m_frameIdx = 0;
    }

    __device__ Device::IsosurfaceExplorer::IsosurfaceExplorer() { }

    __device__ vec2 Mul(const vec2& a, const vec2& b)
    {
        return vec2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
    }

    __device__ void Device::IsosurfaceExplorer::Render()
    {
        const ivec2 xyScreen = kKernelPos<ivec2>();
        if (xyScreen.x < 0 || xyScreen.x >= m_accumBuffer->Width() || xyScreen.y < 0 || xyScreen.y >= m_accumBuffer->Height()) { return; }

        // Transform from screen space to view space
        const vec2 xyView = m_viewCtx.transform.matrix * vec2(xyScreen * m_accum.downsample);

        //if (!m_viewCtx.sceneBounds.Contains(xyView)) { m_accumBuffer->At(xyScreen) = vec4(0.0f, 0.0f, 0.0f, 1.0f); return; }

        Complex z = xyView;
        Complex c;
        if (m_inspectors && m_inspectors->Size() > 0)
        {
            c = (*m_inspectors)[0]->GetWorldSpaceBoundingBox().Centroid();
        }        

        bool inSet = true;
        for(int iterIdx = 0; iterIdx < 10; ++iterIdx)
        {
            if (length2(z) > 4.f) 
            { 
                m_accumBuffer->At(xyScreen) = vec4(kRed, 1.0f);
                return;
            }

            z = z * z + c;
        }    

        m_accumBuffer->At(xyScreen) = vec4(kGreen, 1.0f);
    }
    DEFINE_KERNEL_PASSTHROUGH(Render);

    __device__ void Device::IsosurfaceExplorer::Composite(Cuda::Device::ImageRGBA* deviceOutputImage)
    {
        assert(deviceOutputImage);

        // TODO: Make alpha compositing a generic operation inside the Image class.
        const ivec2 xyScreen = kKernelPos<ivec2>();
        if (xyScreen.x >= 0 && xyScreen.x < m_accumBuffer->Width() && xyScreen.y >= 0 && xyScreen.y < m_accumBuffer->Height())
        {
            deviceOutputImage->Blend(xyScreen, m_accumBuffer->At(xyScreen));
            //vec4& target = deviceOutputImage->At(xyScreen);
            //target = Blend(target, m_accumBuffer->At(xyScreen));
            //target.xyz += m_accumBuffer->At(xyScreen).xyz;
        }
    }
    DEFINE_KERNEL_PASSTHROUGH_ARGS(Composite);

    Host::IsosurfaceExplorer::IsosurfaceExplorer(const std::string& id, AssetHandle<Host::BIH2DAsset>& bih, AssetHandle<TracableContainer>& tracables, 
                                                 AssetHandle<InspectorContainer>& inspectors, const uint width, const uint height, const uint downsample, hipStream_t renderStream) :
        UILayer(id, bih, tracables),
        m_hostInspectors(inspectors)
    {
        // Create some Cuda objects
        m_hostAccumBuffer = CreateChildAsset<Cuda::Host::ImageRGBW>("accumBuffer", width / downsample, height / downsample, renderStream);

        m_deviceObjects.m_inspectors = m_hostInspectors->GetDeviceInstance();
        m_deviceObjects.m_accumBuffer = m_hostAccumBuffer->GetDeviceInstance();

        m_accum.width = width;
        m_accum.height = height;
        m_accum.downsample = downsample;

        cu_deviceData = InstantiateOnDevice<Device::IsosurfaceExplorer>();

        Synchronise(kSyncObjects);
    }

    Host::IsosurfaceExplorer::~IsosurfaceExplorer()
    {
        OnDestroyAsset();
    }


    __host__ void Host::IsosurfaceExplorer::Rebuild(const uint dirtyFlags, const UIViewCtx& viewCtx, const UISelectionCtx& selectionCtx)
    {
        UILayer::Rebuild(dirtyFlags, viewCtx, selectionCtx);

        Synchronise(kSyncParams);
    }

    __host__ void Host::IsosurfaceExplorer::Synchronise(const int syncType)
    {
        UILayer::Synchronise(cu_deviceData, syncType);

        if (syncType & kSyncObjects) { SynchroniseObjects2<IsosurfaceExplorerObjects>(cu_deviceData, m_deviceObjects); }
        if (syncType & kSyncParams) { SynchroniseObjects2<IsosurfaceExplorerParams>(cu_deviceData, *this); }
    }

    __host__ void Host::IsosurfaceExplorer::OnDestroyAsset()
    {
        DestroyOnDevice(cu_deviceData);
        m_hostAccumBuffer.DestroyAsset();
    }

    __host__ void Host::IsosurfaceExplorer::Render()
    {
        if (m_dirtyFlags)
        {
            m_hostAccumBuffer->Clear(vec4(0.0f));
            m_dirtyFlags = 0;
        }

        dim3 blockSize, gridSize;
        KernelParamsFromImage(m_hostAccumBuffer, blockSize, gridSize);

        KernelRender << < gridSize, blockSize, 0 >> > (cu_deviceData);
        IsOk(hipDeviceSynchronize());
    }

    __host__ void Host::IsosurfaceExplorer::Composite(AssetHandle<Cuda::Host::ImageRGBA>& hostOutputImage) const
    {
        dim3 blockSize, gridSize;
        KernelParamsFromImage(hostOutputImage, blockSize, gridSize);

        KernelComposite << < gridSize, blockSize, 0 >> > (cu_deviceData, hostOutputImage->GetDeviceInstance());
        IsOk(hipDeviceSynchronize());
    }
}