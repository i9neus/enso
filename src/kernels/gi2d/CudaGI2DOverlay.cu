#include "hip/hip_runtime.h"
#include "CudaGI2DOverlay.cuh"
#include "kernels/math/CudaColourUtils.cuh"

namespace Cuda
{
    __host__ __device__ GI2DOverlayParams::GI2DOverlayParams()
    {
        majorLineSpacing = 1.0f;
        viewScale = 1.0f;
        lineAlpha = 0.0f;
    }
    
    __device__ Device::GI2DOverlay::GI2DOverlay(const GI2DOverlayParams& params) :
        m_params(params)
    {

    }

    __device__ void Device::GI2DOverlay::Synchronise(const GI2DOverlayParams& params)
    {
        m_params = params;
    }

    __device__ void Device::GI2DOverlay::Render(Device::ImageRGBA* deviceOutputImage)
    {
        assert(deviceOutputImage);

        const ivec2 xyScreen = kKernelPos<ivec2>();
        if (xyScreen.x < 0 || xyScreen.x >= deviceOutputImage->Width() || xyScreen.y < 0 || xyScreen.y >= deviceOutputImage->Height()) { return; }

        // Transform from screen space to view space
        const vec2 xyView = (m_params.viewMatrix * vec3(vec2(xyScreen), 1.0f)).xy;

        vec3 L = vec3(0.1);
        vec2 xyGrid = fract(xyView / vec2(m_params.majorLineSpacing)) * sign(xyView);
        if (cwiseMin(xyGrid) < 0.02f * mix(1.0, 0.1, m_params.lineAlpha)) { L = kOne * 0.3f; }

        xyGrid = fract(xyView / vec2(m_params.minorLineSpacing)) * sign(xyView);
        if (cwiseMin(xyGrid) < 0.02f) { L = max(L, kOne * 0.3f * m_params.lineAlpha); }

        deviceOutputImage->At(xyScreen) = vec4(L, 1.0f);
    }

    DEFINE_KERNEL_PASSTHROUGH_ARGS(Render);

    Host::GI2DOverlay::GI2DOverlay(const std::string& id) :
        Asset(id)
    {
        cu_deviceData = InstantiateOnDevice<Device::GI2DOverlay>(GetAssetID(), m_params);
    }

    Host::GI2DOverlay::~GI2DOverlay()
    {
        OnDestroyAsset();
    }

    __host__ void Host::GI2DOverlay::OnDestroyAsset()
    {
        DestroyOnDevice(GetAssetID(), cu_deviceData);
    }

    __host__ void Host::GI2DOverlay::Render(AssetHandle<Host::ImageRGBA>& hostOutputImage)
    {
        const auto& meta = hostOutputImage->GetMetadata();
        dim3 blockSize(16, 16, 1);
        dim3 gridSize((meta.Width() + 15) / 16, (meta.Height() + 15) / 16, 1);    

        KernelRender << < gridSize, blockSize, 0, m_hostStream >> > (cu_deviceData, hostOutputImage->GetDeviceInstance());
        IsOk(hipDeviceSynchronize());
    }

    __host__ void Host::GI2DOverlay::SetParams(const GI2DOverlayParams& newParams)
    {
        m_params = newParams;

        //Log::Warning("%s", vec2(m_params.viewMatrix.i02, m_params.viewMatrix.i12).format());

        const float logScale = std::log10(m_params.viewScale);
        constexpr float kGridScale = 0.05f;

        m_params.majorLineSpacing = kGridScale * std::pow(10.0f, std::ceil(logScale));
        m_params.minorLineSpacing = kGridScale * std::pow(10.0f, std::floor(logScale));
        m_params.lineAlpha = 1 - (logScale - std::floor(logScale));

        SynchroniseObjects(cu_deviceData, m_params);
    }
}