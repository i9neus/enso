﻿#include "CudaRenderObjectContainer.cuh"

namespace Cuda
{
    __host__ void RenderObjectContainer::Finalise() const
    {
        Log::Debug("Finalising...\n");
        Log::Indent indent;

        Log::Debug("DAG map:\n");
        {
            Log::Indent indent;
            for (auto& object : m_dagMap)
            {
                Log::Debug("%s\n", object.first);
            }
        }
    }

    __host__ void RenderObjectContainer::Emplace(AssetHandle<Host::RenderObject>& newObject)
    {
        AssertMsgFmt(!Exists(newObject->GetAssetID()), "A render object with ID '%s' already exists in the object container.\n", newObject->GetAssetID().c_str());

        m_objectMap[newObject->GetAssetID()] = newObject;

        if (newObject->HasDAGPath())
        {
            if (m_dagMap.find(newObject->GetDAGPath()) == m_dagMap.end())
            {
                m_dagMap[newObject->GetDAGPath()] = newObject.GetWeakHandle();
            }
            else
            {
                Log::Error("Internal error: object '%s' has the same DAG path (%s) as another object.\n", newObject->GetAssetID(), newObject->GetDAGPath());
            }
        }
        else if (!newObject->IsChildObject())
        {
            Log::Error("Error: instantiated object '%s' does not have a valid DAG path. (Did you forget to call UpdateDAGPath() during FromJson()?)\n", newObject->GetAssetID());
            return;
        }
    }

    __host__ void RenderObjectContainer::Bind()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Bind(*this);
        }
    }

    __host__ void RenderObjectContainer::Synchronise()
    {
        for (auto& object : m_objectMap)
        {
            object.second->Synchronise();
        }
    }

    __host__ void RenderObjectContainer::OnDestroyAsset()
    {
        Log::Debug("Unloading scene graph...");
        
        constexpr int kMaxAttempts = 10;
        std::vector<std::string> activeList;
        for (int i = 0; !m_objectMap.empty() && i < kMaxAttempts; i++)
        {
            //Log::Indent indent(tfm::format("Pass %i...", i + 1));
            for (RenderObjectMap::iterator it = m_objectMap.begin(); it != m_objectMap.end();)
            {
                uint flags = kAssetCleanupPass;
                if (i == kMaxAttempts - 1)
                {
                    flags |= kAssetForceDestroy | kAssetAssertOnError;

                    if (it->second.GetReferenceCount() > 1)
                    {
                        activeList.push_back(it->first);
                    }
                }

                // Try to delete the asset
                if (!it->second.DestroyAsset(flags))
                {
                    ++it;
                }
                else
                {
                    auto nextIt = std::next(it);
                    m_objectMap.erase(it);
                    it = nextIt;
                }
            }
        }

        if (activeList.size() > 0)
        {
            Log::Error("ERROR: %i objects were not properly cleaned up:", activeList.size());
            for (const auto& name : activeList)
            {
                Log::Error("  - %s", name);
            }
        }
    }
}