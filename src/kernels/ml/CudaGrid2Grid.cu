﻿#include "CudaGrid2Grid.cuh"
#include "../cameras/CudaLightProbeCamera.cuh"

#include "generic/JsonUtils.h"

#define kBlockSize 256
#define kMaxCoefficients 5

namespace Cuda
{
    __host__ __device__ Grid2GridParams::Grid2GridParams()
    {

    }

    __host__ void Grid2GridParams::ToJson(::Json::Node& node) const
    {
      
    }

    __host__ uint Grid2GridParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        return kRenderObjectDirtyRender;
    }

    __host__ Host::Grid2Grid::Grid2Grid(const std::string& id, const ::Json::Node& node) :
        RenderObject(id),
        m_gridSize(1), m_blockSize(1)
    {
        FromJson(node, Json::kSilent);

        node.GetValue("inputGridID", m_inputGridID, Json::kRequiredAssert);
        node.GetValue("outputGridID", m_outputGridID, Json::kRequiredAssert);
        node.GetValue("modelPath", m_modelPath, Json::kRequiredAssert | Json::kNotBlank);

        AssertMsgFmt(!GlobalResourceRegistry::Get().Exists(m_outputGridID), "Error: an asset with ID '%s' already exists'.", m_outputGridID.c_str());

        // Create some objects
        m_hostOutputGrid = CreateChildAsset<Host::LightProbeGrid>(m_outputGridID);

        // Initialise the grid2grid model
        m_onnxEvaluator.Initialise(m_modelPath);

        m_rawData.reserve(8 * 8 * 8 * 5);
    }

    __host__ AssetHandle<Host::RenderObject> Host::Grid2Grid::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kLightProbeFilter) { return AssetHandle<Host::RenderObject>(); }

        return CreateAsset<Host::Grid2Grid>(id, json);
    }

    __host__ uint Host::Grid2Grid::FromJson(const ::Json::Node& node, const uint flags)
    {
        m_objects->params.FromJson(node, flags);

        Prepare();

        return kRenderObjectDirtyRender;
    }

    __host__ void Host::Grid2Grid::OnDestroyAsset()
    {
        m_hostOutputGrid.DestroyAsset();
    }

    __host__ void Host::Grid2Grid::Bind(RenderObjectContainer& sceneObjects)
    {
        m_hostInputGrid = sceneObjects.FindByID(m_inputGridID).DynamicCast<Host::LightProbeGrid>();
        if (!m_hostInputGrid)
        {
            Log::Error("Error: Grid2Grid::Bind(): the specified input light probe grid '%s' is invalid.\n", m_inputGridID);
            return;
        }

        // Get the light probe camera object and listen out for rebuilds
        auto& probeCamera = sceneObjects.FindFirstOfType<Host::LightProbeCamera>();
        if (probeCamera)
        {
            probeCamera->Listen(*this, "OnBuildGrids", &Host::Grid2Grid::OnBuildInputGrids);
        }

        Prepare();
    }

    __host__ void Host::Grid2Grid::Prepare()
    {
        m_isActive = true;
        m_isValidInput = true;

        // Filter isn't yet bound, so do nothing
        if (!m_hostInputGrid || !m_hostOutputGrid) { return; }

        const auto& gridParams = m_hostInputGrid->GetParams();

        if (gridParams.gridDensity != ivec3(8, 8, 8) || gridParams.shOrder != 1)
        {
            Log::Warning("Warning: Grid2Grid requires 8x8x8 grid of order L1. Input is %s of order L%i.", gridParams.gridDensity.format(), gridParams.shOrder);
            m_isValidInput = false;
            return;
        }

        // Initialise the output grids so it has the same dimensions as the input
        m_hostOutputGrid->Prepare(gridParams);

        m_objects.Upload();
    }

    __host__ void Host::Grid2Grid::OnBuildInputGrids(const RenderObject& originObject, const std::string& eventID)
    {
        // Run the filter every time the input grids are updated
        Execute();
    }

    __host__ void Host::Grid2Grid::Execute()
    {
        // Filter isn't yet bound, so do nothing
        if (!m_hostInputGrid || !m_hostOutputGrid) { return; }

        const LightProbeGridParams& gridParams = m_hostInputGrid->GetParams();

        // Pass-through filter just copies the data
        //if (m_objects->params.filterType == kKernelFilterNull/* || !m_hostInputGrid->IsConverged()*/)
        if (!m_isValidInput)
        {
            m_hostOutputGrid->Replace(*m_hostInputGrid);           
            m_isActive = true;
            return;
        }

        // Invoke the grid2grid model
        m_hostInputGrid->GetRawData(m_rawData);
        m_onnxEvaluator.Evaluate(m_rawData, m_rawData); 
        m_hostOutputGrid->SetRawData(m_rawData);
        
        m_isActive = false;
    }

    __host__ std::vector<AssetHandle<Host::RenderObject>> Host::Grid2Grid::GetChildObjectHandles()
    {    
        return std::vector<AssetHandle<Host::RenderObject>>({ m_hostOutputGrid });
    }

    __host__ void Host::Grid2Grid::OnUpdateSceneGraph(RenderObjectContainer& sceneObjects, const uint dirtyFlags)
    {
        if (m_hostInputGrid && m_hostOutputGrid &&
            m_hostInputGrid->GetParams() != m_hostOutputGrid->GetParams())
        {
            Prepare();
        }
    }
}