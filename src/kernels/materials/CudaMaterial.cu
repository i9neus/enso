#include "hip/hip_runtime.h"
﻿#include "CudaMaterial.cuh"
#include "generic/JsonUtils.h" 

namespace Cuda
{
    __host__ void Device::SimpleMaterial::Params::ToJson(Json::Node& node) const
    {
        node.AddArray("albedo", std::vector<float>({ albedo.x, albedo.y, albedo.z }));
    }

    __host__ void Device::SimpleMaterial::Params::FromJson(const Json::Node& node)
    {
        node.GetVector("albedo", albedo, true);
    }
    
    __device__ vec3 Device::SimpleMaterial::Evaluate(const HitCtx& hit) const
    {
        constexpr float kGridScale = 5.0f;

        vec3 albedo = m_params.albedo;
        vec2 absUv = abs(hit.uv - vec2(0.5f));
        if (absUv.x < 0.52f && absUv.y < 0.52f && !(absUv.x < 0.5f && absUv.y < 0.5f)) 
        { 
            albedo *= 0.7; 
        }
        if (fract(absUv.x * kGridScale) < 0.02f || fract(absUv.y * kGridScale) < 0.02f ||
            fract(absUv.x * 10.0f * kGridScale) < 0.1f || fract(absUv.y * 10.0 * kGridScale) < 0.1f)
        {
            albedo *= 0.7;
        }

        return albedo;
    }
    
    __host__ Host::SimpleMaterial::SimpleMaterial() :
        cu_deviceData(nullptr)
    {
        cu_deviceData = InstantiateOnDevice<Device::SimpleMaterial>();
    }

    __host__ void Host::SimpleMaterial::OnDestroyAsset()
    {
        DestroyOnDevice(&cu_deviceData);
    }

    __host__ void Host::SimpleMaterial::OnJson(const Json::Node& parentNode)
    {
        Json::Node childNode = parentNode.GetChildObject("material", true);
        if (childNode)
        {
            SyncParameters(cu_deviceData, Device::SimpleMaterial::Params(childNode));
        }
    }
}
    