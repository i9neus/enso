#include "hip/hip_runtime.h"
﻿#include "CudaMaterial.cuh"

namespace Cuda
{
    __device__ vec3 Device::SimpleMaterial::Evaluate(const HitCtx& hit) const
    {
        constexpr float kGridScale = 5.0f;

        vec3 albedo = m_params.albedo;
        vec2 absUv = abs(hit.uv - vec2(0.5f));
        if (absUv.x < 0.52f && absUv.y < 0.52f && !(absUv.x < 0.5f && absUv.y < 0.5f)) 
        { 
            albedo *= 0.7; 
        }
        if (fract(absUv.x * kGridScale) < 0.02f || fract(absUv.y * kGridScale) < 0.02f ||
            fract(absUv.x * 10.0f * kGridScale) < 0.1f || fract(absUv.y * 10.0 * kGridScale) < 0.1f)
        {
            albedo *= 0.7;
        }

        return albedo;
    }
    
    __host__ Host::SimpleMaterial::SimpleMaterial() :
        cu_deviceData(nullptr)
    {
        cu_deviceData = InstantiateOnDevice<Device::SimpleMaterial>();
    }

    __host__ void Host::SimpleMaterial::OnDestroyAsset()
    {
        DestroyOnDevice(&cu_deviceData);
    }

    __host__ void Host::SimpleMaterial::OnJson(const Json::Node& jsonNode)
    {
        Device::SimpleMaterial::Params params;

        jsonNode.GetVector("albedo", params.albedo, true);

        SyncParameters(cu_deviceData, params);
    }
}
    