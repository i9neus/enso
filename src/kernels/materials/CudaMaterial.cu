﻿#include "CudaMaterial.cuh"
#include "generic/JsonUtils.h" 
#include "../bxdfs/CudaBxDF.cuh"

namespace Cuda
{
    __host__ Host::Material::Material(const std::string& id, const ::Json::Node& parentNode) :
        RenderObject(id)
    {
        Host::RenderObject::UpdateDAGPath(parentNode);
    }
    
    __host__ uint Host::Material::FromJson(const ::Json::Node& parentNode, const uint flags)
    {
        parentNode.GetValue("bxdf", m_bxdfId, flags);
    }

    __host__ void Host::Material::Bind(RenderObjectContainer& objectContainer)
    {
        // Push the binding to the device
        AssetHandle<Host::BxDF> bxdfAsset = GetAssetHandleForBinding<Host::Material, Host::BxDF>(objectContainer, m_bxdfId);
        if (bxdfAsset)
        {
            SynchroniseObjects(GetDeviceInstance(), bxdfAsset->GetDeviceInstance());
        }
    }
}
    