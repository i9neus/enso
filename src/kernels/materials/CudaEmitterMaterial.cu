﻿#include "CudaEmitterMaterial.cuh"

namespace Cuda
{
    __host__ Host::EmitterMaterial::EmitterMaterial() :
        cu_deviceData(nullptr)
    {
        RenderObject::SetRenderObjectFlags(kIsChildObject);
        cu_deviceData = InstantiateOnDevice<Device::EmitterMaterial>();
    }

    __host__ void Host::EmitterMaterial::OnDestroyAsset()
    {
        DestroyOnDevice(cu_deviceData);
    }

    __host__ void Host::EmitterMaterial::UpdateParams(const vec3& radiance)
    {
        SynchroniseObjects(cu_deviceData, radiance);
    }
}