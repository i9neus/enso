﻿#include "CudaEmitterMaterial.cuh"

namespace Cuda
{
    __host__ Host::EmitterMaterial::EmitterMaterial(const std::string& id, const uint flags) :
        Material(id),
        cu_deviceData(nullptr)
    {
        RenderObject::SetRenderObjectFlags(flags);
        cu_deviceData = InstantiateOnDevice<Device::EmitterMaterial>(id);
    }

    __host__ void Host::EmitterMaterial::OnDestroyAsset()
    {
        DestroyOnDevice(GetAssetID(), cu_deviceData);
    }

    __host__ void Host::EmitterMaterial::UpdateParams(const vec3& radiance)
    {
        SynchroniseObjects(cu_deviceData, radiance);
    }
}