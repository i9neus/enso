#include "hip/hip_runtime.h"
﻿#include "CudaKIFSMaterial.cuh"
#include "generic/JsonUtils.h" 
#include "../bxdfs/CudaBxDF.cuh"
#include "../math/CudaColourUtils.cuh"

namespace Cuda
{
    __host__ __device__ KIFSMaterialParams::KIFSMaterialParams() :
        incandescenceHSV(vec3(0.0f)),
        albedoHSV(vec3(0.0f, 0.0f, 0.7f)),
        incandescenceRGB(vec3(0.0f))
    {
        albedoHSVRange[0] = albedoHSVRange[1] = vec3(0.0f, 0.0f, 1.0f);
    }

    __host__ KIFSMaterialParams::KIFSMaterialParams(const ::Json::Node& node, const uint flags) :
        KIFSMaterialParams()
    {
        FromJson(node, ::Json::kRequiredWarn);
    }

    __host__ void KIFSMaterialParams::ToJson(::Json::Node& node) const
    {
        incandescenceHSV.ToJson("incandescence", node);
        albedoHSV.ToJson("albedo", node);
    }

    __host__ void KIFSMaterialParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        incandescenceHSV.FromJson("incandescence", node, flags);
        albedoHSV.FromJson("albedo", node, flags);
        albedoHSVRange[0] = albedoHSV.p - albedoHSV.dpdt;
        albedoHSVRange[1] = albedoHSV.p + albedoHSV.dpdt;
        
        incandescenceRGB = HSVToRGB(incandescenceHSV());
    }

    __device__ void Device::KIFSMaterial::Evaluate(const HitCtx& hit, vec3& albedo, vec3& incandescence) const
    {
        const uint code = HashOf(*reinterpret_cast<const uint*>(&hit.uv.x));

        vec3 alpha((code & ((1 << 10) - 1)) / float((1 << 10) - 1),
                    ((code >> 10) & ((1 << 10) - 1)) / float((1 << 10) - 1),
                    ((code >> 20) & ((1 << 10) - 1)) / float((1 << 10) - 1));

        incandescence = m_params.incandescenceRGB;
        albedo =  HSVToRGB(cwiseMix(m_params.albedoHSVRange[0], m_params.albedoHSVRange[1], fmod(vec3(alpha) + m_params.albedoHSV.t, kOne)));
    }

    __host__ AssetHandle<Host::RenderObject> Host::KIFSMaterial::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kMaterial) { return AssetHandle<Host::RenderObject>(); }

        return AssetHandle<Host::RenderObject>(new Host::KIFSMaterial(json), id);
    }

    __host__ Host::KIFSMaterial::KIFSMaterial(const ::Json::Node& node) :
        Material(node),
        cu_deviceData(nullptr)
    {        
        cu_deviceData = InstantiateOnDevice<Device::KIFSMaterial>();
        FromJson(node, ::Json::kRequiredWarn);
    }

    __host__ void Host::KIFSMaterial::OnDestroyAsset()
    {
        DestroyOnDevice(cu_deviceData);
    }

    __host__ void Host::KIFSMaterial::FromJson(const ::Json::Node& parentNode, const uint flags)
    {
        Host::Material::FromJson(parentNode, flags);

        SynchroniseObjects(cu_deviceData, KIFSMaterialParams(parentNode, flags));
    }
}
