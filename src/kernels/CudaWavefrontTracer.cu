#include "hip/hip_runtime.h"
﻿#include "CudaWavefrontTracer.cuh"
#include "CudaSampler.cuh"
#include "CudaHash.cuh"
#include "generic/Assert.h"
#include "CudaAsset.cuh"
#include "CudaRay.cuh" 

#include "bxdfs/CudaLambert.cuh"
#include "tracables/CudaSphere.cuh"
#include "tracables/CudaPlane.cuh"
#include "tracables/CudaCornellBox.cuh"
#include "tracables/CudaKIFS.cuh"
#include "materials/CudaMaterial.cuh"
#include "lights/CudaQuadLight.cuh"

#include "CudaPerspectiveCamera.cuh"
#include "CudaManagedArray.cuh"

#include "CudaCommonIncludes.cuh"
#include "CudaImage.cuh"
#include "CudaCtx.cuh"

#include "generic/JsonUtils.h"

namespace Cuda
{
	__host__ WavefrontTracerParams::WavefrontTracerParams() : 
		maxDepth(1),
		ambientRadiance(0.0f),
		debugNormals(false),
		importanceMode(kImportanceMIS)
	{
	}
	
	__host__ void WavefrontTracerParams::ToJson(::Json::Node& node) const
	{
		node.AddValue("maxDepth", maxDepth);
		node.AddArray("ambientRadiance", std::vector<float>({ ambientRadiance.x, ambientRadiance.y, ambientRadiance.z }));
		node.AddValue("debugNormals", debugNormals);

		const std::vector<std::string> importanceModeIds({ "mis", "light", "bxdf" });
		node.AddEnumeratedParameter("importanceMode", importanceModeIds, importanceMode);
	}

	__host__ void WavefrontTracerParams::FromJson(const ::Json::Node& node, const uint flags)
	{
		node.GetValue("maxDepth", maxDepth, flags);
		node.GetVector("ambientRadiance", ambientRadiance, ::Json::kSilent);
		node.GetValue("debugNormals", debugNormals, flags);

		const std::vector<std::string> importanceModeIds({ "mis", "light", "bxdf" });
		node.GetEnumeratedParameter("importanceMode", importanceModeIds, importanceMode, flags);
	}

	__host__ bool WavefrontTracerParams::operator==(const WavefrontTracerParams& rhs) const
	{
		return maxDepth == rhs.maxDepth &&
			ambientRadiance == rhs.ambientRadiance;
	}
	
	__device__ void Device::WavefrontTracer::Synchronise(const Device::WavefrontTracer::Objects& objects)
	{
		m_objects = objects;
	}

	__device__ void Device::WavefrontTracer::Synchronise(const WavefrontTracerParams& params)
	{
		m_params = params;
	}

	__device__ void Device::WavefrontTracer::PreFrame(const float& wallTime, const int frameIdx)
	{
		m_wallTime = wallTime;
		m_frameIdx = frameIdx;
	}

	__device__ void Device::WavefrontTracer::SeedRayBuffer(const ivec2& viewportPos) const
	{
		if (!IsValid(viewportPos)) { return; }

		CompressedRay& compressedRay = (*m_objects.cu_deviceCompressedRayBuffer)[viewportPos.y * 512 + viewportPos.x];
		
		if (!compressedRay.IsAlive())
		{
			compressedRay.viewport.x = viewportPos.x;
			compressedRay.viewport.y = viewportPos.y;
			compressedRay.sampleIdx++;
			compressedRay.depth = 0;

			RenderCtx renderCtx(compressedRay, m_objects.viewportDims);
			m_objects.cu_camera->CreateRay(renderCtx);
		}	
	}
	
	__device__ __forceinline__ float PowerHeuristic(float pdf1, float pdf2)
	{
		return 2.0f * sqr(pdf1) / (sqr(pdf1) + sqr(pdf2));
	}

	__device__ uchar Device::WavefrontTracer::GetImportanceMode(const RenderCtx& ctx) const
	{
		return m_params.importanceMode;
		//return (ctx.emplacedRay.viewport.x < 256) ? m_params.importanceMode : kImportanceMIS;
	}

	__device__ vec3 Device::WavefrontTracer::Shade(const Ray& incidentRay, const Device::Material& material, const HitCtx& hitCtx, RenderCtx& renderCtx) const
	{	
		vec3 albedo, incandescence;
		material.Evaluate(hitCtx, albedo, incandescence);

		if (renderCtx.depth >= m_params.maxDepth) { return incandescence; }

		vec2 xi = renderCtx.Rand<2, 3>();

		const BxDF* bxdf = material.GetBoundBxDF();
		if (!bxdf) 
		{ 
			return incandescence; 
		}

		// If there are no lights in this scene, always sample the BxDF
		if (GetImportanceMode(renderCtx) == kImportanceBxDF || m_objects.cu_deviceLights->Size() == 0)
		{
			xi.x *= 0.5f;
		}

		// Indirect light sampling
		if(xi.x < 0.5f)
		{			
			vec3 extantDir;
			float pdfBxDF;
			if (bxdf->Sample(incidentRay, hitCtx, renderCtx, extantDir, pdfBxDF))
			{
				vec3 L = renderCtx.emplacedRay.weight * albedo;
				if (GetImportanceMode(renderCtx) != kImportanceBxDF) { L *= 2.0f; }

				renderCtx.EmplaceIndirectSample(RayBasic(hitCtx.ExtantOrigin(), extantDir), L);
			}
		}
		// Direct light sampling
		else
		{		
			// Rescale the random number
			xi.x = (GetImportanceMode(renderCtx) == kImportanceLight) ? 0.0f : (xi.x * 2.0f - 1.0f);
			
			// Randomly select a light
			const int lightIdx = min(m_objects.cu_deviceLights->Size() - 1, uint(xi.y * m_objects.cu_deviceLights->Size()));
			const Light& light = *(*m_objects.cu_deviceLights)[lightIdx];

			float pdfBxDF, pdfLight;
			vec3 extantDir, L;

			// Sample the light
			if (xi.x < 0.5f)
			{
				if (light.Sample(incidentRay, hitCtx, renderCtx, extantDir, L, pdfLight))
				{
					float weightBxDF;
					bxdf->Evaluate(incidentRay.od.d, extantDir, hitCtx, weightBxDF, pdfBxDF);

					L *= renderCtx.emplacedRay.weight * albedo * 2.0f * weightBxDF; // Factor of two here accounts for stochastic dithering between direct and indirect sampling

					// If MIS is enabled, weight the ray using the power heuristic
					if (GetImportanceMode(renderCtx) == kImportanceMIS)
					{
						L *= PowerHeuristic(pdfLight, pdfBxDF);
					}

					renderCtx.EmplaceDirectSample(RayBasic(hitCtx.ExtantOrigin(), extantDir), L, pdfLight, lightIdx, kRayDirectLightSample);
				}
			}
			// Sample the BxDF
			else if (bxdf->Sample(incidentRay, hitCtx, renderCtx, extantDir, pdfBxDF))
			{	
				renderCtx.EmplaceDirectSample(RayBasic(hitCtx.ExtantOrigin(), extantDir), 
					renderCtx.emplacedRay.weight * albedo * 2.0f,
					pdfBxDF, lightIdx, kRayDirectBxDFSample);
			}	
		}

		return incandescence;
	}

	__device__ void Device::WavefrontTracer::PreBlock() const
	{
		for (int i = 0; i < m_objects.cu_deviceTracables->Size(); i++)
		{
			(*m_objects.cu_deviceTracables)[i]->InitialiseKernelConstantData();
		}
	}

	__device__ void Device::WavefrontTracer::Trace(const uint rayIdx) const
	{		
		if (rayIdx >= m_objects.cu_deviceCompressedRayBuffer->Size()) { return; }

		CompressedRay& compressedRay = (*m_objects.cu_deviceCompressedRayBuffer)[rayIdx];
		Ray incidentRay(compressedRay);
		RenderCtx renderCtx(compressedRay, m_objects.viewportDims);
		
		//m_objects.cu_deviceAccumBuffer->At(renderCtx.viewportPos) = vec4(renderCtx.viewportPos.x / 512.0f, renderCtx.viewportPos.y / 512.0f, 0.0f, -1.0f);
		//return;

		compressedRay.Kill();

		// INTERSECTION
		HitCtx hitCtx;
		auto& tracables = *m_objects.cu_deviceTracables;
		Device::Tracable* hitObject = nullptr;
		for (int i = 0; i < tracables.Size(); i++)
		{
			if (tracables[i]->Intersect(incidentRay, hitCtx))
			{
				hitObject = tracables[i];
			}
		}
		
		vec3 L(0.0f);
		
		if (!hitObject)
		{
			// Ray didn't hit anything so add the ambient term multiplied by the weight
			L = m_params.ambientRadiance * compressedRay.weight;
		}
		else
		{		
			// Ray is a direct sample 
			if (incidentRay.IsDirectSample())
			{
				// Check that the intersected tracable is the same as the light ID associated with this ray
				if (compressedRay.lightId == hitObject->GetLightID())
				{
					// Light should be evaluated (i.e. BxDF was sampled)
					if (incidentRay.flags & kRayDirectBxDFSample)
					{
						const Light* light = (*m_objects.cu_deviceLights)[compressedRay.lightId];
						if (!light) { L = kPink; }
						else
						{
							float pdfLight;
							light->Evaluate(incidentRay, hitCtx, L, pdfLight);

							L *= compressedRay.weight;
							if (GetImportanceMode(renderCtx) == kImportanceMIS)
							{
								L *= PowerHeuristic(compressedRay.pdf, pdfLight);
							}
						}
					}
					// If the light itself was sampled, everything's baked into the throughput
					else
					{
						L = compressedRay.weight;
					}
				}
			}
			else if(hitObject->GetLightID() == kNotALight || GetImportanceMode(renderCtx) == kImportanceBxDF)
			{			
				// Otherwise, it's a BxDF sample so do a regular shade op
				L = Shade(incidentRay, *(hitObject->GetBoundMaterial()), hitCtx, renderCtx) * compressedRay.weight;
				//if(compressedRay.IsAlive()) L = compressedRay.od.d * 0.5f + vec3(0.5f);
			}	
		}

		// Accumulate radiance if we're above a certain threshold
		//if (cwiseMax(L) > 1e-6f)
		{
			m_objects.cu_deviceAccumBuffer->Accumulate(renderCtx.viewportPos, L, renderCtx.depth, compressedRay.IsAlive());
		}
	}

	__device__ void Device::WavefrontTracer::Composite(const ivec2& viewportPos, Device::ImageRGBA* deviceOutputImage) const
	{
		if (viewportPos.x >= deviceOutputImage->Width() || viewportPos.y >= deviceOutputImage->Height() ||
			viewportPos.x >= m_objects.cu_deviceAccumBuffer->Width() || viewportPos.y >= m_objects.cu_deviceAccumBuffer->Height()) {
			return;
		}

		// If the texel weight is negative, the texel is ready to be rendered
		vec4& texel = m_objects.cu_deviceAccumBuffer->At(viewportPos);
		if (texel.w >= 0.0f) { return; }

		CompressedRay& compressedRay = (*m_objects.cu_deviceCompressedRayBuffer)[kKernelIdx];

		// Flip the weight back to positve
		texel.w = -texel.w;

		const vec3 rgb = texel.xyz / fmax(1.0f, texel.w);
		deviceOutputImage->At(viewportPos) = vec4(rgb, 1.0f);
	}

	__host__ void Host::WavefrontTracer::OnDestroyAsset()
	{
		if (!m_hostCompressedRayBuffer) { return; }

		m_hostCompressedRayBuffer.DestroyAsset();
		m_hostAccumBuffer.DestroyAsset();
		m_hostTracables.DestroyAsset();
		m_hostLights.DestroyAsset();

		DestroyOnDevice(cu_deviceData);
	}

	__host__ AssetHandle<Host::RenderObject> Host::WavefrontTracer::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
	{
		if (expectedType != AssetType::kIntegrator) { return AssetHandle<Host::RenderObject>(); }

		return AssetHandle<Host::RenderObject>(new Host::WavefrontTracer(json), id);
	}

	__host__ Host::WavefrontTracer::WavefrontTracer(const ::Json::Node& node) :
		cu_deviceData(nullptr),
		m_isDirty(true)
	{
		// Create the packed ray buffer
		m_hostCompressedRayBuffer = AssetHandle<Host::CompressedRayBuffer>("id_hostCompressedRayBuffer", 512 * 512, m_hostStream);
		m_hostCompressedRayBuffer->Clear(CompressedRay());

		// Create the accumulation buffer
		m_hostAccumBuffer = AssetHandle<Host::ImageRGBW>("id_hostAccumBuffer", 512, 512, m_hostStream);
		m_hostAccumBuffer->Clear(vec4(0.0f));

		m_hostTracables = AssetHandle<Host::AssetContainer<Host::Tracable>>("wavefront_tracablesContainer");
		m_hostLights = AssetHandle<Host::AssetContainer<Host::Light>>("wavefront_lightsContainer");	

		cu_deviceData = InstantiateOnDevice<Device::WavefrontTracer>();
		FromJson(node, ::Json::kRequiredWarn);
		
		m_block = dim3(16, 16, 1);
		m_grid = dim3((m_hostAccumBuffer->GetHostInstance().Width() + 15) / 16, (m_hostAccumBuffer->GetHostInstance().Height() + 15) / 16, 1);
	}

	__host__ Host::WavefrontTracer::~WavefrontTracer() 
	{ 
		OnDestroyAsset(); 
	}

	__host__ void Host::WavefrontTracer::Bind(RenderObjectContainer& sceneObjects)
	{
		Log::Indent indent;
		for (auto& object : sceneObjects)
		{
			const auto type = object->GetAssetType();
			
			if (type == AssetType::kTracable)
			{
				Log::Debug("Linked tracable '%s' to wavefront tracer.\n", object->GetAssetID());

				Cuda::AssetHandle<Host::Tracable> tracable = object.DynamicCast<Tracable>();
				Assert(tracable);
				m_hostTracables->Push(tracable);
			}
			else if(type == AssetType::kLight)
			{
				Log::Debug("Linked light '%s' to wavefront tracer.\n", object->GetAssetID());

				Cuda::AssetHandle<Host::Light> light = object.DynamicCast<Light>();				
				Assert(light);				
			
				// Set the light ID for this tracable with the index of the light in the array. Crude, but it'll do for now. 
				Cuda::AssetHandle<Host::Tracable> tracable = light->GetTracableHandle();
				const uchar lightId = m_hostLights->Size();
				light->SetLightID(lightId);
				tracable->SetLightID(lightId);

				m_hostLights->Push(light);
			}
		}

		// Synchronise the container objects managed by this instance
		m_hostTracables->Synchronise();
		m_hostLights->Synchronise();

		// Synchronise the wavefront tracer structure on the device
		m_hostObjects.cu_deviceAccumBuffer = m_hostAccumBuffer->GetDeviceInstance();
		m_hostObjects.cu_deviceCompressedRayBuffer = m_hostCompressedRayBuffer->GetDeviceInstance();
		m_hostObjects.cu_deviceTracables = m_hostTracables->GetDeviceInstance();
		m_hostObjects.cu_deviceLights = m_hostLights->GetDeviceInstance();
		m_hostObjects.viewportDims = m_hostAccumBuffer->GetHostInstance().Dimensions();

		m_cameraAsset = GetAssetHandleForBinding<Host::WavefrontTracer, Host::PerspectiveCamera>(sceneObjects, m_cameraId);
		if (m_cameraAsset)
		{
			m_hostObjects.cu_camera = m_cameraAsset->GetDeviceInstance();
		}

		SynchroniseObjects(cu_deviceData, m_hostObjects);
		Log::Write("Bound tracables and lights to wavefront tracer '%s'.\n", GetAssetID());
	}

	__host__ void Host::WavefrontTracer::FromJson(const ::Json::Node& parentNode, const uint flags)
	{		
		Host::RenderObject::FromJson(parentNode, flags);

		SynchroniseObjects(cu_deviceData, WavefrontTracerParams(parentNode, flags));

		parentNode.GetValue("camera", m_cameraId, flags);
		m_isDirty = true;
	}

	__global__ void KernelPreFrame(Device::WavefrontTracer* tracer, const float wallTime, const int frameIdx)
	{
		tracer->PreFrame(wallTime, frameIdx);
	}

	__global__ void KernelSeedRayBuffer(Device::WavefrontTracer* tracer)
	{
		tracer->SeedRayBuffer(kKernelPos<ivec2>());
	}

	__global__ void KernelTrace(Device::WavefrontTracer* tracer)
	{
		if (kThreadIdx == 0)
		{
			tracer->PreBlock();
		}
		__syncthreads();

		tracer->Trace(kKernelIdx);
	}

	__global__ void KernelComposite(Device::ImageRGBA* deviceOutputImage, const Device::WavefrontTracer* tracer)
	{
		//if (*(deviceOutputImage->AccessSignal()) != kImageWriteLocked) { return; }

		tracer->Composite(kKernelPos<ivec2>(), deviceOutputImage);
	}

	__host__ void Host::WavefrontTracer::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage)
	{
		//std::printf("Composite! %i %i %i\n", m_grid.x, m_grid.y, m_grid.z);
	
		hostOutputImage->SignalSetWrite(m_hostStream);
		KernelComposite << < m_grid, m_block, 0, m_hostStream >> > (hostOutputImage->GetDeviceInstance(), cu_deviceData);
		hostOutputImage->SignalUnsetWrite(m_hostStream);
	}

	__host__ void Host::WavefrontTracer::Iterate(const float wallTime, const float frameIdx)
	{
		//std::printf("Iterate! %f\n", wallTime);

		if (m_isDirty)
		{
			m_hostAccumBuffer->Clear(vec4(0.0f));
			m_hostCompressedRayBuffer->Clear(Cuda::CompressedRay());
			//m_hostPixelFlagsBuffer->Clear(0);
			m_isDirty = false;
		}
		
		KernelPreFrame << < 1, 1, 0, m_hostStream >> > (cu_deviceData, wallTime, frameIdx);

		KernelSeedRayBuffer << < m_grid, m_block, 0, m_hostStream >> > (cu_deviceData);

		KernelTrace << <  m_hostCompressedRayBuffer->NumBlocks(), m_hostCompressedRayBuffer->ThreadsPerBlock(), 0, m_hostStream >> > (cu_deviceData);
	}
}