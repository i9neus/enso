#include "hip/hip_runtime.h"
﻿#include "CudaWavefrontTracer.cuh"
#include "CudaSampler.cuh"
#include "CudaHash.cuh"
#include "generic/Assert.h"
#include "CudaAsset.cuh"

namespace Cuda
{
	__global__ void KernelSeedRayBuffer(Device::WavefrontTracer* tracer)
	{		
		tracer->SeedRayBuffer(KERNEL_COORDS_IVEC2);
	}

	__global__ void KernelTrace(Device::WavefrontTracer* tracer)
	{
		tracer->Trace(KERNEL_COORDS_IVEC2);
	}

	__global__ void KernelComposite(Device::ImageRGBA* deviceOutputImage, const Device::WavefrontTracer* tracer)
	{
		//if (*(deviceOutputImage->AccessSignal()) != kImageWriteLocked) { return; }

		tracer->Composite(KERNEL_COORDS_IVEC2, deviceOutputImage);
	}

	Device::WavefrontTracer::WavefrontTracer()
	{
		cu_deviceAccumBuffer = nullptr;
		cu_deviceCompressedRayBuffer = nullptr;		 
	}

	__device__ Device::RenderCtx Device::WavefrontTracer::CreateRenderCtx(const ivec2& viewportPos, const uint depth) const
	{
		int seed = int(hashOf(depth, uint(viewportPos.x), uint(viewportPos.y)));
		
		RenderCtx ctx;
		ctx.pcg.Initialise(seed);
		ctx.viewportPos = viewportPos;
		ctx.viewportDims = m_viewportDims;

		return ctx;
	}

	__device__ void Device::WavefrontTracer::SeedRayBuffer(const ivec2& viewportPos) const
	{
		if (!IsValid(viewportPos)) { return; }
		
		CompressedRay& packedRay = cu_deviceCompressedRayBuffer->At(viewportPos);

		if (!packedRay.IsAlive())
		{
			RenderCtx renderCtx = CreateRenderCtx(viewportPos, 0u);
			m_camera.CreateRay(packedRay, renderCtx);
			packedRay.SetAlive();
		}

		//cu_deviceAccumBuffer->At(viewportPos) = vec4(newRay.od.d, 1.0f);
	}

	__device__ void Device::WavefrontTracer::Trace(const ivec2& viewportPos) const
	{
		if (!IsValid(viewportPos)) { return; }
		
		Ray ray = DeriveRay(cu_deviceCompressedRayBuffer->At(viewportPos));
		HitCtx hit;

		//for (int i = 0; i < cu_deviceTracables->Size(); i++)
		{
			if(cu_sphere->Intersect(ray, hit))
			{
				cu_deviceAccumBuffer->Accumulate(viewportPos, hit.n);
			}
		}
	}

	__device__ void Device::WavefrontTracer::Composite(const ivec2& viewportPos, Device::ImageRGBA* deviceOutputImage) const
	{		
		if (viewportPos.x >= deviceOutputImage->Width() || viewportPos.y >= deviceOutputImage->Height() ||
			viewportPos.x >= cu_deviceAccumBuffer->Width() || viewportPos.y >= cu_deviceAccumBuffer->Height()) {
			return;
		}

		vec4 texel = cu_deviceAccumBuffer->At(viewportPos);
		texel.xyz /= fmax(1.0f, texel.w);
		texel.w = 1.0f;

		deviceOutputImage->At(viewportPos) = texel;
	}

	__host__ void Host::WavefrontTracer::OnDestroyAsset()
	{
		if (!m_hostCompressedRayBuffer) { return; }
		
		m_hostCompressedRayBuffer.DestroyAsset();
		m_hostAccumBuffer.DestroyAsset();
		m_hostTracables.DestroyAsset();
		m_hostSphere.DestroyAsset();

		DestroyOnDevice(&cu_deviceData);
	}

	__host__ Host::WavefrontTracer::WavefrontTracer(hipStream_t hostStream) :
		cu_deviceData(nullptr),
		m_hostStream(hostStream)
	{
		// Create the packed ray buffer
		m_hostCompressedRayBuffer = AssetHandle<Host::CompressedRayBuffer>("id_hostCompressedRayBuffer", 512, 512, m_hostStream);

		// Create the accumulation buffer
		m_hostAccumBuffer = AssetHandle<Host::ImageRGBW>("id_hostAccumBuffer", 512, 512, m_hostStream);
		m_hostAccumBuffer->Clear(vec4(0.0f));

		m_hostTracables = AssetHandle<Host::AssetContainer<Host::Tracable>>("id_tracableContainer");

		m_hostSphere = AssetHandle<Host::Sphere>(new Host::Sphere(vec3(0.0f), 1.0f), "id_sphere");
		//m_hostTracables->Push(newSphere);		
		//m_hostTracables->Sync();

		checkCudaErrors(hipDeviceSynchronize());

		// Create the wavefront tracer structure on the device
		m_hostData.cu_deviceAccumBuffer = m_hostAccumBuffer->GetDeviceInstance();
		m_hostData.cu_deviceCompressedRayBuffer = m_hostCompressedRayBuffer->GetDeviceInstance();
		//cu_deviceTracables = m_hostTracables->GetDeviceInstance();
		m_hostData.m_viewportDims = m_hostAccumBuffer->GetHostInstance().Dimensions();
		m_hostData.cu_sphere = m_hostSphere->GetDeviceInstance();

		InstantiateOnDevice(&cu_deviceData, m_hostData.cu_deviceAccumBuffer,
								 			m_hostData.cu_deviceCompressedRayBuffer, 
											m_hostData.cu_sphere,
											m_hostData.m_viewportDims);
		
		m_block = dim3(16, 16, 1);
		m_grid = dim3((m_hostAccumBuffer->GetHostInstance().Width() + 15) / 16, (m_hostAccumBuffer->GetHostInstance().Height() + 15) / 16, 1);

		std::printf("%i, %i, %i\n", m_grid.x, m_grid.y, m_grid.z);
	}

	__host__ void Host::WavefrontTracer::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage)
	{
		std::printf("Composite! %i %i %i\n", m_grid.x, m_grid.y, m_grid.z);
	
		KernelComposite << < m_grid, m_block, 0, m_hostStream >> > (hostOutputImage->GetDeviceInstance(), cu_deviceData);
	}

	__host__ void Host::WavefrontTracer::Iterate()
	{
		std::printf("Iterate!\n");

		KernelSeedRayBuffer << < m_grid, m_block, 0, m_hostStream >> > (cu_deviceData);

		KernelTrace << < m_grid, m_block, 0, m_hostStream >> > (cu_deviceData);
	}
}