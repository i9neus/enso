#include "hip/hip_runtime.h"
﻿#include "CudaWavefrontTracer.cuh"
#include "CudaSampler.cuh"
#include "CudaHash.cuh"
#include "generic/Assert.h"
#include "CudaAsset.cuh"

namespace Cuda
{
	__device__ void Device::WavefrontTracer::PreFrame(const float& wallTime, const int frameIdx)
	{
		m_wallTime = wallTime;
		m_frameIdx = frameIdx;

		//auto transform = CreateCompoundTransform(vec3(0.8f, 1.1f, 0.9f) * wallTime);
		BidirectionalTransform transform;		
		cu_cornell->SetTransform(transform);
		cu_sphere->SetTransform(transform);
	}

	__device__ void Device::WavefrontTracer::SeedRayBuffer(const ivec2& viewportPos) const
	{
		if (!IsValid(viewportPos)) { return; }
		
		CompressedRay& compressedRay = (*cu_deviceCompressedRayBuffer)[viewportPos.y * 512 + viewportPos.x];

		if (!compressedRay.IsAlive())
		{
			RenderCtx renderCtx(viewportPos, m_viewportDims, m_wallTime, m_frameIdx, 0);
			m_camera.CreateRay(compressedRay, renderCtx);			
		}

		//cu_deviceAccumBuffer->At(viewportPos) = vec4(newRay.od.d, 1.0f);
	}

	__device__ vec3 Device::WavefrontTracer::Shade(const Ray& incidentRay, const HitCtx& hitCtx, RenderCtx& renderCtx) const
	{
		if (incidentRay.depth >= 5) { return kZero; }
		
		const vec4 xi = renderCtx.Rand4();

		vec3 brdfDir;
		float brdfPdf;
		if (cu_lambert->Sample(incidentRay, hitCtx, renderCtx, brdfDir, brdfPdf))
		{
			const vec3 weight = incidentRay.weight * 0.8f;
			
			renderCtx.EmplaceRay(RayBasic(hitCtx.ExtantOrigin(), brdfDir), weight, brdfPdf, incidentRay.lambda, 0, incidentRay.depth);
		}

		return kZero;
	}		 

	__device__ void Device::WavefrontTracer::Trace(const uint rayIdx) const
	{		
		if (rayIdx >= cu_deviceCompressedRayBuffer->Size()) { return; }
		
		CompressedRay& compressedRay = (*cu_deviceCompressedRayBuffer)[rayIdx];
		Ray incidentRay(compressedRay);
		RenderCtx renderCtx(compressedRay.ViewportPos(), m_viewportDims, m_wallTime, m_frameIdx, compressedRay.depth);
		vec3 L(0.0f);
		const vec2 viewportPos = compressedRay.ViewportPos();

		int depth = incidentRay.depth;

		// INTERSECTION 
		HitCtx hitCtx;
		//for (int i = 0; i < cu_deviceTracables->Size(); i++)
		{
			cu_cornell->Intersect(incidentRay, hitCtx);
			//cu_sphere->Intersect(incidentRay, hitCtx);
		}

		// SHADE
		if (!hitCtx.isValid)
		{
			L += incidentRay.weight * vec3(1.0f);
		}
		else
		{
			//L += hitCtx.hit.n * 0.5f + vec3(0.5f);
			L += Shade(incidentRay, hitCtx, renderCtx);
		}

		if (renderCtx.emplacedRay.IsAlive())
		{
			compressedRay = renderCtx.emplacedRay;
		}
		else
		{
			compressedRay.Kill();
		}

		//L += incidentRay.od.d;
		//cu_deviceAccumBuffer->At(viewportPos) = 0.0f;
		cu_deviceAccumBuffer->Accumulate(viewportPos, L, incidentRay.depth);
		//cu_deviceAccumBuffer->At(viewportPos) += vec4(L, 1.0f);
	}

	__device__ void Device::WavefrontTracer::Composite(const ivec2& viewportPos, Device::ImageRGBA* deviceOutputImage) const
	{		
		if (viewportPos.x >= deviceOutputImage->Width() || viewportPos.y >= deviceOutputImage->Height() ||
			viewportPos.x >= cu_deviceAccumBuffer->Width() || viewportPos.y >= cu_deviceAccumBuffer->Height()) {
			return;
		}

		vec4 texel = cu_deviceAccumBuffer->At(viewportPos);
		texel.xyz /= fmax(1.0f, texel.w);
		texel.w = 1.0f;

		deviceOutputImage->At(viewportPos) = texel;
	}

	__host__ void Host::WavefrontTracer::OnDestroyAsset()
	{
		if (!m_hostCompressedRayBuffer) { return; }
		
		m_hostCompressedRayBuffer.DestroyAsset();
		m_hostAccumBuffer.DestroyAsset();
		m_hostTracables.DestroyAsset();
		m_hostCornell.DestroyAsset();

		DestroyOnDevice(&cu_deviceData);
	}

	__host__ Host::WavefrontTracer::WavefrontTracer(hipStream_t hostStream) :
		cu_deviceData(nullptr),
		m_hostStream(hostStream)
	{
		// Create the packed ray buffer
		m_hostCompressedRayBuffer = AssetHandle<Host::CompressedRayBuffer>("id_hostCompressedRayBuffer", 512 * 512, m_hostStream);

		// Create the accumulation buffer
		m_hostAccumBuffer = AssetHandle<Host::ImageRGBW>("id_hostAccumBuffer", 512, 512, m_hostStream);
		m_hostAccumBuffer->Clear(vec4(0.0f));

		m_hostTracables = AssetHandle<Host::AssetContainer<Host::Tracable>>("id_tracableContainer");

		m_hostCornell = AssetHandle<Host::Cornell>(new Host::Cornell(), "id_cornell");
		m_hostSphere = AssetHandle<Host::Sphere>(new Host::Sphere(), "id_sphere");
		//m_hostTracables->Push(newSphere);
		//m_hostTracables->Sync();

		m_hostLambert = AssetHandle<Host::LambertBRDF>(new Host::LambertBRDF(), "id_lambert");

		checkCudaErrors(hipDeviceSynchronize());

		// Create the wavefront tracer structure on the device
		m_hostData.cu_deviceAccumBuffer = m_hostAccumBuffer->GetDeviceInstance();
		m_hostData.cu_deviceCompressedRayBuffer = m_hostCompressedRayBuffer->GetDeviceInstance();
		//cu_deviceTracables = m_hostTracables->GetDeviceInstance();
		m_hostData.m_viewportDims = m_hostAccumBuffer->GetHostInstance().Dimensions();
		m_hostData.cu_cornell = m_hostCornell->GetDeviceInstance();
		m_hostData.cu_sphere = m_hostSphere->GetDeviceInstance();

		InstantiateOnDevice(&cu_deviceData, m_hostData.cu_deviceAccumBuffer,
								 			m_hostData.cu_deviceCompressedRayBuffer, 
											m_hostData.cu_cornell,
											m_hostData.cu_sphere,
											m_hostData.cu_lambert,
											m_hostData.m_viewportDims);
		
		m_block = dim3(16, 16, 1);//
		m_grid = dim3((m_hostAccumBuffer->GetHostInstance().Width() + 15) / 16, (m_hostAccumBuffer->GetHostInstance().Height() + 15) / 16, 1);

		std::printf("%i, %i, %i\n", m_grid.x, m_grid.y, m_grid.z);
	}

	__global__ void KernelPreFrame(Device::WavefrontTracer* tracer, const float wallTime, const int frameIdx)
	{
		tracer->PreFrame(wallTime, frameIdx);
	}

	__global__ void KernelSeedRayBuffer(Device::WavefrontTracer* tracer)
	{
		tracer->SeedRayBuffer(KERNEL_COORDS_IVEC2);
	}

	__global__ void KernelTrace(Device::WavefrontTracer* tracer)
	{
		tracer->Trace(blockIdx.x * blockDim.x + threadIdx.x);
	}

	__global__ void KernelComposite(Device::ImageRGBA* deviceOutputImage, const Device::WavefrontTracer* tracer)
	{
		//if (*(deviceOutputImage->AccessSignal()) != kImageWriteLocked) { return; }

		tracer->Composite(KERNEL_COORDS_IVEC2, deviceOutputImage);
	}

	__host__ void Host::WavefrontTracer::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage)
	{
		//std::printf("Composite! %i %i %i\n", m_grid.x, m_grid.y, m_grid.z);
	
		KernelComposite << < m_grid, m_block, 0, m_hostStream >> > (hostOutputImage->GetDeviceInstance(), cu_deviceData);
	}

	__host__ void Host::WavefrontTracer::Iterate(const float wallTime, const float frameIdx)
	{
		//std::printf("Iterate! %f\n", wallTime);
		
		KernelPreFrame << < 1, 1, 0, m_hostStream >> > (cu_deviceData, wallTime, frameIdx);

		KernelSeedRayBuffer << < m_grid, m_block, 0, m_hostStream >> > (cu_deviceData);

		KernelTrace << <  m_hostCompressedRayBuffer->NumBlocks(), m_hostCompressedRayBuffer->ThreadsPerBlock(), 0, m_hostStream >> > (cu_deviceData);
	}
}