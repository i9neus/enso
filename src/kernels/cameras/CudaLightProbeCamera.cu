#include "hip/hip_runtime.h"
﻿#define CUDA_DEVICE_ASSERTS

#include "CudaLightProbeCamera.cuh"
#include "generic/JsonUtils.h"

#include "../CudaCtx.cuh"
#include "../CudaManagedArray.cuh"
#include "../CudaManagedObject.cuh"

#include "../math/CudaSphericalHarmonics.cuh"

#include "../../io/USDIO.h"

#define kAccumBufferWidth 1024u
#define kAccumBufferHeight 1024u
#define kAccumBufferSize (kAccumBufferWidth * kAccumBufferHeight)

#define kRayBufferSize          (512u * 512u * 2u)
#define kRayBufferNumBuckets    (512u * 512u)

namespace Cuda
{
    __host__ __device__ LightProbeCameraParams::LightProbeCameraParams()
    {
        lightingMode = kBakeLightingCombined;
        traversalMode = kBakeTraversalLinear;
        gridUpdateInterval = 10;
        minViableValidity = 0.0f;
        filterGrids = false;
    }

    __host__ LightProbeCameraParams::LightProbeCameraParams(const ::Json::Node& node, const uint flags) :
        LightProbeCameraParams()
    {
        FromJson(node, flags);
    }

    __host__ void LightProbeCameraParams::ToJson(::Json::Node& node) const
    {
        auto gridNode = node.AddChildObject("grid");
        grid.ToJson(gridNode);
        camera.ToJson(node);

        node.AddEnumeratedParameter("lightingMode", std::vector<std::string>({ "combined", "separated" }), lightingMode);
        node.AddEnumeratedParameter("traversalMode", std::vector<std::string>({ "linear", "hilbert" }), traversalMode);
        node.AddValue("gridUpdateInterval", gridUpdateInterval);
        node.AddValue("minViableValidity", minViableValidity);
        node.AddValue("filterGrids", filterGrids);
    }

    __host__ void LightProbeCameraParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        auto gridNode = node.GetChildObject("grid", flags);
        grid.FromJson(gridNode, flags);
        camera.FromJson(node, flags);

        node.GetEnumeratedParameter("lightingMode", std::vector<std::string>({ "combined", "separated" }), lightingMode, flags);
        node.GetEnumeratedParameter("traversalMode", std::vector<std::string>({ "linear", "hilbert" }), traversalMode, flags);
        node.GetValue("gridUpdateInterval", gridUpdateInterval, flags);
        node.GetValue("minViableValidity", minViableValidity, flags);
        node.GetValue("filterGrids", filterGrids, flags);
    }      

    __device__ Device::LightProbeCamera::LightProbeCamera() {  }

    __device__ void Device::LightProbeCamera::Synchronise(const LightProbeCameraParams& params)
    {
        m_params = params;

        Prepare();
    }
    __device__ void Device::LightProbeCamera::Synchronise(const Objects& objects)
    {
        m_objects = objects;
    }

    __device__ void Device::LightProbeCamera::SeedRayBuffer(const int frameIdx)
    {
        assert(kKernelIdx * 2 < kRayBufferSize);

        int probeIdx = kKernelIdx / m_params.subprobesPerProbe;
        if (probeIdx >= m_params.grid.numProbes) { return; }

        // FIXME: This asserts, but it shouldn't. Find out why.
        //assert(probeIdx < m_params.grid.numProbes);

        // Apply indirection
        if (m_params.traversalMode != kBakeTraversalLinear)
        {
            probeIdx = (*m_objects.cu_indirectionBuffer)[probeIdx];
        }
        
        // If adaptive sampling is enabled and the probe is converged, don't spawn any more rays.
        if (m_params.camera.samplingMode != kCameraSamplingFixed && (*m_objects.cu_convergenceGrid)[probeIdx] == 0) { return; }
        
        CompressedRay* compressedRays = &(*m_objects.renderState.cu_compressedRayBuffer)[kKernelIdx * 2];

        if (kKernelIdx > m_params.totalBuckets) 
        {
            compressedRays[0].Kill();
            compressedRays[1].Kill();
            return;
        }

        // On the first frame, reset the ray and the sample index
        if (frameIdx == 0)
        {
            compressedRays[0].Reset();
            compressedRays[1].Reset();
            compressedRays[0].sampleIdx = m_seedOffset;
        }

        if (!compressedRays[0].IsAlive() && !compressedRays[1].IsAlive() &&
            int(compressedRays[0].sampleIdx - m_seedOffset) < m_params.minMaxSamplesPerSubprobe.y)
        {
            CreateRays(probeIdx, kKernelIdx % m_params.subprobesPerProbe, compressedRays, frameIdx);
        }
    }

    __device__ void Device::LightProbeCamera::Composite(const ivec2& accumPos, Device::ImageRGBA* deviceOutputImage) const
    {
        const ivec2 viewportPos = accumPos + deviceOutputImage->Dimensions() / 2 - ivec2(kAccumBufferWidth, kAccumBufferHeight) / 2;
        if (viewportPos.x < 0 || viewportPos.x >= deviceOutputImage->Width() ||
            viewportPos.y < 0 || viewportPos.y >= deviceOutputImage->Height()) {
            return;
        }

        assert(accumPos.y * kAccumBufferWidth + accumPos.x <= m_objects.cu_reduceBuffer->Size());
        //assert(m_objects.cu_reduceBuffer->Size() == kAccumBufferSize);

        int idx = accumPos.x / (kAccumBufferWidth / 2);

        // Normalise and gamma correct
        const auto& texel = (*m_objects.cu_accumBuffers[idx])[accumPos.y * kAccumBufferWidth + accumPos.x];
        const vec3 rgb = texel.xyz / fmax(1.0f, texel.w);       
        deviceOutputImage->At(viewportPos) = vec4(rgb, 1.0f);
    }

    __device__ void Device::LightProbeCamera::Prepare()
    {
        assert(m_objects.cu_accumBuffers[0] && m_objects.cu_accumBuffers[1]);
        assert(m_objects.cu_probeGrids[0] && m_objects.cu_probeGrids[1]);

        // Only use the lower 31 bits for the seed because we need to deduce the actual sample count from it
        m_seedOffset = HashOf(uint(m_params.camera.seed) & ((1u << 31) - 1u));
    }

    __device__ void Device::LightProbeCamera::CreateRays(const int& probeIdx, const int& subsampleIdx, CompressedRay* rays, const int frameIdx) const
    {        
        //const int probeIdx = kKernelIdx / m_params.subprobesPerProbe;
        const ivec3 gridIdx = GridPosFromProbeIdx(probeIdx, m_params.grid.gridDensity);
        const uint accumIdx = probeIdx * m_params.subprobesPerProbe + subsampleIdx;

        auto& primary = rays[0]; 
        auto& secondary = rays[1];
        
        // NOTE: Ray depth corresponds to the position on the graph at the hit-point of the ray.
        // Probe 0 -> 1 -> 2 -> ...
        primary.accumIdx = accumIdx;
        primary.sampleIdx++;
        primary.depth = 0; // Set depth to zero so the RNG is correctly seeded
        RNG rng(primary);

        const vec2 xi = rng.Rand<0, 1>();
        primary.od.d = SampleUnitSphere(xi);
        primary.probeDir = primary.od.d;

        // Probes data is grouped as follows:
        // Subsample    = [SH data] [Auxilliary data]
        // Probe        = [Subsample 1] .... [Subsample N]
        // Grid         = [Probe 1] ... [Probe M]        

        // Project this direction into SH and pre-normalise
        primary.weight = kOne;
        primary.depth = 1;
        primary.flags = kRayIndirectSample | kRayLightProbe;

        primary.od.o = m_params.grid.aspectRatio * vec3(gridIdx) / vec3(m_params.grid.gridDensity - 1) - vec3(0.5f);
        primary.od.o = m_params.grid.transform.PointToWorldSpace(primary.od.o);

        secondary.accumIdx = accumIdx;
        secondary.probeDir = primary.probeDir;
        secondary.sampleIdx = primary.sampleIdx;
    }

    __device__ void Device::LightProbeCamera::Accumulate(const RenderCtx& ctx, const Ray& incidentRay, const HitCtx& hitCtx, const vec3& value, const bool isAlive)
    {         
        auto& emplacedRay = ctx.emplacedRay[0];
        assert(emplacedRay.accumIdx < kAccumBufferSize);
        
        vec3 L = value;
        if (m_params.camera.splatClamp > 0.0)
        {
            const float intensity = cwiseMax(L);
            if (intensity > m_params.camera.splatClamp)
            {
                L *= m_params.camera.splatClamp / intensity;
            }
        }
  
        // Loop through the direct and indirect accumulation buffers
        for (int gridIdx = 0; gridIdx < kLightProbeNumBuffers; ++gridIdx)
        {
            if (!m_objects.cu_accumBuffers[gridIdx]) { continue; }

            // Don't write into the indirect grid at all when in combined lighting mode
            const int componentIdx = gridIdx % 2;
            if (m_params.lightingMode == kBakeLightingCombined && componentIdx == kLightProbeBufferIndirect) { continue; }
            
            vec4* accumBuffer = &(*m_objects.cu_accumBuffers[gridIdx])[emplacedRay.accumIdx * m_params.grid.coefficientsPerProbe];
            const float weight = !isAlive;

            // Should we accumulate this sample?
            bool accumulate = false;
            if(cwiseMax(L) > 1e-10f && incidentRay.depth >= m_params.camera.overrides.minDepth)
            {
                switch (componentIdx)
                {
                case kLightProbeBufferDirect:
                    accumulate = m_params.lightingMode == kBakeLightingCombined || incidentRay.depth <= 1;
                    break;
                case kLightProbeBufferIndirect:
                    accumulate = m_params.lightingMode == kBakeLightingSeparated && incidentRay.depth > 1;
                    break;
                default:
                    assert(false);
                }

                // Half buffers only get every second sample
                if (gridIdx > kLightProbeBufferIndirect) { accumulate &= emplacedRay.sampleIdx % 2 == 1; }
            }

            if(accumulate)  
            {              
                // Project and accumulate the SH coefficients
                for (int shIdx = 0; shIdx < m_params.grid.shCoefficientsPerProbe; ++shIdx)
                {
                    accumBuffer[shIdx] += vec4(L * SH::Project(ctx.emplacedRay[0].probeDir, shIdx) * kFourPi, weight);
                }
            }
            else if(!isAlive)
            {
                // Just increment the weights if they're non-zero
                for (int shIdx = 0; shIdx < m_params.grid.shCoefficientsPerProbe; ++shIdx) { accumBuffer[shIdx][3] += weight; }
            }

            // Accumulate validity and mean distance       
            if (incidentRay.IsIndirectSample() && incidentRay.depth == 1)
            {                
                // A probe sample is valid if, on the first hit, it intersects with a front-facing surface or it leaves the scene
                accumBuffer[m_params.grid.shCoefficientsPerProbe] += vec4(float(!hitCtx.isValid || !hitCtx.backfacing),
                                              //1.0f / max(1e-10f, incidentRay.tNear),
                                              //min(m_params.grid.transform.scale().x, incidentRay.tNear) / m_params.grid.transform.scale().x,
                                              0.0f, 0.0f, 1.0f);
            }
        }
    }
    __device__ void Device::LightProbeCamera::ReduceAccumulatedSample(vec4& dest, const vec4& source)
    {              
        if (int(dest.w) >= m_params.grid.minMaxSamplesPerProbe.y - 1) { return; }       
        
        if (int(dest.w + source.w) < m_params.grid.minMaxSamplesPerProbe.y)
        {           
            dest += source;
            return;
        }

        dest += source * (m_params.grid.minMaxSamplesPerProbe.y - dest.w) / source.w;
    }

    __device__ void Device::LightProbeCamera::ReduceAccumulationBuffer(Device::Array<vec4>* cu_accumBuffer, Device::LightProbeGrid* cu_probeGrid, const uint batchSize, const uvec2 batchRange)
    {         
        if (kKernelIdx >= m_params.totalBuckets) { return; }

        assert(cu_accumBuffer);
        assert(cu_probeGrid);
        assert(m_objects.cu_reduceBuffer);

        //if (batchRange[0] == batchSize) (*m_objects.cu_reduceBuffer)[kKernelIdx] = 0.0f;

        auto& accumBuffer = *cu_accumBuffer;
        auto& reduceBuffer = *m_objects.cu_reduceBuffer;
        
        const int probeIdx = kKernelIdx / m_params.bucketsPerProbe;
        const int probeSubsampleIdx = (kKernelIdx / m_params.grid.coefficientsPerProbe) % m_params.subprobesPerProbe;
        const int coeffIdx = kKernelIdx % m_params.grid.coefficientsPerProbe;

        for (uint iterationSize = batchRange[0] / 2; iterationSize > batchRange[1] / 2; iterationSize >>= 1)
        {
            if (probeSubsampleIdx < iterationSize)
            {
                // For the first iteration, copy the data out of the accumulation buffer
                if (iterationSize == batchSize / 2)
                {
                    auto& texel = reduceBuffer[kKernelIdx];
                    texel = 0.0f;
                    ReduceAccumulatedSample(texel, accumBuffer[kKernelIdx]);

                    if (probeSubsampleIdx + iterationSize < m_params.subprobesPerProbe)
                    {
                        assert(kKernelIdx + iterationSize * m_params.grid.coefficientsPerProbe < kAccumBufferSize);
                        //if (probeIdx == 0 && coeffIdx == 0) { printf("%i: %f + %f = %f\n", iterationSize, texel.w, accumBuffer[kKernelIdx + iterationSize].w, texel.w + accumBuffer[kKernelIdx + iterationSize].w); }
                        ReduceAccumulatedSample(texel, accumBuffer[kKernelIdx + iterationSize * m_params.grid.coefficientsPerProbe]);

                    }
                    //else
                    //    if (probeIdx == 0 && coeffIdx == 0) { printf("%i: %f\n", iterationSize, texel.w); }
                }
                else
                {
                    assert(kKernelIdx + iterationSize * m_params.grid.coefficientsPerProbe < kAccumBufferSize);
                    assert(probeSubsampleIdx + iterationSize < m_params.subprobesPerProbe);
                    //if (probeIdx == 0 && coeffIdx == 0) { printf("%i: %f + %f = %f\n", iterationSize, reduceBuffer[kKernelIdx].w, reduceBuffer[kKernelIdx + iterationSize].w, reduceBuffer[kKernelIdx].w + reduceBuffer[kKernelIdx + iterationSize].w); }
                    ReduceAccumulatedSample(reduceBuffer[kKernelIdx], reduceBuffer[kKernelIdx + iterationSize * m_params.grid.coefficientsPerProbe]);
                }
            }
            else
            {
                //reduceBuffer[kKernelIdx] = 1.0f;
            }

            __syncthreads();
        } 

        // After the last operation, cache the accumulated value in the probe grid
        if (probeSubsampleIdx == 0 && batchRange[0] == 2)
        {
            auto& texel = reduceBuffer[kKernelIdx];         

            //const int probeIdx = kKernelIdx / m_params.bucketsPerProbe;
            //const int coeffIdx = (kKernelIdx / m_params.bucketsPerCoefficient) % m_params.grid.coefficientsPerProbe;
            if (coeffIdx == m_params.grid.shCoefficientsPerProbe)
            {
               const float norm = max(1.0f, texel.w);

               texel[kProbeValidity] /= norm;                   // Probe validity
               //texel.y = norm / max(1e-10f, texel.y);         // Harmonic mean distance
               //texel.y /= norm;                               // Geometric mean distance
               texel[kProbeFilterWeights] = 1.0f;
               texel[kProbeNumSamples] = texel.w;               // Store the total number of samples
            }
            else
            {
                texel /= max(1.0f, texel.w);
            }

            cu_probeGrid->SetSHCoefficient(probeIdx, coeffIdx, texel.xyz);
        }
    }

    __device__ void Device::LightProbeCamera::BuildLightProbeErrorGrid()
    {
        if (kKernelIdx >= m_params.grid.numProbes) { return; }
        assert(m_objects.cu_filteredProbeGrids[0]);
        assert(m_params.grid.shOrder > 0);

        const vec3* PA = m_objects.cu_filteredProbeGrids[0]->At(kKernelIdx);
        const vec3* PB = m_objects.cu_filteredProbeGrids[1]->At(kKernelIdx);
        const vec3* PAHalf = m_objects.cu_filteredProbeGrids[2]->At(kKernelIdx);
        const vec3* PBHalf = m_objects.cu_filteredProbeGrids[3]->At(kKernelIdx);

        // Square error should be down-weighted according to the reciprocal of the sum of the filter weights
        const vec2 weights[2] = { vec2(1.0f),
                                  vec2(1 / PA[m_params.grid.shCoefficientsPerProbe][kProbeFilterWeights],
                                       1 / PB[m_params.grid.shCoefficientsPerProbe][kProbeFilterWeights]) };

        // Record data in the grid
        vec2& probe = (*m_objects.cu_lightProbeErrorGrids[0])[kKernelIdx];
        probe = 0.0f;
        
        // Go channel by channel to find the peak irradiance
        for (int chnlIdx = 0; chnlIdx < 3; ++chnlIdx)
        {
            // First pass is irradiance, second pass is square error. 
            for (int passIdx = 0; passIdx < 2; ++passIdx)
            {                
                // Load the coefficients from the direct map
                float L0 = PA[0][chnlIdx] * weights[passIdx][0];
                float L0Half = PAHalf[0][chnlIdx] * weights[passIdx][0];
                vec3 L1 = vec3(PA[1][chnlIdx], PA[2][chnlIdx], PA[3][chnlIdx]) * weights[passIdx][0];
                vec3 L1Half = vec3(PAHalf[1][chnlIdx], PAHalf[2][chnlIdx], PAHalf[3][chnlIdx]) * weights[passIdx][0];

                // If we're in separated mode, load the coefficients from the indirect map
                if (m_params.lightingMode == kBakeLightingSeparated)
                {
                    L0 += PB[0][chnlIdx] * weights[passIdx][1];
                    L1 += vec3(PB[1][chnlIdx], PB[2][chnlIdx], PB[3][chnlIdx]) * weights[passIdx][1];
                    L0Half += PBHalf[0][chnlIdx] * weights[passIdx][1];
                    L1Half += vec3(PBHalf[1][chnlIdx], PBHalf[2][chnlIdx], PBHalf[3][chnlIdx]) * weights[passIdx][1];
                }

                // Estimate the peak irradiance over the unit sphere
                float M = max(0.0f, L0Half * SH::Legendre(0) + length(L1Half) * SH::Legendre(1));
                float N = max(0.0f, (L0 - L0Half) * SH::Legendre(0) + (length(L1) - length(L1Half)) * SH::Legendre(1));

                // Gamma ramp
                if (m_params.camera.adaptiveSamplingGamma != 1.0f)
                {
                    M = powf(M, 1 / m_params.camera.adaptiveSamplingGamma);
                    N = powf(N, 1 / m_params.camera.adaptiveSamplingGamma);
                }
                
                // Update the peak irradiance and error over all channels
                probe[passIdx] = max(probe[passIdx], 
                                     (passIdx == 0) ? ((M + N) * 0.5f) : (sqr(M - N) * 2.0f));                
            }
        } 

        //printf("[%i: %f %f] ", kKernelIdx, probe.x, probe.y);
    }

    __device__ void Device::LightProbeCamera::DilateLightProbeErrorGrid()
    {
        if (kKernelIdx >= m_params.grid.numProbes) { return; }
        assert(m_objects.cu_lightProbeErrorGrids[0]);

        // Create validity and edge masks to save time later on.
        const ivec3 gridPos0 = GridPosFromProbeIdx(kKernelIdx, m_params.grid.gridDensity);
        vec2 peakProbe(0.0f);
        for (int z = -1, idx = 0; z <= 1; z++)
        {
            for (int y = -1; y <= 1; y++)
            {
                for (int x = -1; x <= 1; x++, idx++)
                {
                    const ivec3 gridPosK = gridPos0 + ivec3(x, y, z);
                    if (gridPosK.x < 0 || gridPosK.x >= m_params.grid.gridDensity.x ||
                        gridPosK.y < 0 || gridPosK.y >= m_params.grid.gridDensity.y ||
                        gridPosK.z < 0 || gridPosK.z >= m_params.grid.gridDensity.z)
                    {                
                        continue;
                    }
    
                    // The dilated error is simply the maximum value of its neighbours
                    const auto& probe = (*m_objects.cu_lightProbeErrorGrids[0])[ProbeIdxFromGridPos(gridPosK, m_params.grid.gridDensity)];
                    if (probe.y > peakProbe.y)
                    {
                        peakProbe = probe;
                    }
                }
            }
        }

        (*m_objects.cu_lightProbeErrorGrids[1])[kKernelIdx] = peakProbe;
    }

    __device__ void Device::LightProbeCamera::ReduceLightProbeErrorData(LightProbeCameraAggregateStatistics& stats)
    {
        assert(m_objects.cu_lightProbeErrorGrids[0]);
        assert(m_objects.cu_convergenceGrid);
        assert(m_objects.cu_meanI);
        
        constexpr float kMinMSENorm = 1e-10f;
        
        __shared__ int localNumActiveProbes[256], numActiveProbes;
        __shared__ vec2 localI[256], I;

        const int startIdx = m_params.grid.numProbes * kKernelIdx / 256;
        const int endIdx = m_params.grid.numProbes * (kKernelIdx + 1) / 256;

        // Sum peak irradiance over the grid
        localNumActiveProbes[kKernelIdx] = 0;
        localI[kKernelIdx] = vec2(0.0f);
        for (int idx = startIdx; idx < endIdx; idx++)
        {          
            localI[kKernelIdx] += (*m_objects.cu_lightProbeErrorGrids[0])[idx];
        }
        localI[kKernelIdx] /= max(1, endIdx - startIdx);

        __syncthreads();

        if (kKernelIdx == 0)
        {       
            I = vec2(0.0f);
            for (int idx = 0; idx < 256; idx++) { I += localI[idx]; }
            I /= 256.0f;

            // Update the MSE value and normalise it if necessary
            stats.error.meanI =  I.x;
            *m_objects.cu_meanI = I.x;
            stats.error.MSE = I.y;
            if (m_params.camera.samplingMode == kCameraSamplingAdaptiveRelative)
            {
                stats.error.MSE /= max(kMinMSENorm, I.x);
            }
        }

        __syncthreads();

        // Populate the adaptive sampling grid
        for (int i = startIdx; i < endIdx; i++)
        {
            // Relative mode uses the mean probe irradiance as its normalisation factor. 
            // We multiply this by a factor of two assuming irradiance values are evently distributed between N and 0, so
            // the mean is half the value we need to normalise by.
            float sqrError = (*m_objects.cu_lightProbeErrorGrids[0])[i].y;
            if (m_params.camera.samplingMode == kCameraSamplingAdaptiveRelative)
            {
                sqrError /= max(kMinMSENorm, sqr(I.x * 2.0f));
            }

            uchar convergenceFlags = 0;
            const int sampleCount = (*m_objects.cu_probeGrids)[kLightProbeBufferDirect].At(i)[m_params.grid.shCoefficientsPerProbe][kProbeNumSamples];
            
            // If the probe is below the minimum sample count, flag it and mark as active
            if (sampleCount < m_params.camera.minMaxSamples.x)
            {
                convergenceFlags |= kProbeBelowSampleMin;
            }
            // Otherwise, if the probe has reached the maximum number of samples, mark it as converged
            else if (sampleCount >= m_params.camera.minMaxSamples.y)
            {
                convergenceFlags |= kProbeAtSampleMax;
            }
            
            // If the probe is still below the error threshold, mark is active
            if (sqrError > sqr(m_params.camera.errorThreshold)) 
            { 
                convergenceFlags |= kProbeUnconverged; 
            }
            
            // Set the entry in the adaptive sampling grid and accumulate the number of active probes
            (*m_objects.cu_convergenceGrid)[i] = convergenceFlags;
            localNumActiveProbes[kKernelIdx] += uchar((convergenceFlags & (kProbeUnconverged | kProbeBelowSampleMin)) != 0 && !(convergenceFlags & kProbeAtSampleMax));
        }

        __syncthreads();

        if (kKernelIdx == 0)
        {
            int numActiveProbes = 0;
            for (int idx = 0; idx < 256; idx++) { numActiveProbes += localNumActiveProbes[idx]; }
            
            stats.bake.probesConverged = 1.0f - numActiveProbes / float(m_params.grid.numProbes);
        }
    }

    __host__ Host::LightProbeCamera::LightProbeCamera(const ::Json::Node& node, const std::string& id) :
        Host::Camera(node, id, kRayBufferSize),
        m_block(16 * 16, 1, 1),
        m_seedGrid(1, 1, 1),
        m_reduceGrid(1, 1, 1),
        m_hostMeanI(1.0f),
        m_needsRebind(false)
    {
        // Register events for deligates to watch
        RegisterEvent("OnBuildGrids");

        // TODO: This is to maintain backwards compatibility. Deprecate it when no longer required.
        m_gridIDs[0] = "grid_noisy_direct";
        m_gridIDs[1] = "grid_noisy_indirect";

        // Output grid IDs
        node.GetValue("gridDirectID", m_gridIDs[0], Json::kRequiredAssert | Json::kNotBlank);
        node.GetValue("gridIndirectID", m_gridIDs[1], Json::kRequiredAssert | Json::kNotBlank);
        node.GetValue("gridDirectHalfID", m_gridIDs[2], Json::kRequiredAssert | Json::kNotBlank);
        node.GetValue("gridIndirectHalfID", m_gridIDs[3], Json::kRequiredAssert | Json::kNotBlank);
        
        // Input grid IDs for adaptive sampling
        node.GetValue("gridFilteredDirectID", m_filteredGridIDs[0], Json::kNotBlank);
        node.GetValue("gridFilteredIndirectID", m_filteredGridIDs[1], Json::kNotBlank);
        node.GetValue("gridFilteredDirectHalfID", m_filteredGridIDs[2], Json::kNotBlank);
        node.GetValue("gridFilteredIndirectHalfID", m_filteredGridIDs[3], Json::kNotBlank);

        // Create reduction and adaptive sampling buffers
        m_hostReduceBuffer = AssetHandle<Host::Array<vec4>>(tfm::format("%s_probeReduceBuffer", id), kAccumBufferSize, m_hostStream);
        m_hostIndirectionBuffer = AssetHandle<Host::Array<uint>>(tfm::format("%s_indirectionBuffer", id), kAccumBufferSize, m_hostStream);
        m_hostLightProbeErrorGrids[0] = AssetHandle<Host::Array<vec2>>(tfm::format("%s_probeErrorGrids0", id), kAccumBufferSize, m_hostStream);
        m_hostLightProbeErrorGrids[1] = AssetHandle<Host::Array<vec2>>(tfm::format("%s_probeErrorGrids1", id), kAccumBufferSize, m_hostStream);
        m_hostConvergenceGrid = AssetHandle<Host::Array<uchar>>(tfm::format("%s_adaptiveSamplingGrid", id), kAccumBufferSize, m_hostStream);

        // Instantiate the camera object on the device
        cu_deviceData = InstantiateOnDevice<Device::LightProbeCamera>();

        // Create the accumulation buffers and probe grids
        for (int idx = 0; idx < m_hostAccumBuffers.size(); ++idx)
        {
            // Don't create grids that don't have IDs
            Assert(!m_gridIDs[idx].empty());

            m_hostAccumBuffers[idx] = AssetHandle<Host::Array<vec4>>(tfm::format("%s_probeAccumBuffer%i", id, idx), kAccumBufferSize, m_hostStream);
            m_hostAccumBuffers[idx]->Clear(vec4(0.0f));
            m_deviceObjects.cu_accumBuffers[idx] = m_hostAccumBuffers[idx]->GetDeviceInstance();

            // Create the probe grid objects and attach external buffers to them
            m_hostLightProbeGrids[idx] = AssetHandle<Host::LightProbeGrid>(m_gridIDs[idx], m_gridIDs[idx]);
            m_hostLightProbeGrids[idx]->SetExternalBuffers(m_hostConvergenceGrid, m_hostLightProbeErrorGrids[0], m_hostMeanI);

            // Update the device pointers. Adaptive sampling grids might be updated again during the binding stage. 
            m_deviceObjects.cu_probeGrids[idx] = m_hostLightProbeGrids[idx]->GetDeviceInstance();
            m_deviceObjects.cu_filteredProbeGrids[idx] = m_deviceObjects.cu_probeGrids[idx];
        }

        // Sychronise the device objects
        m_deviceObjects.cu_reduceBuffer = m_hostReduceBuffer->GetDeviceInstance();
        m_deviceObjects.renderState.cu_compressedRayBuffer = m_hostCompressedRayBuffer->GetDeviceInstance();
        m_deviceObjects.cu_indirectionBuffer = m_hostIndirectionBuffer->GetDeviceInstance();
        m_deviceObjects.renderState.cu_blockRayOccupancy = m_hostBlockRayOccupancy->GetDeviceInstance();
        m_deviceObjects.renderState.cu_renderStats = m_hostRenderStats->GetDeviceInstance();
        m_deviceObjects.cu_lightProbeErrorGrids[0] = m_hostLightProbeErrorGrids[0]->GetDeviceInstance();
        m_deviceObjects.cu_lightProbeErrorGrids[1] = m_hostLightProbeErrorGrids[1]->GetDeviceInstance();
        m_deviceObjects.cu_convergenceGrid = m_hostConvergenceGrid->GetDeviceInstance();
        m_deviceObjects.cu_meanI = m_hostMeanI.GetDeviceInstance();

        // Objects are re-synchronised at every JSON update
        FromJson(node, ::Json::kRequiredWarn);
    }

    __host__ AssetHandle<Host::RenderObject> Host::LightProbeCamera::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kCamera) { return AssetHandle<Host::RenderObject>(); }

        return AssetHandle<Host::RenderObject>(new Host::LightProbeCamera(json, id), id);
    }

    __host__ void Host::LightProbeCamera::OnDestroyAsset()
    {
        Host::Camera::OnDestroyAsset();

        // Destroy the light probe grids 
        for (auto& grid : m_hostLightProbeGrids) { grid.DestroyAsset(); }

        // Destroy the rest of the objects
        for (auto& accumBuffer : m_hostAccumBuffers) { accumBuffer.DestroyAsset(); }
        for (auto& grid : m_hostLightProbeErrorGrids) { grid.DestroyAsset();  }
        m_hostReduceBuffer.DestroyAsset();
        m_hostConvergenceGrid.DestroyAsset();

        DestroyOnDevice(cu_deviceData);
    }

    __host__ std::vector<AssetHandle<Host::RenderObject>> Host::LightProbeCamera::GetChildObjectHandles()
    {
        std::vector<AssetHandle<Host::RenderObject>> children;
        for (auto& grid : m_hostLightProbeGrids)
        {
            if (grid) { children.push_back(AssetHandle<Host::RenderObject>(grid)); }
        }
        return children;
    }

    __host__ void Host::LightProbeCamera::FromJson(const ::Json::Node& parentNode, const uint flags)
    {       
        Prepare(LightProbeCameraParams(parentNode, flags));
    }

    __host__ void Host::LightProbeCamera::GenerateHilbertBuffer(const LightProbeCameraParams& newParams)
    {
        if (newParams.grid.gridDensity == m_params.grid.gridDensity &&
            newParams.traversalMode == m_params.traversalMode) { return; }

        if (newParams.grid.numProbes == 0)
        {
            m_hostIndirectionBuffer->Resize(0);
            return;
        }        
         
        /*
            Symbol table for Lindenmayer system for 3D Hilbert curve
            0 = Rotate 90 anticlockwise around X axis
            1 = Rotate 90 clockwise around X axis
            2 = Rotate 90 anticlockwise around Y axis,
            ....
            * = Recurse
            + = Forward 1 unit
        */

        static const std::string LSystem = "24*+24*+*1+255*+*3+055*+*1+5*15";

        std::array<int, 36> transformLUT;
        std::array<ivec3, 6> directionLUT;
        static const std::array<imat3, 6> rotMat = 
        {
            imat3(ivec3(0, 1, 0), ivec3(-1, 0, 0), ivec3(0, 0, 1)),
            imat3(ivec3(0, -1, 0), ivec3(1, 0, 0), ivec3(0, 0, 1)),
            imat3(ivec3(0, 0, 1), ivec3(0, 1, 0), ivec3(-1, 0, 0)),
            imat3(ivec3(0, 0, -1), ivec3(0, 1, 0), ivec3(1, 0, 0)),
            imat3(ivec3(1, 0, 0), ivec3(0, 0, 1), ivec3(0, -1, 0)),
            imat3(ivec3(1, 0, 0), ivec3(0, 0, -1), ivec3(0, 1, 0))
        }; 

        std::vector<uint> hilbertIndices;
        std::vector<int> LStack;

        // Reserve some space for the indirection buffer
        hilbertIndices.reserve(newParams.grid.numProbes);
        hilbertIndices.push_back(0);
        LStack.push_back(0);

        // Compute the size of the Hilbert cube that will completely enclose the probe grid
        int hilbertSize;
        int numIterations = 1;
        for (hilbertSize = 2; hilbertSize < cwiseMax(newParams.grid.gridDensity); hilbertSize <<= 1, ++numIterations) {}
        Assert(numIterations < 8); // Sanity check       

        ivec3 turtleP(0);
        imat3 turtleM = imat3::Indentity();

        std::vector<uchar> checksum(newParams.grid.numProbes, 0);

        while (!LStack.empty())
        {
            const char L = LSystem[LStack.back()++];
            switch (L)
            {
            case '*':
                // Recurse by pushing a new rule onto the stack
                if (LStack.size() < numIterations)
                {
                    LStack.push_back(0);
                }
                break;
            case '+':
                // Increment the position by point unit in the current direction
                turtleP += ivec3(turtleM[0][0], -turtleM[1][0], -turtleM[2][0]);

                AssertMsgFmt(!(turtleP.x < 0 || turtleP.x >= hilbertSize || turtleP.y < 0 || turtleP.y >= hilbertSize || turtleP.z < 0 || turtleP.z >= hilbertSize),
                    "Turtle went out of bounds: %s", turtleP.format().c_str());

                if (turtleP.x < newParams.grid.gridDensity.x && turtleP.y < newParams.grid.gridDensity.y && turtleP.z < newParams.grid.gridDensity.z)
                {
                    // Push the index of this voxel into the indirection buffer
                    hilbertIndices.push_back(newParams.grid.gridDensity.x * (turtleP.z * newParams.grid.gridDensity.y + turtleP.y) + turtleP.x);
                    checksum[hilbertIndices.back()]++;
                }
                break;

            default:
                // Transform the turtleP direction
                turtleM = turtleM * rotMat[int(L) - int('0')];
            }

            // If we've reached the end of the rule, pop it off the stack
            if (LStack.back() == LSystem.length()) { LStack.pop_back(); }
        }        

        AssertMsgFmt(hilbertIndices.size() == newParams.grid.numProbes, "Size mismatch: %i -> %i", hilbertIndices.size(), newParams.grid.numProbes); // Sanity check

        // Diagnostics
        /*int count[3] = { 0, 0, 0 };
        for (int i = 0; i < checksum.size(); ++i)
        {
            count[min(2, int(checksum[i]))]++;
        }
        Log::Error("%i -> %i, %i, %i", checksum.size(), count[0], count[1], count[2]);*/

        // Upload the indices to the device
        m_hostIndirectionBuffer->Upload(hilbertIndices);

    }

    __host__ void Host::LightProbeCamera::Prepare(LightProbeCameraParams newParams)
    {
        newParams.grid.Prepare();

        // Reduce the size of the grid if it exceeds the size of the accumulation buffer
        const int maxNumProbes = min(kAccumBufferSize / newParams.grid.coefficientsPerProbe, kRayBufferNumBuckets);
        if (Volume(newParams.grid.gridDensity) > maxNumProbes)
        {
            const auto oldDensity = newParams.grid.gridDensity;
            while (Volume(newParams.grid.gridDensity) > maxNumProbes)
            {
                newParams.grid.gridDensity = max(ivec3(1), newParams.grid.gridDensity - ivec3(1));
            }
            Log::Error("WARNING: The size of the probe grid %s is too large for the accumulation buffer. Reducing to %s.\n", oldDensity.format(), newParams.grid.gridDensity.format());
        }

        // Prepare the light probe grids with the new parameters
        newParams.grid.camera = newParams.camera;
        int q = 0;
        for (auto& grid : m_hostLightProbeGrids)
        {
            if (grid) { grid->Prepare(newParams.grid); }
        }
        for (auto& grid : m_hostFilteredLightProbeGrids)
        {
            if (grid) { grid->SetOutputMode(newParams.grid.outputMode); }
        }

        // Number of light probes in the grid
        newParams.grid.numProbes = Volume(newParams.grid.gridDensity);
        // Number of SH parameter sets per probe, reduced later to get the final value 
        newParams.subprobesPerProbe = min(kRayBufferNumBuckets / newParams.grid.numProbes,
            kAccumBufferSize / (newParams.grid.numProbes * newParams.grid.coefficientsPerProbe));

        // The minimum and maximum number of samples per bucket based on the number of buckets per coefficient
        newParams.grid.minMaxSamplesPerProbe = newParams.camera.minMaxSamples;
        newParams.minMaxSamplesPerSubprobe = ivec2(vec2(1.0f) + vec2(newParams.camera.minMaxSamples) / vec2(newParams.subprobesPerProbe));
        if (newParams.camera.minMaxSamples.x <= 0) { newParams.minMaxSamplesPerSubprobe.x = newParams.grid.minMaxSamplesPerProbe.x = 0; }
        if (newParams.camera.minMaxSamples.y <= 0) { newParams.minMaxSamplesPerSubprobe.y = newParams.grid.minMaxSamplesPerProbe.y = std::numeric_limits<int>::max(); }

        // Derive some more values
        newParams.bucketsPerProbe = newParams.subprobesPerProbe * newParams.grid.coefficientsPerProbe;
        newParams.totalBuckets = newParams.bucketsPerProbe * newParams.grid.numProbes;
        newParams.totalSubprobes = newParams.subprobesPerProbe * newParams.grid.numProbes;

        // Used when parallel reducing the accumluation buffer
        uint reduceBatchSizePow2 = NearestPow2Ceil(newParams.subprobesPerProbe);

        Log::Debug("Light probe camera buffer layout:");
        {
            Log::Indent indent;
            Log::Debug("coefficientsPerProbe: %i", newParams.grid.coefficientsPerProbe);
            Log::Debug("numProbes: %i", newParams.grid.numProbes);
            Log::Debug("subprobesPerProbe: %i", newParams.subprobesPerProbe);
            Log::Debug("Buckets:");
            Log::Debug("  Per probe: %i", newParams.bucketsPerProbe);
            Log::Debug("  Total: %i", newParams.totalBuckets);
            Log::Debug("Min/max samples:");
            Log::Debug("  Per probe: %s", newParams.grid.minMaxSamplesPerProbe.format());
            Log::Debug("  Per sub-probe: %s", newParams.minMaxSamplesPerSubprobe.format());
        }

        // Update the Hilbert indirect buffer if required
        GenerateHilbertBuffer(newParams);

        // Update the camera params object with the new params
        m_params = newParams;

        // Sync everything with the device
        SynchroniseObjects(cu_deviceData, m_deviceObjects);
        SynchroniseObjects(cu_deviceData, m_params);

        const int seedGridSize = m_params.totalSubprobes;
        m_seedGrid = dim3((seedGridSize + (m_block.x - 1)) / m_block.x, 1, 1);
        const int reduceGridSize = m_params.totalBuckets;
        m_reduceGrid = dim3((reduceGridSize + (m_block.x - 1)) / m_block.x, 1, 1);

        m_frameIdx = 0;
        m_needsRebind = true;
    }

    __host__ void Host::LightProbeCamera::ClearRenderState()
    {
        for (auto& accumBuffer : m_hostAccumBuffers)
        {
            if (accumBuffer)
            {
                accumBuffer->Clear(vec4(0.0f));
            }
        }
        m_hostCompressedRayBuffer->Clear(Cuda::CompressedRay());
        m_hostConvergenceGrid->Clear(1);
        *m_aggregateStats = LightProbeCameraAggregateStatistics();
    }

    __global__ void KernelSeedRayBuffer(Device::LightProbeCamera* camera, const int frameIdx)
    {
        camera->SeedRayBuffer(frameIdx);
    }

    __host__ void Host::LightProbeCamera::OnPreRenderPass(const float wallTime, const uint frameIdx)
    {
        m_frameIdx = frameIdx;

        KernelSeedRayBuffer << < m_seedGrid, m_block, 0, m_hostStream >> > (cu_deviceData, frameIdx);
    }

    __global__ void KernelComposite(Device::ImageRGBA* deviceOutputImage, const Device::LightProbeCamera* camera)
    {
        //if (*(deviceOutputImage->AccessSignal()) != kImageWriteLocked) { return; }

        camera->Composite(kKernelPos<ivec2>(), deviceOutputImage);
    }

    __host__ void Host::LightProbeCamera::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage) const
    {
        dim3 blockSize = dim3(16, 16, 1);
        dim3 gridSize(kAccumBufferWidth / 16, kAccumBufferHeight / 16, 1);

        hostOutputImage->SignalSetWrite(m_hostStream);
        KernelComposite << < gridSize, blockSize, 0, m_hostStream >> > (hostOutputImage->GetDeviceInstance(), cu_deviceData);
        hostOutputImage->SignalUnsetWrite(m_hostStream);
    }

    __global__ void KernelReduceAccumulationBuffer(Device::LightProbeCamera* camera, Device::Array<vec4>* cu_accumBuffer, Device::LightProbeGrid* cu_probeGrid,
        const uint reduceBatchSize, const uvec2 batchRange)
    {
        camera->ReduceAccumulationBuffer(cu_accumBuffer, cu_probeGrid, reduceBatchSize, batchRange);
    }

    __global__ void KernelReduceLightProbeErrorData(Device::LightProbeCamera* cu_camera, LightProbeCameraAggregateStatistics* cu_stats)
    {
        cu_camera->ReduceLightProbeErrorData(*cu_stats);
    }

    __host__ void Host::LightProbeCamera::UpdateProbeGridAggregateStatistics()
    {       
        // Reset the stats
        m_aggregateStats = LightProbeCameraAggregateStatistics();
        auto& stats = *m_aggregateStats;
        
        // If we're using adaptive sampling to monitor convergence, construct the grid
        if (m_params.camera.samplingMode != kCameraSamplingFixed)
        {
            // Reduce the adaptive sampling data to find the total number of converged probes
            KernelReduceLightProbeErrorData << <1, 256, 0, m_hostStream >> > (cu_deviceData, m_aggregateStats.GetDeviceInstance());
            IsOk(hipStreamSynchronize(m_hostStream));

            // Sync the host copy 
            m_aggregateStats.Download();
        }

        // Update aggreate stats from each light probe grid in turn. We'll only refer to one grid here, but other routines will refer to the data as well.
        for (int gridIdx = 0; gridIdx < kLightProbeNumBuffers; ++gridIdx)
        {
            Assert(m_hostLightProbeGrids[gridIdx]);
            m_hostLightProbeGrids[gridIdx]->UpdateAggregateStatistics(m_params.grid.minMaxSamplesPerProbe.y);
        }

        // Get a handle to a single grid which will act as the representitive 
        // NOTE: Always prefer whichever grid gets the full complement of indirect energy. This is a hedge against future refactoring but it may need to be updated.
        auto& grid = (m_params.lightingMode == kBakeLightingCombined) ? m_hostLightProbeGrids[0]->GetAggregateStatistics() : m_hostLightProbeGrids[1]->GetAggregateStatistics();
        stats.minMaxSamples = grid.minMaxSamples; 
        stats.meanSamples = grid.meanSamples;
        stats.meanValidity = grid.meanValidity;
        stats.meanDistance = grid.meanDistance;

        // If an upper limit is set on the number of samples, calculate the proportion of probes which have reached it
        if (m_params.camera.minMaxSamples.y > 0)
        {
            stats.bake.probesFull = clamp(std::ceil(grid.minMaxSamples.x) / float(m_params.grid.minMaxSamplesPerProbe.y), 0.0f, 1.0f);
        }     

        // Fixed sampling modes use the total number of full probes to determine convergence. Adaptive sampling uses the error-driven matric.
        stats.bake.progress = (m_params.camera.samplingMode == kCameraSamplingFixed) ? stats.bake.probesFull : stats.bake.probesConverged;
    }

    __host__ const LightProbeCameraAggregateStatistics& Host::LightProbeCamera::PollBakeProgress()
    {
        // FIXME: This is a horrible hack to prevent having to manually scan the accumulation buffer every frame.
        /*if (m_frameIdx < m_params.maxSamplesPerProbe)
        {
            return clamp(m_frameIdx / float(m_params.maxSamplesPerProbe), 0.0f, 1.0f);
        }*/

        if ((m_frameIdx - 2) % m_params.gridUpdateInterval == 0)
        {
            UpdateProbeGridAggregateStatistics();
        }

        return *m_aggregateStats;
    }

    __host__ bool Host::LightProbeCamera::ExportProbeGrid(const LightProbeGridExportParams& params)
    {
        // Recompile the grids to make sure everything is included in the export
        Compile();

        for (int gridIdx = 0; gridIdx <= kLightProbeBufferIndirect; ++gridIdx)
        {
            // Don't write out indirect when running in combined mode
            if (m_params.lightingMode == kBakeLightingCombined && gridIdx == kLightProbeBufferIndirect) { continue; }

            // Only write grids that have valid paths associated with them
            if (gridIdx >= params.exportPaths.size()) { continue; }

            // If the validity is outside the valid range, all grids will be similarly invalid so bail immediately
            const auto& stats = m_hostLightProbeGrids[gridIdx]->GetAggregateStatistics();
            if (stats.meanValidity < params.minGridValidity || stats.meanValidity > params.maxGridValidity)
            {
                Log::Warning("Warning: Cannot export %s probe grid. Mean validity %f is outside valid range [%f, %f]", 
                    (gridIdx == 0) ? "direct" : "indirect", stats.meanValidity, params.minGridValidity, params.maxGridValidity);
                break;
            }

            // Only export to USD if explicitly flagged to do so
            if (params.isArmed)
            {
                Log::Debug("Exporting to '%s'...\n", params.exportPaths[gridIdx]);
                try
                {
                    // Get a handle to the filtered or unfiltered grid and its stats
                    const auto& grid = m_params.filterGrids ? m_hostFilteredLightProbeGrids[gridIdx] : m_hostLightProbeGrids[gridIdx];
                    USDIO::ExportLightProbeGrid(grid, m_params.grid, params.exportPaths[gridIdx], USDIO::SHPackingFormat::kUnity);
                }
                catch (const std::runtime_error& err)
                {
                    Log::Error("Error: %s\n", err.what());
                }
            }
            else
            {
                Log::Warning("Warning: Skipped USD export to '%s' because setting was not enabled.\n", params.exportPaths[gridIdx]);
                break;
            }
        }

        return true;
    }

    __host__ void Host::LightProbeCamera::SetLightProbeCameraParams(const LightProbeCameraParams& newParams)
    {
        Prepare(newParams);
    }

    __host__ void Host::LightProbeCamera::BuildLightProbeGrids()
    {
        // Used when parallel reducing the accumluation buffer
        uint reduceBatchSizePow2 = NearestPow2Ceil(m_params.subprobesPerProbe);

        for (int gridIdx = 0; gridIdx < kLightProbeNumBuffers; ++gridIdx)
        {
            if (!m_hostLightProbeGrids[gridIdx]) { continue; }

            // Indirect buffer isn't used when running in combined mode
            if (m_params.lightingMode == kBakeLightingCombined && gridIdx == kLightProbeBufferIndirect) { continue; }

            auto& grid = *m_hostLightProbeGrids[gridIdx];

            // Reduce until the batch range is equal to the size of the block
            uint batchSize = reduceBatchSizePow2;
            while (batchSize > 1)
            {
                KernelReduceAccumulationBuffer << < m_reduceGrid, m_block, 0, m_hostStream >> > (cu_deviceData, m_deviceObjects.cu_accumBuffers[gridIdx],
                    m_deviceObjects.cu_probeGrids[gridIdx],
                    reduceBatchSizePow2, uvec2(batchSize, batchSize >> 1));
                batchSize >>= 1;
            }
            // Reduce the block in a single operation
            //KernelReduceAccumulationBuffer << < m_reduceGrid, m_block, 0, m_hostStream >> > (cu_deviceData, reduceBatchSizePow2, uvec2(batchSize, 2));

            //const vec2 minMax = GetProbeGridAggregateStatistics();
            //Log::Debug("Samples: %i\n", minMax.x);

            grid.Integrate();

            IsOk(hipStreamSynchronize(m_hostStream));
        }

        // If we're filtering, let any listening filters know that the build is complete
        if (m_params.filterGrids)
        {
            OnEvent("OnBuildGrids");
        }
    }

    __global__ void KernelBuildLightProbeErrorGrid(Device::LightProbeCamera* cu_camera)
    {
        cu_camera->BuildLightProbeErrorGrid();
    }

    __global__ void KernelDilateLightProbeErrorGrid(Device::LightProbeCamera* cu_camera)
    {
        cu_camera->DilateLightProbeErrorGrid();
    }

    __host__ void Host::LightProbeCamera::BuildLightProbeErrorGrid()
    {
        const int gridSize = (m_params.grid.numProbes + 255) / 256;
        KernelBuildLightProbeErrorGrid << < gridSize, 256, 0, m_hostStream >> > (cu_deviceData);
        IsOk(hipStreamSynchronize(m_hostStream));

        // Dilate the adaptive sampling grid
        KernelDilateLightProbeErrorGrid << < gridSize, 256, 0, m_hostStream >> > (cu_deviceData);
        IsOk(hipStreamSynchronize(m_hostStream));
        m_hostLightProbeErrorGrids[0]->Swap(*m_hostLightProbeErrorGrids[1]);
    }

    __host__ void Host::LightProbeCamera::OnPostRenderPass()
    {
        if ((m_frameIdx - 2) % m_params.gridUpdateInterval == 0)
        {
            Compile();
        }
    }

    __host__ void Host::LightProbeCamera::Compile()
    {
        // Compile the data in the accumulation buffer into the grid data structures
        BuildLightProbeGrids();

        // If we're using adaptive sampling to monitor convergence, construct the grid
        if (m_params.camera.samplingMode != kCameraSamplingFixed)
        {
            BuildLightProbeErrorGrid();
        }

        // Compute aggregate statistics about each grid such as min and max sample count
        UpdateProbeGridAggregateStatistics();
    }

    __host__ void Host::LightProbeCamera::Bind(RenderObjectContainer& sceneObjects)
    {                
        // Bind the adaptive sampling grids
        for (int idx = 0; idx < m_hostFilteredLightProbeGrids.size(); ++idx)
        {  
            if (m_params.camera.useFilteredError && !m_filteredGridIDs[idx].empty())
            {
                m_hostFilteredLightProbeGrids[idx] = sceneObjects.FindByID(m_filteredGridIDs[idx]).DynamicCast<Host::LightProbeGrid>();
                AssertMsgFmt(m_hostFilteredLightProbeGrids[idx], "Error: LightProbeCamera::Bind(): the specified light probe grid '%s' is invalid.\n", m_filteredGridIDs[idx].c_str());

                m_hostFilteredLightProbeGrids[idx]->SetExternalBuffers(m_hostConvergenceGrid, m_hostLightProbeErrorGrids[0], m_hostMeanI);
                
                Log::Write("Bound light probe grid '%s' to light probe camera '%s'", m_filteredGridIDs[idx], GetAssetID());
            }
            else
            {
                // If we're not using the filtered error, don't try and bind a different set of adaptive sampling grids
                m_hostFilteredLightProbeGrids[idx] = m_hostLightProbeGrids[idx];
            }

            m_deviceObjects.cu_filteredProbeGrids[idx] = m_hostFilteredLightProbeGrids[idx]->GetDeviceInstance();
        }

        SynchroniseObjects(cu_deviceData, m_deviceObjects);
    }

    __host__ void Host::LightProbeCamera::OnUpdateSceneGraph(RenderObjectContainer& sceneObjects)
    {
        if (m_needsRebind)
        {
            Bind(sceneObjects);
            m_needsRebind = true;
        }
    }

    __host__ bool Host::LightProbeCamera::EmitStatistics(Json::Node& rootNode) const
    {        
        auto& stats = *m_aggregateStats;

        rootNode.AddValue("isActive", m_params.camera.isActive);  
        rootNode.AddValue("frameIdx", m_frameIdx);
        rootNode.AddValue("lightingMode", std::string((m_params.lightingMode == kBakeLightingCombined) ? "combined" : "separated"));
        rootNode.AddValue("minSamples", int(stats.minMaxSamples.x));
        rootNode.AddValue("maxSamples", int(stats.minMaxSamples.y));
        rootNode.AddValue("meanSamples", stats.meanSamples);
        rootNode.AddValue("meanValidity", stats.meanValidity);
        rootNode.AddValue("meanDistance", stats.meanDistance);

        Json::Node bakeNode = rootNode.AddChildObject("bake");
        bakeNode.AddValue("progress", m_aggregateStats->bake.progress);
        bakeNode.AddValue("probesConverged", m_aggregateStats->bake.probesConverged);
        bakeNode.AddValue("probesFull", m_aggregateStats->bake.probesFull);
        
        Json::Node errorNode = rootNode.AddChildObject("error");
        errorNode.AddValue("mse", m_aggregateStats->error.MSE);
        errorNode.AddValue("meanI", m_aggregateStats->error.meanI);       

        Json::Node gridSetNode = rootNode.AddChildObject("grids");
        for (int gridIdx = 0; gridIdx < kLightProbeNumBuffers; ++gridIdx)
        {
            if (!m_hostLightProbeGrids[gridIdx]) { continue; }

            const auto& grid = m_hostLightProbeGrids[gridIdx]->GetAggregateStatistics();
            
            Json::Node gridNode = gridSetNode.AddChildObject(m_gridIDs[gridIdx]);                  

            std::vector<std::vector<uint>> histogramData(4);
            for (int idx = 0; idx < 4; ++idx)
            {
                histogramData[idx].resize(50);
                std::memcpy(histogramData[idx].data(), &grid.coeffHistogram[50 * idx], sizeof(uint) * 50);
            }
            gridNode.AddArray2D("coeffHistograms", histogramData);

            std::vector<float> peakIntensityData(4);
            std:memcpy(peakIntensityData.data(), grid.meanSqrIntensity, sizeof(float) * 4);
            gridNode.AddArray("peakIntensity", peakIntensityData);

        }

        return true;
    }
}