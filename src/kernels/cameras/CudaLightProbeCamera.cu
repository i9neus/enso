#include "hip/hip_runtime.h"
﻿#define CUDA_DEVICE_ASSERTS

#include "CudaLightProbeCamera.cuh"
#include "generic/JsonUtils.h"

#include "../CudaCtx.cuh"
#include "../CudaManagedArray.cuh"
#include "../CudaManagedObject.cuh"

#include "../math/CudaSphericalHarmonics.cuh"

#include "../../io/USDIO.h"

#define kAccumBufferWidth 1024u
#define kAccumBufferHeight 1024u
#define kAccumBufferSize (kAccumBufferWidth * kAccumBufferHeight)

#define kRayBufferSize          (512u * 512u * 2u)
#define kRayBufferNumBuckets    (512u * 512u)

namespace Cuda
{
    __host__ __device__ LightProbeCameraParams::LightProbeCameraParams()
    {
        lightingMode = kBakeLightingCombined;
        gridUpdateInterval = 10;
        minViableValidity = 0.0f;
    }

    __host__ LightProbeCameraParams::LightProbeCameraParams(const ::Json::Node& node) :
        LightProbeCameraParams()
    {
        FromJson(node, ::Json::kRequiredWarn);
    }

    __host__ void LightProbeCameraParams::ToJson(::Json::Node& node) const
    {
        auto gridNode = node.AddChildObject("grid");
        grid.ToJson(gridNode);
        camera.ToJson(node);

        node.AddEnumeratedParameter("lightingMode", std::vector<std::string>({ "combined", "separated" }), lightingMode);
        node.AddValue("gridUpdateInterval", gridUpdateInterval);
        node.AddValue("minViableValidity", minViableValidity);
    }

    __host__ void LightProbeCameraParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        auto gridNode = node.GetChildObject("grid", flags);
        grid.FromJson(gridNode, flags);
        camera.FromJson(node, flags);

        node.GetEnumeratedParameter("lightingMode", std::vector<std::string>({ "combined", "separated" }), lightingMode, flags);
        node.GetValue("gridUpdateInterval", gridUpdateInterval, flags);
        node.GetValue("minViableValidity", minViableValidity, flags);
    }      

    __device__ Device::LightProbeCamera::LightProbeCamera() {  }

    __device__ void Device::LightProbeCamera::Synchronise(const LightProbeCameraParams& params)
    {
        m_params = params;
        if (m_params.camera.maxSamples == 0) { m_params.camera.maxSamples = INT_MAX; }

        Prepare();
    }
    __device__ void Device::LightProbeCamera::Synchronise(const Objects& objects)
    {
        m_objects = objects;
    }

    __device__ void Device::LightProbeCamera::SeedRayBuffer(const int frameIdx)
    {
        assert(kKernelIdx * 2 < kRayBufferSize);
        
        CompressedRay* compressedRays = &(*m_objects.renderState.cu_compressedRayBuffer)[kKernelIdx * 2];

        if (kKernelIdx > m_params.totalBuckets) 
        {
            compressedRays[0].Kill();
            compressedRays[1].Kill();
            return;
        }

        // On the first frame, reset the ray and the sample index
        if (frameIdx == 0)
        {
            compressedRays[0].Reset();
            compressedRays[1].Reset();
            compressedRays[0].sampleIdx = m_seedOffset;
        }

        if (!compressedRays[0].IsAlive() && !compressedRays[1].IsAlive() &&
            (m_params.camera.maxSamples <= 0 || int(compressedRays[0].sampleIdx - m_seedOffset) < m_params.maxSamplesPerBucket))
        {
            CreateRays(kKernelIdx, compressedRays, frameIdx);
        }
    }

    __device__ void Device::LightProbeCamera::Composite(const ivec2& accumPos, Device::ImageRGBA* deviceOutputImage) const
    {
        const ivec2 viewportPos = accumPos + deviceOutputImage->Dimensions() / 2 - ivec2(kAccumBufferWidth, kAccumBufferHeight) / 2;
        if (viewportPos.x < 0 || viewportPos.x >= deviceOutputImage->Width() ||
            viewportPos.y < 0 || viewportPos.y >= deviceOutputImage->Height()) {
            return;
        }

        assert(accumPos.y * kAccumBufferWidth + accumPos.x <= m_objects.cu_reduceBuffer->Size());
        //assert(m_objects.cu_reduceBuffer->Size() == kAccumBufferSize);

        int idx = accumPos.x / (kAccumBufferWidth / 2);

        // Normalise and gamma correct
        const auto& texel = (*m_objects.cu_accumBuffers[idx])[accumPos.y * kAccumBufferWidth + accumPos.x];
        const vec3 rgb = texel.xyz / fmax(1.0f, texel.w);       
        deviceOutputImage->At(viewportPos) = vec4(rgb, 1.0f);
    }

    __device__ void Device::LightProbeCamera::Prepare()
    {
        assert(m_objects.cu_accumBuffers[0] && m_objects.cu_accumBuffers[1]);
        assert(m_objects.cu_probeGrids[0] && m_objects.cu_probeGrids[1]);

        // Only use the lower 31 bits for the seed because we need to deduce the actual sample count from it
        m_seedOffset = HashOf(uint(m_params.camera.seed) & ((1u << 31) - 1u));
    }

    __device__ void Device::LightProbeCamera::CreateRays(const uint& subsampleIdx, CompressedRay* rays, const int frameIdx) const
    {
        assert(subsampleIdx < kAccumBufferSize);
        
        const int probeIdx = subsampleIdx / m_params.subsamplesPerProbe;
        const ivec3 gridIdx = GridPosFromProbeIdx(probeIdx, m_params.grid.gridDensity);

        auto& primary = rays[0]; 
        auto& secondary = rays[1];
        
        primary.accumIdx = subsampleIdx;
        primary.sampleIdx++;
        primary.depth = 0;
        RNG rng(primary);

        const vec2 xi = rng.Rand<0, 1>();
        primary.od.d = SampleUnitSphere(xi);
        primary.probeDir = primary.od.d;

        // Probes data is grouped as follows:
        // Subsample    = [SH data] [Auxilliary data]
        // Probe        = [Subsample 1] .... [Subsample N]
        // Grid         = [Probe 1] ... [Probe M]        

        // Project this direction into SH and pre-normalise
        primary.weight = kOne;
        primary.depth = 1;
        primary.flags = kRayIndirectSample | kRayLightProbe;

        primary.od.o = m_params.grid.aspectRatio * vec3(gridIdx) / vec3(m_params.grid.gridDensity - 1) - vec3(0.5f);
        primary.od.o = m_params.grid.transform.PointToWorldSpace(primary.od.o);

        secondary.accumIdx = subsampleIdx;
        secondary.probeDir = primary.probeDir;
        secondary.sampleIdx = primary.sampleIdx;
    }

    __device__ void Device::LightProbeCamera::Accumulate(const RenderCtx& ctx, const Ray& incidentRay, const HitCtx& hitCtx, const vec3& value, const bool isAlive)
    {          
        auto& emplacedRay = ctx.emplacedRay[0];
        assert(emplacedRay.accumIdx < kAccumBufferSize);
        
        vec3 L = value;
        if (m_params.camera.splatClamp > 0.0)
        {
            const float intensity = cwiseMax(L);
            if (intensity > m_params.camera.splatClamp)
            {
                L *= m_params.camera.splatClamp / intensity;
            }
        }

        // Loop through the direct and indirect accumulation buffers
        for (int gridIdx = 0; gridIdx < kLightProbeNumBuffers; ++gridIdx)
        {
            if (!m_objects.cu_accumBuffers[gridIdx]) { continue; }

            // Don't write into the indirect grid when in combined lighting mode
            if (m_params.lightingMode == kBakeLightingCombined && gridIdx == kLightProbeBufferIndirect) { continue; }
            
            int accumIdx = emplacedRay.accumIdx * m_params.coefficientsPerProbe;
            auto& accumBuffer = *(m_objects.cu_accumBuffers[gridIdx]);
            const float weight = !isAlive;

            // Should we accumulate this sample?
            if ((gridIdx != kLightProbeBufferHalf && (m_params.lightingMode == kBakeLightingCombined || 
                                                      (gridIdx == 0 && incidentRay.depth == 1) || (gridIdx == 1 && incidentRay.depth > 1))) 
                || (gridIdx == kLightProbeBufferHalf && emplacedRay.sampleIdx % 2u == 1u))
                
            {              
                // Project and accumulate the SH coefficients
                for (int shIdx = 0; shIdx < m_params.coefficientsPerProbe - 1; ++shIdx, ++accumIdx)
                {
                    accumBuffer[accumIdx] += vec4(L * SH::Project(ctx.emplacedRay[0].probeDir, shIdx) * kFourPi, weight);
                }
            }
            else
            {
                // Just increment the weights
                for (int shIdx = 0; shIdx < m_params.coefficientsPerProbe - 1; ++shIdx, ++accumIdx) { accumBuffer[accumIdx][3] += weight; }
            }

            if (gridIdx != kLightProbeBufferHalf && incidentRay.depth == 1)
            {
                // Accumulate validity and mean distance
                // A probe sample is valid if, on the first hit, it intersects with a front-facing surface or it leaves the scene
                accumBuffer[accumIdx] += vec4(float(!hitCtx.isValid || !hitCtx.backfacing), 
                                              //1.0f / max(1e-10f, incidentRay.tNear),
                                              min(m_params.grid.transform.scale().x, incidentRay.tNear) / m_params.grid.transform.scale().x,
                                              0.0f, 1.0f);
            }
        }
    }
    __device__ void Device::LightProbeCamera::ReduceAccumulatedSample(vec4& dest, const vec4& source)
    {              
        if (int(dest.w) >= m_params.grid.maxSamplesPerProbe - 1) 
        {             
            return;
        }
        
        if (int(dest.w + source.w) < m_params.grid.maxSamplesPerProbe)
        {           
            dest += source;
            return;
        }

        dest += source * (m_params.grid.maxSamplesPerProbe - dest.w) / source.w;
    }

    __device__ void Device::LightProbeCamera::ReduceAccumulationBuffer(Device::Array<vec4>* cu_accumBuffer, Device::LightProbeGrid* cu_probeGrid, const uint batchSize, const uvec2 batchRange)
    {         
        if (kKernelIdx >= m_params.totalBuckets) { return; }

        assert(cu_accumBuffer);
        assert(cu_probeGrid);
        assert(m_objects.cu_reduceBuffer);

        //if (batchRange[0] == batchSize) (*m_objects.cu_reduceBuffer)[kKernelIdx] = 0.0f;

        auto& accumBuffer = *cu_accumBuffer;
        auto& reduceBuffer = *m_objects.cu_reduceBuffer;
        
        const int probeIdx = kKernelIdx / m_params.bucketsPerProbe;
        const int probeSubsampleIdx = (kKernelIdx / m_params.coefficientsPerProbe) % m_params.subsamplesPerProbe;
        const int coeffIdx = kKernelIdx % m_params.coefficientsPerProbe;

        for (uint iterationSize = batchRange[0] / 2; iterationSize > batchRange[1] / 2; iterationSize >>= 1)
        {
            if (probeSubsampleIdx < iterationSize)
            {
                // For the first iteration, copy the data out of the accumulation buffer
                if (iterationSize == batchSize / 2)
                {
                    auto& texel = reduceBuffer[kKernelIdx];
                    texel = 0.0f;
                    ReduceAccumulatedSample(texel, accumBuffer[kKernelIdx]);

                    if (probeSubsampleIdx + iterationSize < m_params.subsamplesPerProbe)
                    {
                        assert(kKernelIdx + iterationSize * m_params.coefficientsPerProbe < kAccumBufferSize);
                        //if (probeIdx == 0 && coeffIdx == 0) { printf("%i: %f + %f = %f\n", iterationSize, texel.w, accumBuffer[kKernelIdx + iterationSize].w, texel.w + accumBuffer[kKernelIdx + iterationSize].w); }
                        ReduceAccumulatedSample(texel, accumBuffer[kKernelIdx + iterationSize * m_params.coefficientsPerProbe]);

                    }
                    //else
                    //    if (probeIdx == 0 && coeffIdx == 0) { printf("%i: %f\n", iterationSize, texel.w); }
                }
                else
                {
                    assert(kKernelIdx + iterationSize * m_params.coefficientsPerProbe < kAccumBufferSize);
                    assert(probeSubsampleIdx + iterationSize < m_params.subsamplesPerProbe);
                    //if (probeIdx == 0 && coeffIdx == 0) { printf("%i: %f + %f = %f\n", iterationSize, reduceBuffer[kKernelIdx].w, reduceBuffer[kKernelIdx + iterationSize].w, reduceBuffer[kKernelIdx].w + reduceBuffer[kKernelIdx + iterationSize].w); }
                    ReduceAccumulatedSample(reduceBuffer[kKernelIdx], reduceBuffer[kKernelIdx + iterationSize * m_params.coefficientsPerProbe]);
                }
            }
            else
            {
                //reduceBuffer[kKernelIdx] = 1.0f;
            }

            __syncthreads();
        } 

        // After the last operation, cache the accumulated value in the probe grid
        if (probeSubsampleIdx == 0 && batchRange[0] == 2)
        {
            auto& texel = reduceBuffer[kKernelIdx];         

            //const int probeIdx = kKernelIdx / m_params.bucketsPerProbe;
            //const int coeffIdx = (kKernelIdx / m_params.bucketsPerCoefficient) % m_params.coefficientsPerProbe;
            if (coeffIdx == m_params.coefficientsPerProbe - 1)
            {
               const float norm = max(1.0f, texel.w);

               texel.x /= norm;                         // Probe validity
               //texel.y = norm / max(1e-10f, texel.y); // Harmonic mean distance
               texel.y /= norm;                         // Geometric mean distance
               texel.z = texel.w;                       // Store the total number of samples
            }
            else
            {
                texel /= max(1.0f, texel.w);
            }

            cu_probeGrid->SetSHCoefficient(probeIdx, coeffIdx, texel.xyz);
        }
    }

    __device__ void Device::LightProbeCamera::GetProbeGridAggregateStatistics(Device::LightProbeCamera::AggregateStatistics& result, uint* distanceHistogram) const
    {
        __shared__ AggregateStatistics localStats[256];
        __shared__ uint sharedHistogram[50];

        if (kThreadIdx == 0)
        {
            for (int i = 0; i < 256; i++)
            {
                localStats[i].minMaxSamples = vec2(kFltMax, 0.0f);
                localStats[i].meanValidity = 0.0f;
                localStats[i].meanDistance = 0.0f;
                localStats[i].probeCount = 0;
            }
            for (int i = 0; i < 50; ++i) { sharedHistogram[i] = 0; }
        }

        __syncthreads();

        const int startIdx = (m_params.numProbes - 1) * kKernelIdx / 256;
        const int endIdx = (m_params.numProbes - 1) * (kKernelIdx + 1) / 256;
        localStats[kKernelIdx].probeCount = 1 + endIdx - startIdx;

        for (int i = startIdx; i <= endIdx; i++)
        {
            const auto& coeffs = m_objects.cu_probeGrids[0]->At(startIdx)[m_params.coefficientsPerProbe - 1];

            localStats[kKernelIdx].minMaxSamples = vec2(min(localStats[kKernelIdx].minMaxSamples.x, coeffs.z), max(localStats[kKernelIdx].minMaxSamples.y, coeffs.z));
            localStats[kKernelIdx].meanValidity += coeffs.x;
            localStats[kKernelIdx].meanDistance += coeffs.y;

            atomicInc(&sharedHistogram[uint(clamp(coeffs.y, 0.0f, 1.0f) * 49)], 0xffffffff);
        }

        __syncthreads();

        if (kThreadIdx == 0)
        {
            result = localStats[0];
            result.meanValidity /= float(max(1, localStats[0].probeCount));
            result.meanDistance /= float(max(1, localStats[0].probeCount));
            for (int i = 1; i < 256; i++)
            {
                result.minMaxSamples = vec2(min(localStats[i].minMaxSamples.x, result.minMaxSamples.x), max(localStats[i].minMaxSamples.y, result.minMaxSamples.y));
                result.meanValidity += localStats[i].meanValidity / float(max(1, localStats[i].probeCount));
                result.meanDistance += localStats[i].meanDistance / float(max(1, localStats[i].probeCount));
            }
            result.meanValidity /= 256.0f;
            result.meanDistance /= 256.0f;

            memcpy(distanceHistogram, sharedHistogram, sizeof(int) * 50);
        }
    }

    __host__ Host::LightProbeCamera::LightProbeCamera(const ::Json::Node& node, const std::string& id) :
        Host::Camera(node, id, kRayBufferSize),
        m_block(16 * 16, 1, 1),
        m_seedGrid(1, 1, 1),
        m_reduceGrid(1, 1, 1),
        m_exporterState(kDisarmed),
        m_bakeProgress(0.0f)        
    {        
        std::string gridIDs[3];

        // TODO: This is to maintain backwards compatibility. Deprecate it when no longer required.
        gridIDs[0] = "grid_noisy_direct";
        gridIDs[1] = "grid_noisy_indirect";

        node.GetValue("gridDirectID", gridIDs[0], Json::kRequiredWarn | Json::kNotBlank);
        node.GetValue("gridIndirectID", gridIDs[1], Json::kRequiredWarn | Json::kNotBlank);
        node.GetValue("gridHalfID", gridIDs[2], Json::kSilent);

        // Create the accumulation buffers and probe grids
        for (int idx = 0; idx < m_hostAccumBuffers.size(); ++idx)
        {
            // Don't create grids that don't have IDs
            if (gridIDs[idx].empty()) { continue; }

            m_hostAccumBuffers[idx] = AssetHandle<Host::Array<vec4>>(tfm::format("%s_probeAccumBuffer%i", id, idx), kAccumBufferSize, m_hostStream);
            m_hostAccumBuffers[idx]->Clear(vec4(0.0f));
            m_deviceObjects.cu_accumBuffers[idx] = m_hostAccumBuffers[idx]->GetDeviceInstance();

            m_hostLightProbeGrids[idx] = AssetHandle<Host::LightProbeGrid>(gridIDs[idx], gridIDs[idx]);

            m_deviceObjects.cu_probeGrids[idx] = m_hostLightProbeGrids[idx]->GetDeviceInstance();
        }

        // Create the reduction buffer
        m_hostReduceBuffer = AssetHandle<Host::Array<vec4>>(tfm::format("%s_probeReduceBuffer", id), kAccumBufferSize, m_hostStream);

        // Instantiate the camera object on the device
        cu_deviceData = InstantiateOnDevice<Device::LightProbeCamera>();

        // Sychronise the device objects
        m_deviceObjects.cu_reduceBuffer = m_hostReduceBuffer->GetDeviceInstance();
        m_deviceObjects.renderState.cu_compressedRayBuffer = m_hostCompressedRayBuffer->GetDeviceInstance();
        m_deviceObjects.renderState.cu_blockRayOccupancy = m_hostBlockRayOccupancy->GetDeviceInstance();
        m_deviceObjects.renderState.cu_renderStats = m_hostRenderStats->GetDeviceInstance();

        // Objects are re-synchronised at every JSON update
        FromJson(node, ::Json::kRequiredWarn);
    }

    __host__ AssetHandle<Host::RenderObject> Host::LightProbeCamera::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kCamera) { return AssetHandle<Host::RenderObject>(); }

        return AssetHandle<Host::RenderObject>(new Host::LightProbeCamera(json, id), id);
    }

    __host__ void Host::LightProbeCamera::OnDestroyAsset()
    {
        Host::Camera::OnDestroyAsset();

        // Destroy the light probe grids and accumulation buffers
        for (auto& accumBuffer : m_hostAccumBuffers) { accumBuffer.DestroyAsset(); }
        for (auto& grid : m_hostLightProbeGrids) { grid.DestroyAsset(); }

        // Destroy the rest of the objects
        m_hostReduceBuffer.DestroyAsset();
        DestroyOnDevice(cu_deviceData);
    }

    __host__ std::vector<AssetHandle<Host::RenderObject>> Host::LightProbeCamera::GetChildObjectHandles()
    {
        std::vector<AssetHandle<Host::RenderObject>> children;
        for (auto& grid : m_hostLightProbeGrids)
        {
            if (grid) { children.push_back(AssetHandle<Host::RenderObject>(grid)); }
        }
        return children;
    }

    __host__ void Host::LightProbeCamera::FromJson(const ::Json::Node& parentNode, const uint flags)
    {
        // FIXME: Should this just be called once on construction instead of every time the object updates?        
        Host::RenderObject::UpdateDAGPath(parentNode);

        m_params.FromJson(parentNode, flags);

        Prepare();
    }

    __host__ void Host::LightProbeCamera::Prepare()
    {
        m_params.coefficientsPerProbe = SH::GetNumCoefficients(m_params.grid.shOrder) + 1;

        // Reduce the size of the grid if it exceeds the size of the accumulation buffer
        const int maxNumProbes = min(kAccumBufferSize / m_params.coefficientsPerProbe, kRayBufferNumBuckets);
        if (Volume(m_params.grid.gridDensity) > maxNumProbes)
        {
            const auto oldDensity = m_params.grid.gridDensity;
            while (Volume(m_params.grid.gridDensity) > maxNumProbes)
            {
                m_params.grid.gridDensity = max(ivec3(1), m_params.grid.gridDensity - ivec3(1));
            }
            Log::Error("WARNING: The size of the probe grid %s is too large for the accumulation buffer. Reducing to %s.\n", oldDensity.format(), m_params.grid.gridDensity.format());
        }

        // Prepare the light probe grid with the new parameters
        for (auto& grid : m_hostLightProbeGrids)
        {
            if (grid) { grid->Prepare(m_params.grid); }
        }

        // Number of light probes in the grid
        m_params.numProbes = Volume(m_params.grid.gridDensity);
        // Number of SH parameter sets per probe, reduced later to get the final value 
        m_params.subsamplesPerProbe = min(kRayBufferNumBuckets / m_params.numProbes,
            kAccumBufferSize / (m_params.numProbes * m_params.coefficientsPerProbe));

        // The maximum number of samples per bucket based on the number of buckets per coefficient
        m_params.grid.maxSamplesPerProbe = m_params.maxSamplesPerBucket = std::numeric_limits<int>::max();
        if (m_params.camera.maxSamples > 0)
        {
            m_params.grid.maxSamplesPerProbe = m_params.camera.maxSamples;
            m_params.maxSamplesPerBucket = int(1.0f + float(m_params.camera.maxSamples) / float(m_params.subsamplesPerProbe));
        }

        // Derive some more values
        m_params.bucketsPerProbe = m_params.subsamplesPerProbe * m_params.coefficientsPerProbe;
        m_params.totalBuckets = m_params.bucketsPerProbe * m_params.numProbes;
        m_params.totalSubsamples = m_params.subsamplesPerProbe * m_params.numProbes;

        // Used when parallel reducing the accumluation buffer
        uint reduceBatchSizePow2 = NearestPow2Ceil(m_params.subsamplesPerProbe);

        Log::Debug("coefficientsPerProbe: %i\n", m_params.coefficientsPerProbe);
        Log::Debug("numProbes: %i\n", m_params.numProbes);
        Log::Debug("subsamplesPerProbe: %i\n", m_params.subsamplesPerProbe);
        Log::Debug("bucketsPerProbe: %i\n", m_params.bucketsPerProbe);
        Log::Debug("totalBuckets: %i\n", m_params.totalBuckets);
        Log::Debug("maxSamplesPerBucket: %i\n", m_params.maxSamplesPerBucket);
        Log::Debug("reduceBatchSizePow2: %i\n", reduceBatchSizePow2);

        // Sync everything with the device
        SynchroniseObjects(cu_deviceData, m_deviceObjects);
        SynchroniseObjects(cu_deviceData, m_params);

        const int seedGridSize = m_params.totalSubsamples;
        m_seedGrid = dim3((seedGridSize + (m_block.x - 1)) / m_block.x, 1, 1);
        const int reduceGridSize = m_params.totalBuckets;
        m_reduceGrid = dim3((reduceGridSize + (m_block.x - 1)) / m_block.x, 1, 1);

        Log::Debug("m_seedGrid: [%i, %i, %i]\n", m_seedGrid.x, m_seedGrid.y, m_seedGrid.z);
        Log::Debug("m_reduceGrid: [%i, %i, %i]\n", m_reduceGrid.x, m_reduceGrid.y, m_reduceGrid.z);

        m_frameIdx = 0;
    }

    __host__ void Host::LightProbeCamera::ClearRenderState()
    {
        for (auto& accumBuffer : m_hostAccumBuffers)
        {
            if (accumBuffer)
            {
                accumBuffer->Clear(vec4(0.0f));
            }
        }
        m_hostCompressedRayBuffer->Clear(Cuda::CompressedRay());
        m_bakeProgress = 0.0f;
        //m_hostPixelFlagsBuffer->Clear(0);
    }

    __global__ void KernelSeedRayBuffer(Device::LightProbeCamera* camera, const int frameIdx)
    {
        camera->SeedRayBuffer(frameIdx);
    }

    __host__ void Host::LightProbeCamera::OnPreRenderPass(const float wallTime, const uint frameIdx)
    {
        m_frameIdx = frameIdx;

        KernelSeedRayBuffer << < m_seedGrid, m_block, 0, m_hostStream >> > (cu_deviceData, frameIdx);
    }

    __global__ void KernelComposite(Device::ImageRGBA* deviceOutputImage, const Device::LightProbeCamera* camera)
    {
        //if (*(deviceOutputImage->AccessSignal()) != kImageWriteLocked) { return; }

        camera->Composite(kKernelPos<ivec2>(), deviceOutputImage);
    }

    __host__ void Host::LightProbeCamera::Composite(AssetHandle<Host::ImageRGBA>& hostOutputImage) const
    {
        dim3 blockSize = dim3(16, 16, 1);
        dim3 gridSize(kAccumBufferWidth / 16, kAccumBufferHeight / 16, 1);

        hostOutputImage->SignalSetWrite(m_hostStream);
        KernelComposite << < gridSize, blockSize, 0, m_hostStream >> > (hostOutputImage->GetDeviceInstance(), cu_deviceData);
        hostOutputImage->SignalUnsetWrite(m_hostStream);
    }

    __global__ void KernelReduceAccumulationBuffer(Device::LightProbeCamera* camera, Device::Array<vec4>* cu_accumBuffer, Device::LightProbeGrid* cu_probeGrid,
        const uint reduceBatchSize, const uvec2 batchRange)
    {
        camera->ReduceAccumulationBuffer(cu_accumBuffer, cu_probeGrid, reduceBatchSize, batchRange);
    }

    __global__ void KernelGetProbeGridAggregateStatistics(Device::LightProbeCamera* camera, Device::LightProbeCamera::AggregateStatistics* stats, uint* histogram)
    {
        assert(stats);
        camera->GetProbeGridAggregateStatistics(*stats, histogram);
    }

    __host__ void Host::LightProbeCamera::GetProbeGridAggregateStatistics()
    {
        KernelGetProbeGridAggregateStatistics << <1, 256, 0, m_hostStream >> > (cu_deviceData, m_probeAggregateData.GetDeviceObject(), m_distanceHistogram.GetDeviceObject());
        IsOk(hipStreamSynchronize(m_hostStream));

        m_probeAggregateData.Download();
        m_distanceHistogram.Download();

        m_bakeProgress = -1.0f;
        if (m_params.camera.maxSamples > 0)
        {
            m_bakeProgress = clamp((m_probeAggregateData->minMaxSamples.x + 1.0f) / float(m_params.grid.maxSamplesPerProbe), 0.0f, 1.0f);
        }
    }

    __host__ float Host::LightProbeCamera::GetBakeProgress()
    {
        // FIXME: This is a horrible hack to prevent having to manually scan the accumulation buffer every frame.
        /*if (m_frameIdx < m_params.maxSamplesPerProbe)
        {
            return clamp(m_frameIdx / float(m_params.maxSamplesPerProbe), 0.0f, 1.0f);
        }*/

        if ((m_frameIdx - 2) % m_params.gridUpdateInterval == 0)
        {
            GetProbeGridAggregateStatistics();
        }

        return m_bakeProgress;
    }

    __host__ bool Host::LightProbeCamera::ExportProbeGrid(const std::vector<std::string>& usdExportPaths, const bool exportToUSD)
    {
        if (m_exporterState != kArmed) { return false; }

        BuildLightProbeGrids();

        for (int gridIdx = kLightProbeBufferDirect; gridIdx != kLightProbeBufferIndirect + 1; ++gridIdx)
        {
            // Don't write out indirect when running in combined mode
            if (m_params.lightingMode == kBakeLightingCombined && gridIdx == kLightProbeBufferIndirect) { continue; }

            if (exportToUSD)
            {
                Log::Debug("Exporting to '%s'...\n", usdExportPaths[gridIdx]);
                try
                {
                    USDIO::ExportLightProbeGrid(m_hostLightProbeGrids[gridIdx], usdExportPaths[gridIdx]);
                }
                catch (const std::runtime_error& err)
                {
                    Log::Error("Error: %s\n", err.what());
                }
            }
            else
            {
                Log::Warning("Warning: Skipped USD export to '%s' because setting was not enabled.\n", usdExportPaths[gridIdx]);
                break;
            }
        }

        m_exporterState = kFired;
        return true;
    }

    __host__ void Host::LightProbeCamera::SetLightProbeCameraParams(const LightProbeCameraParams& params)
    {
        m_params = params;
        Prepare();
    }

    __host__ void Host::LightProbeCamera::BuildLightProbeGrids()
    {
        // Used when parallel reducing the accumluation buffer
        uint reduceBatchSizePow2 = NearestPow2Ceil(m_params.subsamplesPerProbe);

        for (int gridIdx = 0; gridIdx < kLightProbeNumBuffers; ++gridIdx)
        {
            if (!m_hostLightProbeGrids[gridIdx]) { continue; }

            // Indirect buffer isn't used when running in combined mode
            if (m_params.lightingMode == kBakeLightingCombined && gridIdx == kLightProbeBufferIndirect) { continue; }

            // Reduce until the batch range is equal to the size of the block
            uint batchSize = reduceBatchSizePow2;
            while (batchSize > 1)
            {
                KernelReduceAccumulationBuffer << < m_reduceGrid, m_block, 0, m_hostStream >> > (cu_deviceData, m_deviceObjects.cu_accumBuffers[gridIdx],
                    m_deviceObjects.cu_probeGrids[gridIdx],
                    reduceBatchSizePow2, uvec2(batchSize, batchSize >> 1));
                batchSize >>= 1;
            }
            // Reduce the block in a single operation
            //KernelReduceAccumulationBuffer << < m_reduceGrid, m_block, 0, m_hostStream >> > (cu_deviceData, reduceBatchSizePow2, uvec2(batchSize, 2));

            //const vec2 minMax = GetProbeGridAggregateStatistics();
            //Log::Debug("Samples: %i\n", minMax.x);

            IsOk(hipStreamSynchronize(m_hostStream));
        }
    }

    __host__ void Host::LightProbeCamera::OnPostRenderPass()
    {
        if ((m_frameIdx - 2) % m_params.gridUpdateInterval == 0)
        {
            BuildLightProbeGrids();

            GetProbeGridAggregateStatistics();
        }
    }

    __host__ bool Host::LightProbeCamera::EmitStatistics(Json::Node& node) const
    {
        node.AddValue("isActive", m_params.camera.isActive);
        node.AddValue("minSamples", int(m_probeAggregateData->minMaxSamples.x));
        node.AddValue("maxSamples", int(m_probeAggregateData->minMaxSamples.y));
        node.AddValue("meanProbeValidity", m_probeAggregateData->meanValidity);
        node.AddValue("meanProbeDistance", m_probeAggregateData->meanDistance);
        node.AddValue("bakeProgress", m_bakeProgress);
        
        std::vector<uint> histogramData(50, 0);
        std::memcpy(histogramData.data(), &m_distanceHistogram[0], sizeof(uint) * 50);
        node.AddArray("distanceHistogram", histogramData);

        return true;
    }
}