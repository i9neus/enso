﻿#include "CudaTransform.cuh"
#include "generic/JsonUtils.h"

namespace Cuda
{
    __host__ void BidirectionalTransform::FromJson(const ::Json::Node& node, const uint flags)
    {
        const auto transNode = node.GetChildObject("transform", flags);
        if (!transNode) { return; }

        trans = rot = 0.0f;
        scale = 1.0f;

        transNode.GetVector("pos", trans, flags);       
        transNode.GetVector("rot", rot, flags);
        transNode.GetVector("sca", scale, flags);

        // Convert from degrees to radians
        rot = toRad(rot);
        
        // Build the transform
        Create(trans, rot, scale);
    }

    __host__ void BidirectionalTransform::ToJson(Json::Node& parentNode) const
    {
        auto transNode = parentNode.AddChildObject("transform");

        transNode.AddArray("pos", std::vector<float>({ trans.x, trans.y, trans.z }));
        transNode.AddArray("rot", std::vector<float>({ toDeg(rot.x), toDeg(rot.y), toDeg(rot.z) }));
        transNode.AddArray("sca", std::vector<float>({ scale.x, scale.y, scale.z }));
    }

    __host__ BidirectionalTransform::BidirectionalTransform(const ::Json::Node& node, const uint flags)
    {
        FromJson(node, flags);
    }

    __host__ bool BidirectionalTransform::operator==(const BidirectionalTransform& rhs) const
    {
        return trans == rhs.trans && rot == rhs.rot && scale == rhs.scale;
    }
}