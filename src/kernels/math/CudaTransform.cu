﻿#include "CudaTransform.cuh"
#include "generic/JsonUtils.h"

#include <random>

namespace Cuda
{
    __host__ __device__ BidirectionalTransform::BidirectionalTransform()
    {
        jitterable.trans = vec3(0.0f);
        jitterable.rot = vec3(0.0f);
        jitterable.scale = vec3(1.0f);
        
        MakeIdentity();
    }

    __host__ void BidirectionalTransform::Randomise(const float xi0, const float xi1)
    {
        jitterable.trans.Randomise(xi0, xi1);
        jitterable.rot.Randomise(xi0, xi1);
        jitterable.scale.Randomise(xi0, xi1);

        EvaulateJitterables();
    }
    
    __host__ void BidirectionalTransform::FromJson(const ::Json::Node& node, const uint flags)
    {
        const auto transNode = node.GetChildObject("transform", flags);
        if (!transNode) { return; }        

        jitterable.trans.FromJson("pos", transNode, flags);
        jitterable.rot.FromJson("rot", transNode, ::Json::kSilent);
        jitterable.scale.FromJson("sca", transNode, ::Json::kSilent);

        EvaulateJitterables();
        
        // Build the transform
        Create(trans, rot, scale);
    }

    __host__ void BidirectionalTransform::ToJson(Json::Node& parentNode) const
    {
        auto transNode = parentNode.AddChildObject("transform");

        jitterable.trans.ToJson("pos", transNode);
        jitterable.rot.ToJson("rot", transNode);
        jitterable.scale.ToJson("sca", transNode);
    }

    __host__ BidirectionalTransform::BidirectionalTransform(const ::Json::Node& node, const uint flags)
    {
        FromJson(node, flags);
    }

    __host__ void BidirectionalTransform::EvaulateJitterables()
    {
        trans = jitterable.trans.Evaluate();
        rot = jitterable.rot.Evaluate();
        scale = jitterable.scale.Evaluate();
    }
}