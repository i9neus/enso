﻿#include "CudaSphere.cuh"

namespace Cuda
{
    __device__  bool Device::Sphere::Intersect(Ray& ray, HitCtx& hitCtx) const
    {
        const RayBasic localRay = RayToObjectSpace(ray.od, m_transform);

        // A ray intersects a sphere in at most two places which means we can find t by solving a quadratic
        float a = dot(localRay.d, localRay.d);
        float b = 2.0 * dot(localRay.d, localRay.o);
        float c = dot(localRay.o, localRay.o) - 1.0;

        float t0, t1;
        if (!quadraticSolve(a, b, c, t0, t1)) { return false; }

        if (t1 < t0)
        {
            float swap = t1;
            t1 = t0;
            t0 = swap;
        }

        float tNear = ray.tNear;
        HitPoint hit;
        if (t0 > 0.0 && t0 < tNear)
        {
            hit.n = localRay.o + localRay.d * t0;
            tNear = t0;
        }
        else if (t1 > 0.0 && t1 < tNear)
        {
            hit.n = localRay.o + localRay.d * t1;
            tNear = t1;
        }
        else { return false; }

        ray.tNear = tNear;
        hit.p = ray.HitPoint();
        hit.n = NormalToWorldSpace(hit.n, m_transform);

        hitCtx.Set(hit, false, vec2(0.0f), 1e-5f);

        return true;
    }

    __host__  Host::Sphere::Sphere()
        : cu_deviceData(nullptr)
    {
        m_hostData.m_transform.MakeIdentity();

        cu_deviceData = InstantiateOnDevice<Device::Sphere>(m_hostData.m_transform);
    }

    __host__ void Host::Sphere::OnDestroyAsset()
    {
        DestroyOnDevice(&cu_deviceData);
    }
}