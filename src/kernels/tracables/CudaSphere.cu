﻿#include "CudaSphere.cuh"
#include "generic/JsonUtils.h"

namespace Cuda
{     
    __device__  bool Device::Sphere::Intersect(Ray& ray, HitCtx& hitCtx) const
    {
        const RayBasic localRay = RayToObjectSpace(ray.od, m_transform);

        // A ray intersects a sphere in at most two places which means we can find t by solving a quadratic
        float a = dot(localRay.d, localRay.d);
        float b = 2.0 * dot(localRay.d, localRay.o);
        float c = dot(localRay.o, localRay.o) - 1.0;

        float t0, t1;
        if (!QuadraticSolve(a, b, c, t0, t1)) { return false; }

        if (t1 < t0)
        {
            float swap = t1;
            t1 = t0;
            t0 = swap;
        }

        float tNear = ray.tNear;
        HitPoint hit;
        if (t0 > 0.0 && t0 < tNear)
        {
            hit.n = localRay.o + localRay.d * t0;
            tNear = t0;
        }
        else if (t1 > 0.0 && t1 < tNear)
        {
            hit.n = localRay.o + localRay.d * t1;
            tNear = t1;
        }
        else { return false; }

        ray.tNear = tNear;
        hit.p = ray.HitPoint();
        hit.n = NormalToWorldSpace(hit.n, m_transform);

        hitCtx.Set(hit, false, vec2(0.0f), 1e-5f);

        return true;
    }

     __host__ AssetHandle<Host::RenderObject> Host::Sphere::Instantiate(const std::string& id, const AssetType& expectedType, const Json::Node& json)
     {
         if (expectedType != AssetType::kTracable) { return AssetHandle<Host::RenderObject>(); }

         return AssetHandle<Host::RenderObject>(new Host::Sphere(json), id);
     }

    __host__  Host::Sphere::Sphere(const Json::Node& node)
        : cu_deviceData(nullptr)
    {
        cu_deviceData = InstantiateOnDevice<Device::Sphere>();
        FromJson(node);
    }

    __host__ void Host::Sphere::FromJson(const Json::Node& parentNode)
    {
        SyncParameters(cu_deviceData, BidirectionalTransform(parentNode));
    }

    __host__ void Host::Sphere::OnDestroyAsset()
    {
        DestroyOnDevice(&cu_deviceData);
    }
}