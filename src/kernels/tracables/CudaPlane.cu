#include "hip/hip_runtime.h"
﻿#include "CudaPlane.cuh"
#include "generic/JsonUtils.h"

namespace Cuda
{
    __host__ void PlaneParams::ToJson(::Json::Node& node) const
    {
        node.AddValue("bounded", isBounded);
        node.AddValue("isDoubleSided", isDoubleSided);
        tracable.ToJson(node);
    }

    __host__ void PlaneParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        node.GetValue("bounded", isBounded, flags);
        node.GetValue("isDoubleSided", isDoubleSided, flags);
        tracable.FromJson(node, flags);
    }

    __device__  bool Device::Plane::Intersect(Ray& ray, HitCtx& hitCtx) const
    { 
        if (ray.flags & kRayLightProbe && m_params.tracable.renderObject.flags() & kRenderObjectExcludeFromBake) { return false; }
        
        const RayBasic localRay = RayToObjectSpace(ray.od, m_params.tracable.transform);

        // A ray intersects a sphere in at most two places which means we can find t by solving a quadratic
        if (fabs(localRay.d.z) < 1e-10) { return false; }

        float t = localRay.o.z / -localRay.d.z;
        if (t <= 0.0 || t >= ray.tNear) { return false; }

        float u = (localRay.o.x + localRay.d.x * t) + 0.5;
        float v = (localRay.o.y + localRay.d.y * t) + 0.5;

        if (m_params.isBounded && (u < 0.0 || u > 1.0 || v < 0.0 || v > 1.0)) { return false; }
        
        ray.tNear = t;
        //HitPoint hit = m_transform.HitToWorldSpace(HitPoint(ray.HitPoint(), vec3(0.0f, 0.0f, 1.0f)));
        //if (dot(hit.n, ray.od.o - hit.o) < 0.0f) { hit.n = -hit.n; }

        hitCtx.Set(HitPoint(ray.HitPoint(), 
                   NormalToWorldSpace((localRay.o.z < 0.0f && m_params.isDoubleSided) ? vec3(0.0f, 0.0f, -1.0f) : vec3(0.0f, 0.0f, 1.0f), m_params.tracable.transform)),
                   localRay.o.z < 0.0f && !m_params.isDoubleSided, 
                   vec2(u, v), 1e-5f, 
                   m_objects.lightId);

        return true;
    }

    const RenderObjectParams* Host::Plane::GetRenderObjectParams() const 
    { 
        return &m_params.tracable.renderObject; 
    }

    __host__ AssetHandle<Host::RenderObject> Host::Plane::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kTracable) { return AssetHandle<Host::RenderObject>(); }

        return CreateAsset<Host::Plane>(id, json);
    }

    // Constructor used to instantiate child objects e.g. from quad lights
    __host__  Host::Plane::Plane(const std::string& id, const uint flags) :
        Tracable(id)
    {        
        cu_deviceData = InstantiateOnDevice<Device::Plane>(id);
        RenderObject::SetRenderObjectFlags(flags);
    }

    // Constructor for user instantiations
    __host__  Host::Plane::Plane(const std::string& id, const ::Json::Node& node) :
        Tracable(id)
    {
        cu_deviceData = InstantiateOnDevice<Device::Plane>(id);
        FromJson(node, ::Json::kSilent);
    }

    __host__ void Host::Plane::OnDestroyAsset()
    {
        DestroyOnDevice(GetAssetID(), cu_deviceData);
    }

    __host__ void Host::Plane::FromJson(const ::Json::Node& node, const uint flags)
    {
        Host::Tracable::FromJson(node, flags);

        m_params.FromJson(node, flags);
        RenderObject::SetUserFacingRenderObjectFlags(m_params.tracable.renderObject.flags());

        SynchroniseObjects(cu_deviceData, PlaneParams(node, flags));
    }

    __host__ void Host::Plane::UpdateParams(const BidirectionalTransform& transform, const bool isBounded)
    {
        m_params = PlaneParams(transform, isBounded);
        SynchroniseObjects(cu_deviceData, m_params);
    }
}