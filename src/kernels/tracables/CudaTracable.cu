﻿#include "CudaTracable.cuh"
#include "../materials/CudaMaterial.cuh"
#include "generic/JsonUtils.h"

namespace Cuda
{

    __host__ __device__ TracableParams::TracableParams()
    {
        excludeFromBake = false;
    }

    __host__ void TracableParams::ToJson(::Json::Node& node) const
    {
        transform.ToJson(node);

        node.AddValue("excludeFromBake", excludeFromBake);
    }

    __host__ void TracableParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        transform.FromJson(node, flags);

        node.GetValue("excludeFromBake", excludeFromBake, flags);
    }
    
    __host__ void Host::Tracable::FromJson(const ::Json::Node& node, const uint flags)
    {
        Host::RenderObject::UpdateDAGPath(node);
        
        node.GetValue("material", m_materialId, flags);
    }
    
    __host__ void Host::Tracable::Bind(RenderObjectContainer& objectContainer)
    {
        if (m_materialId.empty())
        {
            Log::Error("Error: no material binding ID was specified for tracable '%s'.\n", GetAssetID());
            return;
        }
                
        // Get a handle to the material asset for this tracable
        m_materialAsset = GetAssetHandleForBinding<Host::Tracable, Host::Material>(objectContainer, m_materialId);        
    }

    __host__ void Host::Tracable::Synchronise()
    {
        Device::Tracable::Objects deviceObjects;
        deviceObjects.lightId = m_lightId;
        if (m_materialAsset)
        {
            deviceObjects.cu_material = m_materialAsset->GetDeviceInstance();
        }

        // Push the binding to the device
        SynchroniseObjects(static_cast<Device::Tracable*>(GetDeviceInstance()), deviceObjects);

        Log::Debug("Synchronised tracable '%s'.\n", GetAssetID());
    }
}