#include "hip/hip_runtime.h"
﻿#include "CudaLightProbeRegressionFilter.cuh"
#include "../CudaManagedArray.cuh"

#include "generic/JsonUtils.h"

#define kBlockSize 256
#define kMaxCoefficients 5

namespace Cuda
{
    __host__ __device__ LightProbeRegressionFilterParams::LightProbeRegressionFilterParams() :
        polynomialOrder(0),
        radius(1),
        isNullFilter(true)
    {

    }

    __host__ void LightProbeRegressionFilterParams::ToJson(::Json::Node& node) const
    {
        node.AddValue("polynomialOrder", polynomialOrder);
        node.AddValue("radius", radius);
        node.AddValue("isNullFilter", isNullFilter);
    }

    __host__ void LightProbeRegressionFilterParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        node.GetValue("polynomialOrder", polynomialOrder, flags);
        node.GetValue("radius", radius, flags);
        node.GetValue("isNullFilter", isNullFilter, flags);

        radius = clamp(radius, 1, 10);
    }

    __host__ Host::LightProbeRegressionFilter::LightProbeRegressionFilter(const ::Json::Node& node, const std::string& id) :
        m_gridSize(1), m_blockSize(1)
    {
        FromJson(node, Json::kRequiredWarn);

        node.GetValue("inputGridID", m_inputGridID, Json::kRequiredAssert);
        node.GetValue("inputGridHalfID", m_inputGridHalfID, Json::kNotBlank);
        node.GetValue("outputGridID", m_outputGridID, Json::kRequiredAssert);

        AssertMsgFmt(!GlobalAssetRegistry::Get().Exists(m_outputGridID), "Error: an asset with ID '%s' already exists'.", m_outputGridID.c_str());

        // Create some objects
        m_hostOutputGrid = AssetHandle<Host::LightProbeGrid>(m_outputGridID, m_outputGridID);

        m_hostPolyCoeffs = AssetHandle<Host::Array<vec3>>(new Host::Array<vec3>(m_hostStream), tfm::format("%s_polyCoeffs", id));

        m_hostRegressionWeights = AssetHandle<Host::Array<float>>(new Host::Array<float>(m_hostStream), tfm::format("%s_regressionWeights", id));
        m_hostRegressionWeights->Resize(1024 * 1024);
    }

    __host__ AssetHandle<Host::RenderObject> Host::LightProbeRegressionFilter::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kLightProbeFilter) { return AssetHandle<Host::RenderObject>(); }

        return AssetHandle<Host::RenderObject>(new Host::LightProbeRegressionFilter(json, id), id);
    }

    __host__ void Host::LightProbeRegressionFilter::FromJson(const ::Json::Node& node, const uint flags)
    {
        m_objects->params.FromJson(node, flags);

        Prepare();
    }

    __host__ void Host::LightProbeRegressionFilter::OnDestroyAsset()
    {
        m_hostOutputGrid.DestroyAsset();
        m_hostPolyCoeffs.DestroyAsset();
        m_hostRegressionWeights.DestroyAsset();
    }

    __host__ void Host::LightProbeRegressionFilter::Bind(RenderObjectContainer& sceneObjects)
    {
        m_hostInputGrid = sceneObjects.FindByID(m_inputGridID).DynamicCast<Host::LightProbeGrid>();
        if (!m_hostInputGrid)
        {
            Log::Error("Error: LightProbeRegressionFilter::Bind(): the specified input light probe grid '%s' is invalid.\n", m_inputGridID);
            return;
        }

        m_hostInputHalfGrid = nullptr;
        if (!m_inputGridHalfID.empty())
        {
            m_hostInputHalfGrid = sceneObjects.FindByID(m_inputGridHalfID).DynamicCast<Host::LightProbeGrid>();
            if (!m_hostInputHalfGrid)
            {
                Log::Error("Error: LightProbeRegressionFilter::Bind(): the specified half input light probe grid '%s' is invalid.\n", m_inputGridHalfID);
                return;
            }
        }

        Prepare();
    }

    __global__ void KernelRandomisePolynomialCoefficients(Host::LightProbeRegressionFilter::Objects* objects)
    {

    }

    __host__ void Host::LightProbeRegressionFilter::Prepare()
    {
        // Filter isn't yet bound, so do nothing
        if (!m_hostInputGrid || !m_hostOutputGrid) { return; }

        Assert(m_hostPolyCoeffs);

        // Establish the dimensions of the kernel
        auto& gridData = m_objects->gridData.Prepare(m_hostInputGrid, m_hostInputHalfGrid, m_hostOutputGrid);
        Assert(m_objects->gridData.coefficientsPerProbe <= kMaxCoefficients);

        m_objects->polyCoeffsPerCoefficient = cub(m_objects->params.polynomialOrder + 1);
        m_objects->polyCoeffsPerProbe = m_objects->polyCoeffsPerCoefficient * gridData.coefficientsPerProbe;
        m_objects->numPolyCoeffs = m_objects->polyCoeffsPerProbe * gridData.numProbes;

        // Resize the polynomial coefficient array as a power of two 
        if(m_hostPolyCoeffs->ExpandToNearestPow2(m_objects->numPolyCoeffs))
        {
            Log::Debug("Resized m_hostPolyCoeffs to %i\n", m_hostPolyCoeffs->Size());
        }

        // Initialise the output grid so it has the same dimensions as the input
        m_hostOutputGrid->Prepare(m_hostInputGrid->GetParams());
                
        m_objects->cu_polyCoeffs = m_hostPolyCoeffs->GetDeviceInstance();
        m_objects->cu_regressionWeights = m_hostRegressionWeights->GetDeviceInstance();

        m_objects.Upload();
    }

    __global__ void KernelComputeRegressionWeights(Host::LightProbeRegressionFilter::Objects* objects, const int probeStartIdx)
    {

    }

    __global__ void KernelApplyRegressionIteration(Host::LightProbeRegressionFilter::Objects* objects)
    {
        assert(objects->gridData.cu_inputGrid);
        assert(objects->gridData.cu_outputGrid);


    }

    __host__ void Host::LightProbeRegressionFilter::OnPostRenderPass()
    {
        // Filter isn't yet bound, so do nothing
        if (!m_hostInputGrid || !m_hostOutputGrid) { return; }

        // Pass-through filter just copies the data
        if (m_objects->params.isNullFilter)
        {
            m_hostOutputGrid->Replace(*m_hostInputGrid);
            return;
        }
    }

    __host__ std::vector<AssetHandle<Host::RenderObject>> Host::LightProbeRegressionFilter::GetChildObjectHandles()
    {
        std::vector<AssetHandle<Host::RenderObject>> objects;
        objects.emplace_back(m_hostOutputGrid);
        return objects;
    }

    __host__ void Host::LightProbeRegressionFilter::OnUpdateSceneGraph(RenderObjectContainer& sceneObjects)
    {
        if (m_hostInputGrid && m_hostOutputGrid &&
            m_hostInputGrid->GetParams() != m_hostOutputGrid->GetParams())
        {
            Prepare();
        }
    }
}