#include "hip/hip_runtime.h"
﻿#include "CudaLightProbeRegressionFilter.cuh"
#include "../CudaManagedArray.cuh"
#include "../CudaSampler.cuh"

#include "generic/JsonUtils.h"

#define kBlockSize 256
#define kMaxCoefficients 5

namespace Cuda
{
    __host__ __device__ LightProbeRegressionFilterParams::LightProbeRegressionFilterParams() :
        polynomialOrder(0),
        regressionRadius(1),
        reconstructionRadius(1),
        regressionIterations(1),
        learningRate(0.005f)
    {

    }

    __host__ void LightProbeRegressionFilterParams::ToJson(::Json::Node& node) const
    {
        node.AddValue("polynomialOrder", polynomialOrder);
        node.AddValue("regressionRadius", regressionRadius);
        node.AddValue("regressionIterations", regressionIterations);
        node.AddValue("reconstructionRadius", reconstructionRadius);
        node.AddValue("learningRate", learningRate);
        node.AddEnumeratedParameter("filterType", std::vector<std::string>({ "null", "box", "gaussian", "nlm" }), filterType);

        Json::Node nlmNode = node.AddChildObject("nlm");
        nlm.ToJson(nlmNode);
    }

    __host__ void LightProbeRegressionFilterParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        node.GetValue("polynomialOrder", polynomialOrder, flags);
        node.GetValue("regressionRadius", regressionRadius, flags);
        node.GetValue("regressionIterations", regressionIterations, flags);
        node.GetValue("reconstructionRadius", reconstructionRadius, flags);
        node.GetValue("learningRate", learningRate, flags);
        node.GetEnumeratedParameter("filterType", std::vector<std::string>({ "null", "box", "gaussian", "nlm" }), filterType, flags);

        Json::Node nlmNode = node.GetChildObject("nlm", flags);
        if (nlmNode) { nlm.FromJson(nlmNode, flags); }

        regressionRadius = clamp(regressionRadius, 0, 10);
        regressionIterations = clamp(regressionIterations, 1, 100);
        reconstructionRadius = clamp(reconstructionRadius, 0, 10);
        polynomialOrder = clamp(polynomialOrder, 0, 3);
    }

    __host__ Host::LightProbeRegressionFilter::LightProbeRegressionFilter(const ::Json::Node& node, const std::string& id)
    {
        FromJson(node, Json::kRequiredWarn);

        node.GetValue("inputGridID", m_inputGridID, Json::kRequiredAssert);
        node.GetValue("inputGridHalfID", m_inputGridHalfID, Json::kNotBlank);
        node.GetValue("outputGridID", m_outputGridID, Json::kRequiredAssert);

        AssertMsgFmt(!GlobalAssetRegistry::Get().Exists(m_outputGridID), "Error: an asset with ID '%s' already exists'.", m_outputGridID.c_str());

        // Create the output grid
        m_hostOutputGrid = AssetHandle<Host::LightProbeGrid>(m_outputGridID, m_outputGridID);

        // Create the buffers used by the regressor
        m_hostC = AssetHandle<Host::Array<vec3>>(new Host::Array<vec3>(m_hostStream), tfm::format("%s_C", id));
        m_hostD = AssetHandle<Host::Array<float>>(new Host::Array<float>(m_hostStream), tfm::format("%s_D", id));
        m_hostdLdC = AssetHandle<Host::Array<vec3>>(new Host::Array<vec3>(m_hostStream), tfm::format("%s_dLdC", id));
        m_hostW = AssetHandle<Host::Array<float>>(new Host::Array<float>(m_hostStream), tfm::format("%s_regressionWeights", id));

        // TODO: Make weight map dynamic
        m_hostW->Resize(1024 * 1024);
    }

    __host__ AssetHandle<Host::RenderObject> Host::LightProbeRegressionFilter::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kLightProbeFilter) { return AssetHandle<Host::RenderObject>(); }

        return AssetHandle<Host::RenderObject>(new Host::LightProbeRegressionFilter(json, id), id);
    }

    __host__ void Host::LightProbeRegressionFilter::FromJson(const ::Json::Node& node, const uint flags)
    {
        m_objects->params.FromJson(node, flags);

        Prepare();
    }

    __host__ void Host::LightProbeRegressionFilter::OnDestroyAsset()
    {
        m_hostOutputGrid.DestroyAsset();
        m_hostC.DestroyAsset();
        m_hostD.DestroyAsset();
        m_hostdLdC.DestroyAsset();
        m_hostW.DestroyAsset();
    }

    __host__ void Host::LightProbeRegressionFilter::Bind(RenderObjectContainer& sceneObjects)
    {
        m_hostInputGrid = sceneObjects.FindByID(m_inputGridID).DynamicCast<Host::LightProbeGrid>();
        if (!m_hostInputGrid)
        {
            Log::Error("Error: LightProbeRegressionFilter::Bind(): the specified input light probe grid '%s' is invalid.\n", m_inputGridID);
            return;
        }

        m_hostInputHalfGrid = nullptr;
        if (!m_inputGridHalfID.empty())
        {
            m_hostInputHalfGrid = sceneObjects.FindByID(m_inputGridHalfID).DynamicCast<Host::LightProbeGrid>();
            if (!m_hostInputHalfGrid)
            {
                Log::Error("Error: LightProbeRegressionFilter::Bind(): the specified half input light probe grid '%s' is invalid.\n", m_inputGridHalfID);
                return;
            }
        }

        Prepare();
    }

    __global__ void KernelRandomisePolynomialCoefficients(Device::Array<vec3>* coeffBuffer)
    {
        assert(coeffBuffer);
        assert(kKernelIdx < coeffBuffer->Size());

        PseudoRNG rng(HashOf(uint(kKernelIdx)));
        (*coeffBuffer)[kKernelIdx] = rng.Rand<0, 1, 2>() * 0.5f;
    }

    __host__ void Host::LightProbeRegressionFilter::Prepare()
    {
        // Filter isn't yet bound, so do nothing
        if (!m_hostInputGrid || !m_hostOutputGrid) { return; }

        Assert(m_hostC); // Sanity check

        // Establish the dimensions of the kernel
        auto& gridData = m_objects->gridData.Prepare(m_hostInputGrid, m_hostInputHalfGrid, m_hostOutputGrid);
        Assert(m_objects->gridData.coefficientsPerProbe <= kMaxCoefficients);
     
        // Precompute some values for the regression step
        m_objects->regression.radius = m_objects->params.regressionRadius;
        m_objects->regression.span = 2 * m_objects->regression.radius + 1;
        m_objects->regression.volume = cub(m_objects->regression.span);
        m_objects->regression.numMonomials = cub(m_objects->params.polynomialOrder + 1);

        if (m_objects->regression.volume < m_objects->regression.numMonomials)
        {
            Log::Error("Warning: the regression system with %i coeffients is under-determined given the window size of %i probes", m_objects->regression.numMonomials, m_objects->regression.volume);
        }

        // Precompute some values for the reconstruction step
        m_objects->reconstruction.radius = m_objects->params.reconstructionRadius;
        m_objects->reconstruction.span = 2 * m_objects->reconstruction.radius + 1;
        m_objects->reconstruction.volume = cub(m_objects->reconstruction.span);
        
        // Each coefficient stores the monomial coefficients of the fitted polynomial plus 2 additional coefficients which represent the max and min values of the kernel   
        m_objects->regrCoeffsPerSHCoeff = m_objects->regression.numMonomials + 2;
        m_objects->regrCoeffsPerProbe = m_objects->regrCoeffsPerSHCoeff * gridData.coefficientsPerProbe;
        m_objects->totalRegrCoeffs = m_objects->regrCoeffsPerProbe * gridData.numProbes;

        // Rather than store every weight for every probe, we do the regression step in batches to cap the memory needed
        m_objects->probesPerBatch = min(gridData.numProbes, int(m_hostW->Size() / m_objects->regression.volume));
        Log::Debug("Probes per batch: %i", m_objects->probesPerBatch);

        // Resize the polynomial coefficient array as a power of two 
        m_hostdLdC->ExpandToNearestPow2(m_objects->totalRegrCoeffs);
        if (m_hostC->ExpandToNearestPow2(m_objects->totalRegrCoeffs))
        {
            Log::Debug("Resized C/dLdC to %i", m_hostC->Size());
        }

        // Generate a precomputed set of matrices containing the monomial constants for the kernel
        PrecomputeMonomialMatrices();

        // Initialise the output grid so it has the same dimensions as the input
        m_hostOutputGrid->Prepare(m_hostInputGrid->GetParams());

        // Set the device objects and sync
        m_objects->cu_C = m_hostC->GetDeviceInstance();
        m_objects->cu_dLdC = m_hostdLdC->GetDeviceInstance();
        m_objects->cu_D = m_hostD->GetDeviceInstance();
        m_objects->cu_W = m_hostW->GetDeviceInstance();
        m_objects.Upload();

        // Generate some random numbers to seed the polynomial coefficients
        KernelRandomisePolynomialCoefficients << < (m_objects->totalRegrCoeffs + 255) / 256, 256, 0, m_hostStream >> > (m_objects->cu_C);

        // Compute initialisation data for the regression kernel
        auto& rk = m_regressionKernel; 
        constexpr int kMaxSharedMemory = 40 * 1024;
        constexpr int kMinBlockSize = 8;
        rk.blockSize = 256;
        do
        {
            rk.sharedMemoryBytes = m_objects->regression.volume * sizeof(float) * rk.blockSize;
            rk.blockSize >>= 1;
        }
        while(rk.sharedMemoryBytes >= kMaxSharedMemory && rk.blockSize >= kMinBlockSize);
        Assert(rk.blockSize >= kMinBlockSize);

        rk.gridSize = ((m_objects->probesPerBatch * m_objects->gridData.shCoeffsPerProbe * 3) + (rk.blockSize - 1)) / rk.blockSize;

        Log::Debug("Regression kernel:");
        Log::Debug("  - Grid size: %i", rk.gridSize);
        Log::Debug("  - Block size: %i", rk.blockSize);
        Log::Debug("  - Shared memory: %i", rk.sharedMemoryBytes);
    }

    __host__ void Host::LightProbeRegressionFilter::PrecomputeMonomialMatrices()
    {
        // Allocate some temporary memory
        const int numElements = m_objects->regression.volume * m_objects->regression.numMonomials;
        std::vector<float> D(numElements);
        const int radius = m_objects->regression.radius;
        const int polynomialOrder = m_objects->params.polynomialOrder;

        // Generate a monomial matrix for every point in the regression kernel. 
        // TODO: There's some redundancy here, but we can see to that later.
        for (int z = -radius, dIdx = 0; z <= radius; ++z)
        {
            const float nz = z / float(max(1, radius));
            for (int y = -radius; y <= radius; ++y)
            {
                const float ny = y / float(max(1, radius));
                for (int x = -radius; x <= radius; ++x)
                {
                    // Construct the monomial matrix
                    const float nx = x / float(max(1, radius));
                    float zExp = 1.0f;
                    for (int zt = 0, cIdx = 0; zt <= polynomialOrder; zt++)
                    {
                        float yExp = 1.0f;
                        for (int yt = 0; yt <= polynomialOrder; yt++)
                        {
                            float xExp = 1.0f;
                            for (int xt = 0; xt <= polynomialOrder; ++xt)
                            {
                                D[dIdx++] = xExp * yExp * zExp;
                                xExp *= nx;
                            }
                            yExp *= ny;
                        }
                        zExp *= nz;
                    }
                }
            }
        }

        // Upload the matrices to the Cuda array
        m_hostD->Upload(D);
    }

    __global__ void KernelComputeRegressionWeights(Host::LightProbeRegressionFilter::Objects* objectsPtr, const int probeStartIdx)
    {
        __shared__ Host::LightProbeRegressionFilter::Objects objects;
        if (kThreadIdx == 0)
        {
            assert(objectsPtr);
            objects = *objectsPtr;
            assert(objects.gridData.cu_inputGrid);
            assert(objects.gridData.cu_outputGrid);
        }

        __syncthreads();

        // Get the index of the probe in the grid and the sample in the kernel
        auto& gridData = objects.gridData;
        const auto& params = objects.params;

        // Compute the index and position of the probe and bail out if we're out of bounds
        const int probeIdx0 = probeStartIdx + kKernelIdx / objects.regression.volume;
        if (probeIdx0 >= gridData.numProbes) { return; }
        const ivec3 pos0 = GridPosFromProbeIdx(probeIdx0, objects.gridData.density);

        // Compute the sample position relative to the centre of the kernel
        const int probeIdxK = kKernelIdx % objects.regression.volume;
        const ivec3 posK = GridPosFromProbeIdx(probeIdxK, objects.regression.span) - ivec3(objects.regression.radius);

        // If the neighbourhood probe lies outside the bounds of the grid, set the weight to zero
        const int probeIdxN = gridData.cu_inputGrid->IdxAt(pos0 + posK);
        if (probeIdxN < 0)
        {
            (*objects.cu_W)[kKernelIdx] = 0.0f;
        }
        else
        {
            // Calculate the weight for the sample
            float weight;
            switch (params.filterType)
            {
            case kKernelFilterGaussian:
            {
                const float len = length(vec3(posK));
                if (len <= objects.regression.radius)
                {
                    weight = Integrate1DGaussian(len - 0.5f, len + 0.5f, objects.regression.radius);
                }
                break;
            }
            case kKernelFilterNLM:
            {
                weight = ComputeNLMWeight(gridData, params.nlm, pos0, posK);
                break;
            }
            default:
                weight = 1.0f;
            };

            (*objects.cu_W)[kKernelIdx] = weight;
        }
    }

    __global__ void KernelComputeRegressionIteration(Host::LightProbeRegressionFilter::Objects* objectsPtr, const int probeStartIdx)
    {
        /*
            p -> pixel values i.e. what we're regressing onto
            C -> polynomial coefficients
            D -> monomial constants over the spread of the kernel
            W -> kernel weights
        */

        extern __shared__ int __block[];
        float* pBlock = reinterpret_cast<float*>(__block);
        __shared__ Host::LightProbeRegressionFilter::Objects objects;
        __shared__  const float* D;

        if (kThreadIdx == 0)
        {
            assert(objectsPtr);
            objects = *objectsPtr;
            assert(objects.gridData.cu_inputGrid);
            assert(objects.gridData.cu_outputGrid);
            assert(objects.cu_D);
            assert(objects.cu_C);
            assert(objects.cu_W);

            D = objects.cu_D->GetData();
        }

        __syncthreads();

        // Get a pointer to the shared memory used to cache the kernel values for this channel
        float* p = &pBlock[kThreadIdx * objects.regression.volume];

        const auto& gridData = objects.gridData;
        const auto& params = objects.params;

        const int probeIdx0 = probeStartIdx + kKernelIdx / (gridData.shCoeffsPerProbe * 3);
        if (probeIdx0 >= gridData.numProbes) { return; }

        const int coeffIdx = (kKernelIdx / 3) % gridData.shCoeffsPerProbe;
        const int channelIdx = kKernelIdx % 3;        

        // Get pointers to the polynomial coefficients and associated partial derivatives
        int dataIdx0 = probeIdx0 * objects.regrCoeffsPerProbe + coeffIdx * objects.regrCoeffsPerSHCoeff;
        assert(dataIdx0 < objects.cu_C->Size());
        vec3* C = &(*objects.cu_C)[dataIdx0];
        vec3* dLdC = &(*objects.cu_dLdC)[dataIdx0];
        // Get a pointer to the weights
        float* W = &(*objects.cu_W)[(kKernelIdx / (gridData.shCoeffsPerProbe * 3)) * objects.regression.volume];

        // Fill the cache with the local pixel values
        float sumW = 0.0f;
        float maxP = -kFltMax, minP = kFltMax;
        const ivec3 pos0 = GridPosFromProbeIdx(probeIdx0, gridData.density);
        for (int z = -objects.regression.radius, pIdx = 0; z <= objects.regression.radius; ++z)
        {
            for (int y = -objects.regression.radius; y <= objects.regression.radius; ++y)
            {
                for (int x = -objects.regression.radius; x <= objects.regression.radius; ++x, pIdx++)
                {
                    ivec3 posK = pos0 + ivec3(x, y, z);
                    if (gridData.cu_inputGrid->IdxAt(posK) < 0)
                    {
                        p[pIdx] = -kFltMax;
                        continue;
                    }
                    p[pIdx] = gridData.cu_inputGrid->At(posK)[coeffIdx][channelIdx];
                    maxP = max(maxP, p[pIdx]);
                    minP = min(minP, p[pIdx]);
                    sumW += W[pIdx];
                }
            }
        }

        // Normalise the p-values
        for (int pIdx = 0; pIdx < objects.regression.volume; ++pIdx)
        {
            if (p[pIdx] != -kFltMax)
            {
                p[pIdx] = (p[pIdx] - minP) / max(1e-5f, maxP - minP);
            }
        }

        // Do the polynomial regression
        for (int itIdx = 0; itIdx < params.regressionIterations; ++itIdx)
        {
            // Clear the loss and derivatives
            float L2Loss = 0.0f;
            for (int t = 0; t < objects.regression.numMonomials; t++) { dLdC[t][channelIdx] = 0.0f; }

            // Loop over every element in the kernel and compute partial derivatives ready for the gradient descent
            for (int pIdx = 0; pIdx < objects.regression.volume; ++pIdx)
            {
                if (p[pIdx] == -kFltMax) { continue; }

                // Accumulate the sum of polynomial coefficients multiplied by the monomial constants associated with them
                float sigma = -p[pIdx];
                for (int cIdx = 0, dIdx = pIdx * objects.regression.numMonomials; cIdx < objects.regression.numMonomials; ++cIdx, ++dIdx)
                {
                    assert(dIdx < objects.cu_D->Size());
                    sigma += C[cIdx][channelIdx] * D[dIdx];
                }

                // Accumulate the partial derivatives for each constant
                for (int cIdx = 0, dIdx = pIdx * objects.regression.numMonomials; cIdx < objects.regression.numMonomials; ++cIdx, ++dIdx)
                {
                    dLdC[cIdx][channelIdx] += 2.0 * D[dIdx] * sigma * W[pIdx];
                }

                // Accumulate the weighted sum of the derivatives as the L2 loss
                L2Loss += sqr(sigma) * W[pIdx];
            }
            L2Loss /= sumW;

            // Perform the gradient descent step
            for (int cIdx = 0; cIdx < objects.regression.numMonomials; ++cIdx)
            {
                dLdC[cIdx][channelIdx] /= sumW;
                C[cIdx][channelIdx] -= params.learningRate * dLdC[cIdx][channelIdx] / max(L2Loss, 1e-2f);
            }
        }

        // Update the min/max components 
        C[objects.regression.numMonomials][channelIdx] = minP;
        C[objects.regression.numMonomials + 1][channelIdx] = maxP;
    }

    __global__ void KernelReconstructPolynomial(Host::LightProbeRegressionFilter::Objects* objectsPtr)
    {
        __shared__ Host::LightProbeRegressionFilter::Objects objects;
        __shared__  const float* D;
        if (kThreadIdx == 0)
        {
            assert(objectsPtr);
            objects = *objectsPtr;
            assert(objects.gridData.cu_inputGrid);
            assert(objects.gridData.cu_outputGrid);
            
            // Get the monomial matrix at the centre of the 
            D = &(objects.cu_D->GetData()[(objects.regression.volume / 2) * objects.regression.numMonomials]);
        }

        __syncthreads();

        // Probes -> SH coefficients -> Polynomial coefficients + max/min

        const auto& gridData = objects.gridData;
        const auto& params = objects.params;
        const int probeIdx0 = kKernelIdx / gridData.shCoeffsPerProbe;
        if (probeIdx0 >= objects.gridData.numProbes) { return; }
        const int coeffIdx = kKernelIdx % gridData.shCoeffsPerProbe;

        if (coeffIdx > 0)
        {
            gridData.cu_outputGrid->SetSHCoefficient(probeIdx0, coeffIdx, kZero);
            return;
        }

        const ivec3 pos0 = GridPosFromProbeIdx(probeIdx0, gridData.density);

        vec3 LSum(0.0f);
        int sumWeights = 0;
        float radiusNorm = max(1, objects.regression.radius);
        for (int z = -objects.reconstruction.radius; z <= objects.reconstruction.radius; ++z)
        {
            const float nz = -z / radiusNorm;
            for (int y = -objects.reconstruction.radius; y <= objects.reconstruction.radius; ++y)
            {
                const float ny = -y / radiusNorm;
                for (int x = -objects.reconstruction.radius; x <= objects.reconstruction.radius; ++x)
                {
                    const float nx = -x / radiusNorm;
                    ivec3 posK = pos0 + ivec3(x, y, z);
                    if (gridData.cu_inputGrid->IdxAt(posK) < 0) { continue; }

                    const int probeIdxK = ProbeIdxFromGridPos(posK, gridData.density);
                    const int dataIdxK = probeIdxK * objects.regrCoeffsPerProbe + coeffIdx * objects.regrCoeffsPerSHCoeff;
                    assert(dataIdxK < objects.cu_C->Size());
                    const vec3* C = &(*objects.cu_C)[dataIdxK];
                    
                    vec3 L(0.0f);                    
                    int t = 0;
                    float zExp = 1.0f;
                    for (int zt = 0; zt <= params.polynomialOrder; zt++)
                    {
                        float yExp = 1.0;
                        for (int yt = 0; yt <= params.polynomialOrder; yt++)
                        {
                            float xExp = 1.0;
                            for (int xt = 0; xt <= params.polynomialOrder; xt++, t++)
                            {
                                L += C[t] * xExp * yExp * zExp;
                                xExp *= nx;
                            }
                            yExp *= ny;
                        }
                        zExp *= nz;
                    }

                    // Denormalise
                    const vec3& kernelMin = C[t];
                    const vec3& kernelMax = C[t + 1];
                    L = kernelMin + L * (kernelMax - kernelMin);

                    // Accumulate
                    LSum += L;
                    sumWeights += 1;
                }
            }
        }
        
        gridData.cu_outputGrid->SetSHCoefficient(probeIdx0, coeffIdx, LSum / float(sumWeights));
    }

    __host__ void Host::LightProbeRegressionFilter::OnPostRenderPass()
    {
        // Filter isn't yet bound, so do nothing
        if (!m_hostInputGrid || !m_hostOutputGrid) { return; }

        // Pass-through filter just copies the data
        if (m_objects->params.filterType == kKernelFilterNull)
        {
            m_hostOutputGrid->Replace(*m_hostInputGrid);
            return;
        }

        for (int probeStartIdx = 0; probeStartIdx < m_objects->gridData.numProbes; probeStartIdx += m_objects->probesPerBatch)
        {
            // Populate the kernel weights buffer ready for the regression step
            int numElements = m_objects->probesPerBatch * m_objects->regression.volume;
            KernelComputeRegressionWeights << < (numElements + 255) / 256, 256, 0, m_hostStream >> > (m_objects.GetDeviceObject(), probeStartIdx);

            // Run the regression step
            KernelComputeRegressionIteration <<< m_regressionKernel.gridSize, m_regressionKernel.blockSize, m_regressionKernel.sharedMemoryBytes, m_hostStream >>> (m_objects.GetDeviceObject(), probeStartIdx);
        }       

        KernelReconstructPolynomial << < (m_objects->gridData.totalSHCoefficients + 255) / 256, 256, 0, m_hostStream >> > (m_objects.GetDeviceObject());
    }

    __host__ std::vector<AssetHandle<Host::RenderObject>> Host::LightProbeRegressionFilter::GetChildObjectHandles()
    {
        std::vector<AssetHandle<Host::RenderObject>> objects;
        objects.emplace_back(m_hostOutputGrid);
        return objects;
    }

    __host__ void Host::LightProbeRegressionFilter::OnUpdateSceneGraph(RenderObjectContainer& sceneObjects)
    {
        if (m_hostInputGrid && m_hostOutputGrid &&
            m_hostInputGrid->GetParams() != m_hostOutputGrid->GetParams())
        {
            Prepare();
        }
    }
}