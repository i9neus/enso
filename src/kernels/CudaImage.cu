#include "hip/hip_runtime.h"
﻿#include "CudaImage.cuh"

namespace Cuda
{
	template<typename T>
	__global__ void KernelSignalChange(Device::Image<T>* image, const unsigned int currentState, const unsigned int newState) { atomicCAS(image->AccessSignal(), currentState, newState); }

	template<typename T>
	__global__ void KernelClear(Device::Image<T>* image, const T value) 
	{ 
		//if (*(image->AccessSignal()) != kImageWriteLocked) { return; }
		
		image->Clear(KERNEL_COORDS_IVEC2, value);
	}
	
	template<typename T>
	__device__ void Device::Image<T>::Clear(const ivec2& xy, const T& value)
	{
		if(IsValid(xy))
		{
			At(xy) = value;
		}
	}

	template<typename T>
	__global__ void KernelCopyImageToD3DTexture(unsigned int clientWidth, unsigned int clientHeight, Device::Image<T>* image, hipSurfaceObject_t cuSurface)
	{
		if (*(image->AccessSignal()) != kImageReadLocked) { return; }

		unsigned int kx = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int ky = blockIdx.y * blockDim.y + threadIdx.y;

		if (kx >= clientWidth || ky >= clientHeight) { return; }

		int px = kx - clientWidth / 2 + image->Width() / 2;
		int py = ky - clientHeight / 2 + image->Height() / 2;

		if (px < 0 || px >= image->Width() || py < 0 || py >= image->Height()) { return; }

		surf2Dwrite(*reinterpret_cast<float4*>(&(image->At(px, py))), cuSurface, kx * 16, ky);
	}

	template<typename T>
	__host__ Host::Image<T>::Image(unsigned int width, unsigned int height, hipStream_t hostStream) :
		cu_deviceData(nullptr)
	{		
		// Prepare the host data
		m_hostData.m_width = width;
		m_hostData.m_height = height;
		m_hostData.m_accessSignal = kImageUnlocked;

		SafeAllocDeviceMemory(&m_hostData.cu_data, width * height);

		InstantiateOnDevice(&cu_deviceData, width, height, m_hostData.cu_data);

		m_hostStream = hostStream;
		m_block = dim3(16, 16, 1);
		m_grid = dim3((width + 15) / 16, (height + 15) / 16, 1);
	}

	template<typename T>
	__host__ void Host::Image<T>::SignalChange(hipStream_t otherStream, const unsigned int currentState, const unsigned int newState)
	{ 
		hipStream_t hostStream = otherStream ? otherStream : m_hostStream;
		KernelSignalChange << < 1, 1, 0, hostStream >> > (cu_deviceData, currentState, newState);
	}	

	template<typename T>
	__host__ void Host::Image<T>::Clear(const T& value)
	{ 
		KernelClear << < m_grid, m_block, 0, m_hostStream >> > (cu_deviceData, value);
	}

	template<typename T>
	__host__ void Host::Image<T>::OnDestroyAsset()
	{		
		DestroyOnDevice(&cu_deviceData);
		SafeFreeDeviceMemory(&m_hostData.cu_data);
	}

	template<typename T>
	__host__ void Host::Image<T>::CopyImageToD3DTexture(unsigned int clientWidth, unsigned int clientHeight, hipSurfaceObject_t cuSurface, hipStream_t hostStream)
	{		
		dim3 block(16, 16, 1);
		dim3 grid((clientWidth + 15) / 16, (clientHeight + 15) / 16, 1);
		
		SignalSetRead(hostStream);
		KernelCopyImageToD3DTexture << < grid, block, 0, hostStream >> > (clientWidth, clientHeight, cu_deviceData, cuSurface);
		SignalUnsetRead(hostStream);

		getLastCudaError("CopyImageToD3DTexture execution failed.\n");
	}
}