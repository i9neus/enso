#include "hip/hip_runtime.h"
﻿#include "CudaImage.cuh"

namespace Cuda
{
	template<typename T>
	__global__ void KernelSignalChange(Device::Image<T>* image, const unsigned int currentState, const unsigned int newState) { atomicCAS(image->AccessSignal(), currentState, newState); }

	template<typename T>
	__global__ void KernelClear(Device::Image<T>* image, const T& value) 
	{ 
		//if (*(image->AccessSignal()) != kImageWriteLocked) { return; }

		const uint kx = blockIdx.x * blockDim.x + threadIdx.x;
		const uint ky = blockIdx.y * blockDim.y + threadIdx.y;
		if (kx < image->Width() && ky < image->Height())
		{
			image->At(ky, ky) = value;
		}
		
		//image->Clear(KERNEL_COORDS_IVEC2, value);
	}
	
	template<typename T>
	__device__ void Device::Image<T>::Clear(const ivec2& xy, const T& value)
	{
		if (xy.x < m_width && xy.y < m_height)
		{
			At(xy.x, xy.y) = value;
		}
	}

	template<typename T>
	__global__ void KernelCopyImageToD3DTexture(unsigned int clientWidth, unsigned int clientHeight, Device::Image<T>* image, hipSurfaceObject_t cuSurface)
	{
		if (*(image->AccessSignal()) != kImageReadLocked) { return; }

		unsigned int kx = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int ky = blockIdx.y * blockDim.y + threadIdx.y;

		if (kx >= clientWidth || ky >= clientHeight) { return; }

		int px = kx - clientWidth / 2 + image->Width() / 2;
		int py = ky - clientHeight / 2 + image->Height() / 2;

		if (px < 0 || px >= image->Width() || py < 0 || py >= image->Height()) { return; }

		surf2Dwrite(*reinterpret_cast<float4*>(&(image->At(px, py))), cuSurface, kx * 16, ky);
	}

	template<typename T>
	__host__ Host::Image<T>::Image(unsigned int width, unsigned int height, hipStream_t hostStream)
	{
		m_width = width;
		m_height = height;
		m_hostStream = hostStream;
		m_accessSignal = kImageUnlocked;

		m_block = dim3(16, 16, 1);
		m_grid = dim3((m_width + 15) / 16, (m_height + 15) / 16, 1);

		checkCudaErrors(hipMalloc((void**)&cu_data, sizeof(T) * width * height));
		checkCudaErrors(hipMalloc((void**)&cu_deviceImage, sizeof(Device::Image<T>)));
		checkCudaErrors(hipMemcpy(cu_deviceImage, static_cast<Device::Image<T>*>(this), sizeof(Device::Image<T>), hipMemcpyHostToDevice));
	}

	template<typename T>
	__host__ void Host::Image<T>::SignalChange(hipStream_t otherStream, const unsigned int currentState, const unsigned int newState)
	{ 
		hipStream_t hostStream = otherStream ? otherStream : m_hostStream;
		KernelSignalChange << < 1, 1, 0, hostStream >> > (cu_deviceImage, currentState, newState);
	}	

	template<typename T>
	__host__ void Host::Image<T>::Clear(const T& value) 
	{ 
		KernelClear << < m_grid, m_block, 0, m_hostStream >> > (cu_deviceImage, value);
	}

	template<typename T>
	__host__ void Host::Image<T>::OnDestroyAsset()
	{
		SafeFreeDeviceMemory(&cu_data);
		SafeFreeDeviceMemory(&cu_deviceImage);
	}

	// The host CPU Sinewave thread spawner
	template<typename T>
	__host__ void Host::Image<T>::CopyImageToD3DTexture(unsigned int clientWidth, unsigned int clientHeight, hipSurfaceObject_t cuSurface, hipStream_t hostStream)
	{		
		dim3 block(16, 16, 1);
		dim3 grid((clientWidth + 15) / 16, (clientHeight + 15) / 16, 1);

		SignalSetRead(hostStream);
		KernelCopyImageToD3DTexture << < grid, block, 0, hostStream >> > (clientWidth, clientHeight, cu_deviceImage, cuSurface);
		SignalUnsetRead(hostStream);

		getLastCudaError("CopyImageToD3DTexture execution failed.\n");
	}
}