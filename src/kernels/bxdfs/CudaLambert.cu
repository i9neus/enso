#include "hip/hip_runtime.h"
﻿#include "CudaLambert.cuh"
#include "../CudaLightProbeGrid.cuh"
#include "../cameras/CudaLightProbeCamera.cuh"

#include "generic/JsonUtils.h"

namespace Cuda
{
    __host__ LambertBRDFParams::LambertBRDFParams(const ::Json::Node& node) : LambertBRDFParams() { FromJson(node, ::Json::kRequiredWarn); }

    __host__ void LambertBRDFParams::ToJson(::Json::Node& node) const
    {
        node.AddValue("lightProbeGridIndex", lightProbeGridIdx);
    }

    __host__ void LambertBRDFParams::FromJson(const ::Json::Node& node, const uint flags)
    {
        node.GetValue("lightProbeGridIndex", lightProbeGridIdx, ::Json::kRequiredWarn);
        lightProbeGridIdx = clamp(lightProbeGridIdx, 0, 1);
    }
    
    __device__ bool Device::LambertBRDF::Sample(const Ray& incident, const HitCtx& hitCtx, RenderCtx& renderCtx, vec3& extant, float& pdf) const
    {
        const vec2 xi = renderCtx.rng.Rand<0, 1>();

        // Sample the Lambertian direction
        vec3 r = vec3(SampleUnitDisc(xi), 0.0f);
        r.z = sqrt(1.0 - sqr(r.x) - sqr(r.y));

        pdf = r.z / kPi;
        extant = CreateBasis(hitCtx.hit.n) * r;

        return true;
    }
    __device__ bool Device::LambertBRDF::Evaluate(const vec3& incident, const vec3& extant, const HitCtx& hitCtx, float& weight, float& pdf) const
    {
        weight = dot(extant, hitCtx.hit.n) / kPi;
        pdf = weight;

        return true;
    }

    __device__ vec3 Device::LambertBRDF::EvaluateCachedRadiance(const HitCtx& hitCtx) const
    {
        return (cu_lightProbeGrid) ? cu_lightProbeGrid->Evaluate(hitCtx) : vec3(0.0f);
    }

    __host__ AssetHandle<Host::RenderObject> Host::LambertBRDF::Instantiate(const std::string& id, const AssetType& expectedType, const ::Json::Node& json)
    {
        if (expectedType != AssetType::kBxDF) { return AssetHandle<Host::RenderObject>(); }

        return AssetHandle<Host::RenderObject>(new Host::LambertBRDF(json), id);
    }

    __host__ Host::LambertBRDF::LambertBRDF(const ::Json::Node& parentNode) :
        cu_deviceData(nullptr)
    {
        cu_deviceData = InstantiateOnDevice<Device::LambertBRDF>();

        FromJson(parentNode, ::Json::kRequiredWarn);
    }

    __host__ void Host::LambertBRDF::OnDestroyAsset()
    {
        m_hostLightProbeGrid = nullptr;
        DestroyOnDevice(cu_deviceData);
    }

    __host__ void Host::LambertBRDF::FromJson(const ::Json::Node& parentNode, const uint flags)
    {      
        Host::BxDF::FromJson(parentNode, ::Json::kRequiredWarn);
        m_params.FromJson(parentNode, ::Json::kRequiredWarn);

        parentNode.GetValue("lightProbeGrid", m_lightProbeGridID, ::Json::kRequiredWarn);
    }

    __host__ void Host::LambertBRDF::Bind(RenderObjectContainer& sceneObjects)
    {
        if (m_lightProbeGridID.empty()) { return; }
        
        AssetHandle<Host::LightProbeCamera> probeCamera = sceneObjects.FindByID<Host::LightProbeCamera>(m_lightProbeGridID);
        if (!probeCamera)
        {
            Log::Error("Error: could not bind probe grid '%s' to Lambert BRDF '%s': camera not found.\n", m_lightProbeGridID, GetAssetID());
            return;
        }

        Device::LightProbeGrid* cu_grid = nullptr;
        if (probeCamera->GetLightProbeCameraParams().camera.isActive)
        {
            m_hostLightProbeGrid = probeCamera->GetLightProbeGrid(m_params.lightProbeGridIdx);
            cu_grid = m_hostLightProbeGrid->GetDeviceInstance();
            Log::Write("Bound probe grid %i from camera '%s' to Lambert BRDF '%s'.\n", m_params.lightProbeGridIdx, m_lightProbeGridID, GetAssetID());
        }

        Cuda::SynchroniseObjects(cu_deviceData, cu_grid);
    }

    __host__ void Host::LambertBRDF::OnUpdateSceneGraph(RenderObjectContainer& sceneObjects)
    {
        // Do a complete re-bind when the scene graph updates
        Bind(sceneObjects);
    }
}