#include "hip/hip_runtime.h"
﻿#include "CudaCommonIncludes.cuh"

void CudaImage::create(unsigned int width, unsigned int height)
{
	m_width = width;
	m_height = height;
	checkCudaErrors(hipMalloc((void**)&c_data, sizeof(float4) * m_width * m_height));
}

void CudaImage::destroy()
{
	checkCudaErrors(hipFree((void*)c_data));
	c_data = nullptr;
}